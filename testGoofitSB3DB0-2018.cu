#include "hip/hip_runtime.h"
//
//
// 
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/stat.h>
#include "Riostream.h"
#include <map>
#include <string>
//#include <boost/algorithm/string.hpp>
//#include <boost/algorithm/string/trim.hpp>f
#include <vector>
#include <math.h>
//#include <TCint.h>
#include <TGenericClassInfo.h> 
#include <TMath.h>
#include <TH1.h>
#include <TH2.h>
#include <TH3.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TH3F.h>
#include <TH1D.h>
#include <TH2D.h>
#include <TH3D.h>
#include <TF1.h>
#include <TF2.h>
#include <TF3.h>
#include <TMatrixD.h>
#include <TVirtualFitter.h>
#include <TROOT.h>
#include <TEnv.h>
#include <TSystem.h>
#include <TTree.h>
#include "TBranch.h"
#include <TApplication.h>
#include <TFile.h>
#include <TCanvas.h>
#include <TStyle.h> 
#include <TF1.h>  
#include <TF2.h> 
#include <TLorentzVector.h>
#include <TVector3.h>
#include "TDSet.h"
#include "TChain.h"
#include <time.h> 
#include <TSystemDirectory.h>
#include <TEfficiency.h>
#include <TLegend.h>
#include <TMinuit.h>
#include "Math/WrappedMultiTF1.h"
#include "TRandom.h" 
#include "TRandom3.h" 
#include  <TStopwatch.h>
#include "TH1F.h"
#include "TH2F.h"			// unused?
#include "TStyle.h"
#include "TCanvas.h"
#include <TGraphAsymmErrors.h>
#include <TFrame.h>
#include <TFitResult.h>
#include <TFitter.h>
#include "Fit/Fitter.h"
#include <TMatrixDSym.h>
#include <TBinomialEfficiencyFitter.h>
#include <TKDTreeBinning.h>
#include <TH2Poly.h>
//#if !defined(__CINT__) || defined(__MAKECINT__)
#include <RooFit.h>
#include <RooMinuit.h>
#include <RooRealVar.h>
#include <RooConstVar.h>
#include <RooPlot.h>
#include <RooDataSet.h>
#include <RooDataHist.h>
#include <RooHistPdf.h>
#include <RooRealVar.h>
#include <RooAbsReal.h>
#include <RooArgSet.h>
#include <RooRealProxy.h>
#include <RooCategoryProxy.h>
#include <RooAbsCategory.h>
#include <RooProdPdf.h>
#include <RooAddPdf.h>
#include <RooProduct.h>
#include <RooGaussian.h>
#include <RooCBShape.h>
#include <RooBifurGauss.h>
#include <RooPolynomial.h>
#include <RooChebychev.h>
#include <RooWorkspace.h>
#include <RooExponential.h>
#include <RooBernstein.h>
#include <RooErrorVar.h>
#include <RooFitResult.h>
#include <RooRangeBinning.h>
#include <RooBinning.h>
#include <TRatioPlot.h>
#include "GBRMath.h"
#include "RooDoubleCBFast.h"
#include "RooBernsteinSideband.h"
#include <sys/time.h>
#include <sys/times.h>
#include <iostream>
//#endif
// GooFit stuff
#include <goofit/Application.h>
#include <goofit/FitManager.h>
#include <goofit/PDFs/GooPdf.h>
#include <goofit/PDFs/basic/GaussianPdf.h>
#include <goofit/PDFs/basic/PolynomialPdf.h>
#include <goofit/PDFs/combine/ProdPdf.h>
#include <goofit/PDFs/combine/ConvolutionPdf.h>
#include <goofit/PDFs/combine/EventWeightedAddPdf.h>
#include <goofit/PDFs/combine/MappedPdf.h>
#include <goofit/PDFs/combine/AddPdf.h>
#include <goofit/UnbinnedDataSet.h>
#include <goofit/fitting/FitManagerMinuit1.h>
#include <goofit/Variable.h>
//#include <goofit/PDFs/mypdf/EffiBernsteinPdf.h>
//#include <goofit/PDFs/mypdf/EffiTestBernsteinPdf.h>
//#include <goofit/PDFs/mypdf/EffiPolynomialPdf.h>
//#include <goofit/PDFs/mypdf/BernsteinPdf.h>
#include <goofit/PDFs/mypdf/RGaussianPdf.h>
#include <goofit/PDFs/mypdf/SimpleCheby2Pdf.h>
#include <goofit/PDFs/mypdf/ErfcMassPdf.h>
#include <goofit/PDFs/mypdf/BernsteinTestPdf.h>
#include <goofit/PDFs/mypdf/FastBernsteinPdf.h>
//#include <goofit/PDFs/mypdf/NormProdEffiPdf.h>
//#include <goofit/PDFs/mypdf/NormProdEffiTestPdf.h>
//#include "ExpGausPEEPdf.h" 
//#include "ExpGausMPdf.h" 
//#include "ExpGausWithIntPdf.h"
//#include "ExpGausPEEfixSigmaPdf.h" 
//#include "ExpGausProdBPdf.h"
//#include "ExpGausProdEffiBPdf.h"
//#include "ExpGausPEESigmaBPdf.h" 
//#include "PolyEffiPdf.h" 
//#include "ErfcPolyPdf.h"
//#include "ErfcMassPdf.h"
//#include "SigmoidB0Pdf.h"
//#include <goofit/PDFs/mypdf/SigmoidB0Pdf.h>
//#include "ErfEffiBpPdf.h"
//#include "SigmoidGausPdf.h"
//#include "GooFit/BivarGaussianConstrPdf.h"
//#include "GooFit/TrivarGaussianConstrPdf.h"
// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 

using namespace std; 
using namespace GooFit;
using namespace ROOT;
using namespace RooFit;

void FitSBModel();
void CreateInputHistoFile();
int RunEra=2018;
char MCW2MassTXT[10] ="MCw";
char MCW2DirTXT[20]  ="reweightV2";
char MCW3MassTXT[20] ="MCw_scaleErr";
char MCW3DirTXT[20]  ="reweightV3";
char MCW4MassTXT[30] ="MCw_scaleErr_noIP2D_xgbv4";
char MCW4DirTXT[20]  ="reweightV4";
char MCW5MassTXT[30] ="MCw_scaleErr_noIP2D_xgbv5";
char MCW5DirTXT[20]  ="reweightV5";
double FitMassSpectrum(UnbinnedDataSet* dataMass, TCanvas* c2, TH1D* masHist, TH1D* pdfHist, TH1D*sigHist, TH1D* bckHist, int MaxDegree);
double FitMassSpectrumRoofit(RooDataSet* RooDataMass, TCanvas* c2, TH1D* masHist, TH1D* pdfHist, TH1D*sigHist, TH1D* bkgHist, int MaxDegreeBckg);
RooGaussian* _constrainVar(RooRealVar *var,RooWorkspace *w=0);
float*  _getFittedVar(const char* varName,RooWorkspace *w=0);
void replaceAll(std::string& str, const std::string& from, const std::string& to) ;
void replaceChar(char * txt, const char * txt1, const char * txt2) ;

//R__LOAD_LIBRARY(libRooBernsteinSideband)

//class RooBernsteinSideband;

bool minuit1;
bool wrongTagged = false;
bool SetMinuit2  = false;
bool Folded      = false;
bool integral    = false;
bool MCW2         = false;
bool MCW3         = false;
bool MCW4         = false;
bool MCW5         = false;

  char RecoDir[100]                 =  "~/p5prime/[RunEra]/skims/newphi/noIP2D/";// RunEra  will be set after...
//  char RecoDir[100]                 =  "~/p5prime/[RunEra]/skims/newphi/fixBkg/";// RunEra  will be set after...
  char InputRecoB0TreeName[10]	    = "ntuple";
  char OutputRecoB0TreeName[10]	    = "ntuple";
  char InputFileNameRecoB0[300]     = "[RunEra]Data_All_finalSelection.root";// RunEra  will be set after...
  char ListParName[400] 	    =  "ListParValues-[RunEra]-Q2Bin-2-Bins-.txt";
  char ListPloName[400] 	    =  "ListParValues-[RunEra]-Q2Bin-2-Bins-.plo";
  char ListParNorm[410] 	    =  "ListParValues-[RunEra]-Q2Bin-2-Bins-.txt_norm";
  char ListPloNorm[410] 	    =  "ListParValues-[RunEra]-Q2Bin-2-Bins-.plo_norm";
  char FitStraName[400] 	    =  "namelist-[RunEra]-Q2Bin-2-Bins-.stra";
  char OutFileName[400] 	    =  "testGoofitSB3DB0-[RunEra]-Q2Bin-1.root";
  char OutFileNameInputHisto[300]   =  "testGoofitSB3DB0-[RunEra]-InputHisto-Q2Bin-1.root";
  char OutSaveFileName[400]	    =  "";
  char PDFNameRecoHisto[350]	    =  "B0-RecoHist-[RunEra]-Q2Bin-1.pdf";
  char PDFNameGeneHisto[350]	    =  "B0-GeneHist-[RunEra]-Q2Bin-1.pdf";
  char PNGNameMassHist[350]	    =  "B0-MassCheck-[RunEra]-Q2Bin-1.png";
  char PNGNameMassCheck[350]	    =  "B0-MassHist-[RunEra]-Q2Bin-1.png";
  char PNGNameMassQ2Hist[350]	    =  "B0-MassHist-[RunEra]-Q2Bin-1.png";
  char PNGNameProjXYHist[350]	    =  "";
  char PNGNameProjZYHist[350]	    =  "";
  char PNGNameProjZXHist[350]	    =  "";
  char ProjectTXT[300]		    =  "";
  char SigmaMethodTXT[100]	    =  "";
  char TaggedVarTXT[100]	    =  "";
  char FoldedTXT[100]		    =  "";
//   char fitMassFileName[300]         =  "~/p5prime/massFits/results_fits_[RunEra]_fM_newbdt.root";// RunEra  will be set after...
//   char fitMassFileNameJpsi[300]     =  "~/p5prime/massFits/results_fits_[RunEra]_fM_Jpsi_newbdt.root";// RunEra  will be set after...
//   char fitMassFileNamePsi[300]      =  "~/p5prime/massFits/results_fits_[RunEra]_fM_Psi_newbdt.root";// RunEra  will be set after...
//   char fitMassFileNameQ2Bin7[300]   =  "~/p5prime/massFits/results_fits_[RunEra]_fM_newbin7.root";// RunEra  will be set after...
  char fitMassFileName[300]         =  "~/p5prime/massFits/noIP2D/xgbv8/results_fits_[RunEra]_fM.root";// RunEra  will be set after...
  char fitMassFileNameJpsi[300]     =  "~/p5prime/massFits/noIP2D/xgbv8/results_fits_[RunEra]_fM_Jpsi.root";// RunEra  will be set after...
  char fitMassFileNamePsi[300]      =  "~/p5prime/massFits/noIP2D/xgbv8/results_fits_[RunEra]_fM_Psi.root";// RunEra  will be set after...
  char FMTNSigma1L[10]		    ="";
  char FMTNSigma2L[10]		    ="";
  char FMTNSigma1R[10]		    ="";
  char FMTNSigma2R[10]		    ="";
//  char fitMassFileName[100]         =  "results_fits_[RunEra].root";
//  char fitMassFileName[100]         =  "rf607_fitresult.root";

  
  TFile*OutFile = 0;

 

  char PDFNameMass[350]          = "B0-Mass-[RunEra]-global.pdf";
  char PDFNameFitSB3D[350]       = "B0-FitSB3DMass-[RunEra]-global.pdf";

  char PNGNameFitSB3D[350]       = "B0-FitSB3D-[RunEra]-global.png";
  char PNGNameFitSB3DMass[350]   = "B0-FitSB3DMass-[RunEra]-global.png";
  char PNGNameFitSB3DProjX[350]  = "B0-FitSB3DProjX-[RunEra]-global.png";
  char PNGNameFitSB3DProjY[350]  = "B0-FitSB3DProjY-[RunEra]-global.png";
  char PNGNameFitSB3DProjZ[350]  = "B0-FitSB3DProjZ-[RunEra]-global.png";
  char testo[300]     = "" ;
  float MarkerSizeSet = 0.35;
  int   PlotLineWidth = 1.;
//                                        0      1      2      3       4       5      6       7
  std::vector<double> fM_sigmas_2016 = {0.023, 0.015, 0.017, 0.013, 0.0005 , 0.010, 0.0018, 0.013};
  std::vector<double> fM_sigmas_2017 = {0.018, 0.014, 0.015, 0.010, 0.0004 , 0.008, 0.0016, 0.011};
  std::vector<double> fM_sigmas_2018 = {0.015, 0.010, 0.011, 0.008, 0.00027, 0.006, 0.0011, 0.008};
  double fM_sigmas = -99.;
//============================
// maxDegree START
// now defined in NAMELIST 
//============================
  int maxDegree1 =0;
  int maxDegree2 =0;
  int maxDegree3 =0;
// Mass Spectrum Bernstein   
  int maxDegree  =0;
  
  int fixParam   =1;
//============================
// now defined in NAMELIST 
// maxDegree END
//============================


//
// Il Bin in q^2 !!!!
//   
  double Q2Min = 0.; 
  double Q2Max = 0.; 
  int    Q2Bin = -1;
//=================  
// Number of Normalization Integrals
//=================
  int    NormInteg = 11;
//=================
//=================
  int SETNumBinsX=100;
  int SETNumBinsY=100;
  int SETNumBinsZ=100;
//=================
//=================
  double SetMinRatio=0;
  double SetMaxRatio=3;
  double SetMinProj=0;
//   
//=================
//=================
// NFact!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  int   NFact = 4; 
// NFact!!!!!!!!!!!!!!!!!!!!!!!!!!!!
//=================
//=================
//     MCToys
//=================

  int   NFactGen = 1;

//===================================================
//=====significance cut definition for params  ======
//=====> should be redefined for each Q2Bin!  <======
//===================================================
       double CutSignificance   = 2;
//===================================================
//===================================================
  int SigmaProbSign=1;
//  
  int FitPrintLevel=1;
  bool boolHesse = true;
  bool AutoFixPar= false;
  bool FirstFit  = false;
  int  iFitLoop  = 3;
  double xCoeffNorm =0.0;
  int    xCoeffIndex=-1;
//=================
//=================
//=================
//=================
  double ParMin =  0.;
//  double ParMax =  100.;
  double ParMax =  10000;
  double RndMin =  0.01;
  double RndMax =  0.1;
//=================
//=================
//=================
//=================

  TCanvas *csignstudy=0;
  double  NumFittedData = -99;
  double  tagged_mass_rangeMin=5.0;
  double  tagged_mass_rangeMax=5.6;

  double XMinSign = 4.9;
  double XMaxSign = 5.6;
//  double B0Mass   = 5.27962;
  double B0Mass   = 5.27958;
  double B0Sigma  = 0.030;
  double JPsiMass = 3.096916;
  double PsiPMass = 3.686109;
  double piMass = 0.13957039;
  double kMass = 0.493677;
  double BpMass = 5.2791;
//  double KstarMass = 0.892;
  double KstarMass = 0.896;

  double XMinSBL  = 0.;
  double XMaxSBL  = 0.;
  double XMinSBR  = 0.;
  double XMaxSBR  = 0.;
//
  double NSigma1L = 0.;
  double NSigma2L = 0.;
  double NSigma1R = 0.;
  double NSigma2R = 0.;
  
  float HistMassL1 = 4.935;
  float HistMassL2 = 5.65;
//
//   double NSigmaSBL = -2.;
//   double NSigmaSBR = 0;
//   double BiasSB   = 6;
  double XMinCosThetaL	       = -1.;
  double XMaxCosThetaL	       =  1.;
  double XMinCosThetaK	       = -1.;
  double XMaxCosThetaK	       =  1.;
  double XMinPhi	       =-TMath::Pi();
  double XMaxPhi	       = TMath::Pi();

  double XMinCosThetaLUnfolded = -1.;
  double XMaxCosThetaLUnfolded =  1.;
  double XMinCosThetaKUnfolded = -1.;
  double XMaxCosThetaKUnfolded =  1.;
  double XMinPhiUnfolded       =-TMath::Pi();
  double XMaxPhiUnfolded       = TMath::Pi();

  int	 xCosLHBin =  25;
  int	 xCosKHBin =  25;
  int	 xPhiHBin  =  25;
  double BinWCosThetaL= 0;
  double BinWCosThetaK= 0;
  double BinWPhi      = 0;
  
  
  double xMinQMuMu = 1.;
  double xMaxQMuMu = 19.;
  double NSigma  = 3.;
//  double XMinSignW = XMinSign;
//  double XMaxSignW = XMaxSign;
//  double XMinSignW = B0Mass - NSigma*B0Sigma;
//  double XMaxSignW = B0Mass + NSigma*B0Sigma;
  double NSignInt2Sigma =0.;
  double NBckgInt2Sigma =0.;
  double XLeftSet =0.;
  double XRightSet =0.;
  double XStepSign = 0.0025;
  double XStepMinuit = 0.00001;
  float xMassHBin = (XMaxSign -XMinSign)/XStepSign;
  float xQ2HBin   = (xMaxQMuMu -xMinQMuMu)/0.1;
  double XHScale = 10;
 
  
  double yieldSignal = 0;
  double yieldBckg   = 0;
//   double ParMin = -1000;
//   double ParMax =  1000;
//   double RndMin = -0.1;
//   double RndMax =  0.1;
//  double c_const       = 0.0299792458;

  

  float xMassHBin2   =  xMassHBin /5; // plot only!
  GooFit::Observable xMass("xMass",XMinSign,XMaxSign) 	     ;
  
  TH1D* HxMass         = new TH1D( "HxMass"     , "B^{0} Mass"		 ,	      xMassHBin2, XMinSign,  XMaxSign);
  TH1D* HxMassQ2       = new TH1D( "HxMassQ2"   , "B^{0} Mass"		 ,	      xMassHBin2, XMinSign,  XMaxSign);
  TH1D* HxMassQ2SB     = new TH1D( "HxMassQ2SB" , "B^{0} Mass"		 ,	      xMassHBin2, XMinSign,  XMaxSign);
  TH1D* pdfHxMass      = new TH1D( "pdfHxMass"  , "B^{0} Mass Fit"	 ,  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1D* sigHxMass      = new TH1D( "sigHxMass"  , "B^{0} Mass Fit"	 ,  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1D* bkgHxMass      = new TH1D( "bkgHxMass"  , "B^{0} Mass Fit"	 ,  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1D* pdfHxMassQ2    = new TH1D( "pdfHxMassQ2", "B^{0} Mass Fit Q2 Bin",  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1D* sigHxMassQ2    = new TH1D( "sigHxMassQ2", "B^{0} Mass Fit Q2 Bin",  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
  TH1D* bkgHxMassQ2    = new TH1D( "bkgHxMassQ2", "B^{0} Mass Fit Q2 Bin",  XHScale * xMassHBin , xMass.getLowerLimit(), xMass.getUpperLimit());
//
  TH2D* HxMassVsCosL   = new TH2D( "HxMassVsCosL","B^{0} Mass%CosL"    ,(int)xMassHBin2, XMinSign,  XMaxSign, NFact*xCosLHBin, XMinCosThetaL, XMaxCosThetaL);
  TH2D* HxMassVsCosK   = new TH2D( "HxMassVsCosK","B^{0} Mass%CosK"    ,(int)xMassHBin2, XMinSign,  XMaxSign, NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK);
  TH2D* HxMassVsPhi    = new TH2D( "HxMassVsPhi", "B^{0} Mass%Phi"     ,(int)xMassHBin2, XMinSign,  XMaxSign, NFact*xPhiHBin , XMinPhi, XMaxPhi);


//TH1D* pdfHist        = new TH1D( "pdfHist", "B^{0} Mass Fit",     xMassHBin2, XMinSign,  XMaxSign);
//TH1D* sigHist        = new TH1D( "sigHist", "B^{0} Mass Fit",     xMassHBin2, XMinSign,  XMaxSign);
//TH1D* bkgHist        = new TH1D( "bkgHist", "B^{0} Mass Fit",     xMassHBin2, XMinSign,  XMaxSign);
// Goofit Reco Observables..
//   GooFit::Observable xCosThetaL ("xCosThetaL"	    ,XMinCosThetaL,XMaxCosThetaL)    ;
//   GooFit::Observable xCosThetaK ("xCosThetaK"	    ,XMinCosThetaK,XMaxCosThetaK)    ;
//   GooFit::Observable xPhiKstMuMu("xPhiKstMuMu"      ,XMinPhi,XMaxPhi)  ; 
//  GooFit::Observable xQ2MuMu("xQ2MuMu",xMinQMuMu    ,xMaxQMuMu)                      ;

RooRealVar *tagged_mass=0;
RooRealVar *mumuMass=0;
RooRealVar *mumuMassE=0;
RooAbsPdf  *bkg_mass_sb           =0;
RooAbsPdf  *bkg_exp               =0;
TTree* RecoB0TreeOut  =0;
TFile*OutFileInputHisto;
double EffiFunc3D(Double_t *var, Double_t *par);
double EffiFunc2D(Double_t *var, Double_t *par);
GooFit::Application *app_ptr;
std::map<std::string, std::string>  ReadNamelist(int argc, char** argv);
//
TRatioPlot* RatioDataModel3DX = 0;
TRatioPlot* RatioDataModel3DY = 0; 
TRatioPlot* RatioDataModel3DZ = 0; 
//      
Minuit1 * Minuit = 0;
TMatrixD * covMatrix=0;
RooBernsteinSideband * BernSideBand =0;
RooRealVar* ctL = new RooRealVar("ctL", "ctL",  XMinCosThetaK,XMaxCosThetaK);
RooRealVar* ctK = new RooRealVar("ctK", "ctK",  XMinCosThetaL,XMaxCosThetaL);
RooRealVar* phi = new RooRealVar("phi", "phi",  XMinPhi,XMaxPhi);
//
//==========================================
// Adaptive Binning...
//==========================================
TKDTreeBinning* RecoAdaptBinsC = 0;
TKDTreeID* TKDTreeIDC =0;
int   xAdaptNumBinC = 1;
int   MinContAdaptBin = 5;  
int   NDim = 3;
//
//==========================================
//==========================================
//=========    MAIN    =====================
//==========================================
//==========================================

int main (int argc, char** argv) {
//gSystem->Load("libRIO.so");
//gSystem->Load("libTree.so");
gSystem->Load("libRooDoubleCBFast.so");
//gSystem->Load("RooBernsteinSideband.so");


// if (argc>1 ){
//     Q2Bin = (int) (*argv)[1];
// }
//     cout<<Q2Bin<<endl;
//     cout<<argv[0]<<endl;
//     exit(1);

if (argc<=1 ){
    cout<<"Q2Bin not set"<<endl;
    cout<<"Usage: "<<argv[0]<< " QBin2 [where QBin2=0,1,2,3,4,5,6,7,8]\n or... \n"<<endl;
    cout<<"Usage: "<<argv[0]<< " QBin2 [where QBin2=0,1,2,3,4,5,6,7,8] mcw[2,3,4] [for the MC reweighting option version [2,3,4]] \n"<<endl;
    exit(1);
}   


 
switch ( *argv[1] ) {

  case '0' : 
   Q2Min = 1.; 
   Q2Max = 2.;
   Q2Bin = 0;
   if(RunEra==2016) CutSignificance =2;
   if(RunEra==2017) CutSignificance =2;
   if(RunEra==2018) CutSignificance =2;
//    xCosLHBin =   8;
//    xCosKHBin =   15;
//    xPhiHBin  =   8;
//       xCosLHBin =  25;
//       xCosKHBin =  25;
//       xPhiHBin  =  25;
// xCosLHBin =   5;
// xCosKHBin =   5;
// xPhiHBin  =   5;
    break;
  case '1' : 
   Q2Min = 2.; 
   Q2Max = 4.3; 
   Q2Bin = 1;
   if(RunEra==2016) CutSignificance =2;
   if(RunEra==2017) CutSignificance =2;
   if(RunEra==2018) CutSignificance =2;
//    xCosLHBin =  25;
//    xCosKHBin =  25;
//    xPhiHBin  =  25;
    break;
  case '2' : 
   Q2Min = 4.3; 
   Q2Max = 6.; 
   Q2Bin = 2;
   if(RunEra==2016) CutSignificance =2;
   if(RunEra==2017) CutSignificance =3;
   if(RunEra==2018) CutSignificance =3;
//    xCosLHBin =  5;
//    xCosKHBin =  5;
//    xPhiHBin  =  8;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
    break;
  case '3' : 
   Q2Min = 6.;  
   Q2Max = 8.68; 
   Q2Bin = 3;
   if(RunEra==2016) CutSignificance =2;
   if(RunEra==2017) CutSignificance =2;
   if(RunEra==2018) CutSignificance =3;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
//    xCosLHBin =  4;
//    xCosKHBin =  4;
//    xPhiHBin  =  7;
    break;
  case '4' : 
   Q2Min = 8.68; 
   Q2Max = 10.09; 
   Q2Bin = 4;
   sprintf(fitMassFileName,fitMassFileNameJpsi);
   if(RunEra==2016) CutSignificance =3;
   if(RunEra==2017) CutSignificance =3;
   if(RunEra==2018) CutSignificance =4.;
//   CutSignificance =4;
//    xCosLHBin =  5;
//    xCosKHBin =  5;
//    xPhiHBin  =  8;
//    xCosLHBin =  25;
//    xCosKHBin =  25;
//    xPhiHBin  =  25;
    break;
  case '5' :  
   Q2Min = 10.09; 
   Q2Max = 12.86; 
   Q2Bin = 5;
   if(RunEra==2016) CutSignificance =3;
   if(RunEra==2017) CutSignificance =3;
   if(RunEra==2018) CutSignificance =3;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
//    xCosLHBin =  5;
//    xCosKHBin =  5;
//    xPhiHBin  =  7;
    break;
  case '6' : 
   Q2Min = 12.86; 
   Q2Max = 14.18; 
   Q2Bin = 6;
   sprintf(fitMassFileName,fitMassFileNamePsi);
//   CutSignificance =4;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
    break;
  case '7' : 
   Q2Min = 14.18; 
   Q2Max = 16.; 
   Q2Bin = 7;
//   sprintf(fitMassFileName,fitMassFileNameQ2Bin7);
   if(RunEra==2016) CutSignificance =2;
   if(RunEra==2017) CutSignificance =2;
   if(RunEra==2018) CutSignificance =2;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
    break;
  case '8' : 
   Q2Min = 16; 
   Q2Max = 19.; 
   Q2Bin = 8;
//    xCosLHBin = 25;
//    xCosKHBin = 25;
//    xPhiHBin  = 25;
    break;

  default : 
    // Process for all other cases.
    cout<<"Q2Bin not set correctly!!!"<<endl;
    cout<<"Usage: "<<argv[0]<< " QBin2 [where QBin2=0,1,2,3,4,5,6,7,8]\n or... \n"<<endl;
    cout<<"Usage: "<<argv[0]<< " QBin2 [where QBin2=0,1,2,3,4,5,6,7,8] mcw[2,3,4] [for the MC reweighting option version [2,3,4]] \n"<<endl;
    exit(1);

}
   if (argc>2 && ((strcmp(argv[2],"MCW2") == 0)||(strcmp(argv[2],"mcw2") == 0)) ){
    MCW2=true;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"Setting the option: MC reweighting 2"<<std::endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    sprintf(RecoDir,"%s/%s",RecoDir,MCW2DirTXT);
    replaceChar(fitMassFileName,".root",Form("_%s.root",MCW2MassTXT));
   }
   if (argc>2 && ((strcmp(argv[2],"MCW3") == 0)||(strcmp(argv[2],"mcw3") == 0)) ){
    MCW3=true;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"Setting the option: MC reweighting 3"<<std::endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    sprintf(RecoDir,"%s/%s",RecoDir,MCW3DirTXT);
    replaceChar(fitMassFileName,".root",Form("_%s.root",MCW3MassTXT));
   }
   if (argc>2 && ((strcmp(argv[2],"MCW4") == 0)||(strcmp(argv[2],"mcw4") == 0)) ){
    MCW4=true;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"Setting the option: MC reweighting 4"<<std::endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    sprintf(RecoDir,"%s/%s",RecoDir,MCW4DirTXT);
    replaceChar(fitMassFileName,".root",Form("_%s.root",MCW4MassTXT));
   }
   if (argc>2 && ((strcmp(argv[2],"MCW5") == 0)||(strcmp(argv[2],"mcw5") == 0)) ){
    MCW5=true;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"Setting the option: MC reweighting 5"<<std::endl;
    std::cout<<"========================================================================="<<endl;
    std::cout<<"========================================================================="<<endl;
    sprintf(RecoDir,"%s/%s",RecoDir,MCW5DirTXT);
    replaceChar(fitMassFileName,".root",Form("_%s.root",MCW5MassTXT));
   }
//
//  Setting Input Files/Dir for RunEra..
//
replaceChar(RecoDir,"[RunEra]",Form("%d",RunEra));
replaceChar(InputFileNameRecoB0,"[RunEra]",Form("%d",RunEra));
replaceChar(fitMassFileName,"[RunEra]",Form("%d",RunEra));
//sprintf(fitMassFileName,"results_fits_%d_fixDataPdf.root",RunEra);
//======================================================================
//======================================================================
//===========		InputFileNameMCGene              ===============
//======================================================================
//======================================================================
//  sprintf(InputFileNameMCGene,"testGene-[RunEra]-Q2Bin-%d.root",Q2Bin);
//======================================================================
//======================================================================
//======================================================================
   std::cout<<"====================================="<<endl;

   if( Q2Bin<=8){
    if      (RunEra==2016){
     fM_sigmas=fM_sigmas_2016[Q2Bin];
     std::cout<<Form("=> Setting fM_sigmas_2016[%d] = %f",Q2Bin,fM_sigmas )<<std::endl;
    }else if(RunEra==2017){
     fM_sigmas=fM_sigmas_2017[Q2Bin];
     std::cout<<Form("=> Setting fM_sigmas_2017[%d] = %f",Q2Bin,fM_sigmas )<<std::endl;
    }else if(RunEra==2018){
     fM_sigmas=fM_sigmas_2018[Q2Bin];
     std::cout<<Form("=> Setting fM_sigmas_2018[%d] = %f",Q2Bin,fM_sigmas )<<std::endl;
    }else{
     std::cout<<"Q2Bin and fM_sigmas_2018not set correctly!!!"<<std::endl;
     exit(1);
    }
   }
   std::cout<<"====================================="<<endl;


//   if (argc>3 && (strcmp(argv[3],"i") == 0) ){
//     integral = true;
//     sprintf(SigmaMethodTXT,"-integraBin");
//     std::cout<<"===================================================================="<<endl;
//     std::cout<<"======== DEFAULT: INTEGRAL of the SB function  =============="<<std::endl;
//     std::cout<<"===================================================================="<<endl;
//   }else{
//     integral = false;
//     sprintf(SigmaMethodTXT,"-centerBin");
//     std::cout<<"========================================================================="<<endl;
//     std::cout<<"Setting the option: SB function evaluated in the CENTER of the bin"<<std::endl;
//     std::cout<<"========================================================================="<<endl;
//   }
  char NameList[300];;
  
  
  
  sprintf(NameList,"namelist-SB3DB0-%d-Q2Bin-%d.lis",RunEra, Q2Bin);
//  
//   if(Folded){
//     sprintf(FoldedTXT,"-PhiFolded");
//     XMinPhi = 0.;
//     ParMin = 0.;
//     RndMin = 0.;
//     std::cout<<"===================================================================="<<endl;
//     std::cout<<"======== SETTING: Phi Ang.Variable FOLDED             =============="<<std::endl;
//     std::cout<<"===================================================================="<<endl;
//   };
  
  
  char*argn[]={NameList};
  
  std::map<std::string, std::string> mappa = ReadNamelist(1,argn );
//
  maxDegree1	    =	 atoi (mappa["maxDegree1"].c_str() ) ;
  maxDegree2	    =	 atoi (mappa["maxDegree2"].c_str() ) ;
  maxDegree3	    =	 atoi (mappa["maxDegree3"].c_str() ) ;
  xCosLHBin	    =	 atof (mappa["xCosLHBin" ].c_str() ) ;
  xCosKHBin	    =	 atof (mappa["xCosKHBin" ].c_str() ) ;
  xPhiHBin	    =	 atof (mappa["xPhiHBin"  ].c_str() ) ;
  NSigma1L	    =	 atof (mappa["NSigma1L"  ].c_str() ) ;
  NSigma2L	    =	 atof (mappa["NSigma2L"  ].c_str() ) ;
  NSigma1R	    =	 atof (mappa["NSigma1R"  ].c_str() ) ;
  NSigma2R	    =	 atof (mappa["NSigma2R"  ].c_str() ) ;
  maxDegree	    =	 atoi (mappa["maxDegree" ].c_str() ) ;
  fixParam	    =	 atoi (mappa["fixParam"  ].c_str() ) ; 
  map<string,string>::iterator  it= mappa.find("MinContAdaptBin");
  if(it != mappa.end()) {
   MinContAdaptBin   =    atoi (mappa["MinContAdaptBin"  ].c_str() ) ;
  }
  map<string,string>::iterator  it2= mappa.find("AutoFixPar");
  if(it2 != mappa.end()) {
   AutoFixPar   =    atol (mappa["AutoFixPar"  ].c_str() ) ;
  }
  if(AutoFixPar){
   std::cout<<"Warning: setting search for Param to fix = "<<AutoFixPar<<std::endl;
  }
  map<string,string>::iterator  it3= mappa.find("NFactGen");
  if(it3 != mappa.end()) {
   NFactGen   =    atoi (mappa["NFactGen"  ].c_str() ) ;
   std::cout<<"Warning: setting  NFactGen from namelist= "<<NFactGen<<std::endl;
  }
  map<string,string>::iterator  it4= mappa.find("SigmaProbSign");
  if(it4 != mappa.end()) {
   SigmaProbSign   =    atoi (mappa["SigmaProbSign"  ].c_str() ) ;
   std::cout<<"Warning: setting  SigmaProbSign from namelist= "<<SigmaProbSign<<std::endl;
  }
  map<string,string>::iterator  it7= mappa.find("tagged_mass_rangeMin");
  if(it7 != mappa.end()) {
   tagged_mass_rangeMin   =    atof (mappa["tagged_mass_rangeMin"  ].c_str() ) ;
   std::cout<<"Warning: setting  tagged_mass_rangeMin from namelist= "<<tagged_mass_rangeMin<<std::endl;
   if(tagged_mass_rangeMin<XMinSign){
    std::cout<<Form("Error: setting  tagged_mass_rangeMin=%f < XMinSign=%f",tagged_mass_rangeMin,XMinSign)<<std::endl;
    exit(0);
   }
  }
  map<string,string>::iterator  it8= mappa.find("tagged_mass_rangeMax");
  if(it8 != mappa.end()) {
   tagged_mass_rangeMax   =    atof (mappa["tagged_mass_rangeMax"  ].c_str() ) ;
   std::cout<<"Warning: setting  tagged_mass_rangeMax from namelist= "<<tagged_mass_rangeMax<<std::endl;
   if(tagged_mass_rangeMax>XMaxSign){
    std::cout<<Form("Error: setting  tagged_mass_rangeMax=%f < XMaxSign=%f",tagged_mass_rangeMax,XMaxSign)<<std::endl;
    exit(0);
   }
  }

  std::cout<<" Num Param Bernstein polynomial CosL :  "<<maxDegree1<<std::endl;
  std::cout<<" Num Param Bernstein polynomial CosK :  "<<maxDegree2<<std::endl;
  std::cout<<" Num Param Bernstein polynomial Phi  :  "<<maxDegree3<<std::endl;
  std::cout<<" Binning choice for CosL		   :  "<<xCosLHBin<<std::endl;
  std::cout<<" Binning choice for CosK		   :  "<<xCosKHBin<<std::endl;
  std::cout<<" Binning choice for Phi		   :  "<<xPhiHBin <<std::endl;
//
  std::cout<<" Min CosL XMinCosThetaL		   :  "<<XMinCosThetaL<<std::endl;
  std::cout<<" Max CosL XMaxCosThetaL		   :  "<<XMaxCosThetaL<<std::endl;
  std::cout<<" Min CosK XMinCosThetaK		   :  "<<XMinCosThetaK<<std::endl;
  std::cout<<" Min CosK XMaxCosThetaK		   :  "<<XMaxCosThetaK<<std::endl;
  std::cout<<" Min Phi  XMinPhi 		   :  "<<XMinPhi<<std::endl;
  std::cout<<" Min Phi  XMaxPhi 		   :  "<<XMaxPhi<<std::endl;
//
  if(SigmaProbSign==0){
   sprintf(SigmaMethodTXT,"-SigmaGauss");
   std::cout<<" NSigma1L [sigma gauss model]:  "<<NSigma1L<<std::endl;
   std::cout<<" NSigma2L [sigma gauss model]:  "<<NSigma2L<<std::endl;
   std::cout<<" NSigma1R [sigma gauss model]:  "<<NSigma1R<<std::endl;
   std::cout<<" NSigma2R [sigma gauss model]:  "<<NSigma2R<<std::endl;
  }else if(SigmaProbSign==-1){
   sprintf(SigmaMethodTXT,"-SigmaBare");
   std::cout<<" NSigma1L [bare min limit left ]:"<<NSigma1L<<std::endl;
   std::cout<<" NSigma2L [bare max limit left ]:"<<NSigma2L<<std::endl;
   std::cout<<" NSigma1R [bare min limit right]:"<<NSigma1R<<std::endl;
   std::cout<<" NSigma2R [bare max limit right]:"<<NSigma2R<<std::endl;
  }else if (SigmaProbSign==1){
   sprintf(SigmaMethodTXT,"-SigmaProb");
   std::cout<<" NSigma1L [Limit in GeV Left ]:  "<<NSigma1L<<std::endl;
   std::cout<<" NSigma2L [n. gauss stand. dev. sign]:  "<<NSigma2L<<std::endl;
   std::cout<<" NSigma1R [n. gauss stand. dev. sign]:  "<<NSigma1R<<std::endl;
   std::cout<<" NSigma2R [Limit in GeV Right]:  "<<NSigma2R<<std::endl;
  }else{
   std::cout<<Form(" SigmaProbSign: 	INVALID OPTION: %f !!! Exit...",SigmaProbSign)<<std::endl;
   exit(1);
  }
//
  std::cout<<" Num Param Bernstein polynomial Mass :  "<<maxDegree<<std::endl;
  std::cout<<" Parameter to Fix for normalization  :  "<<fixParam<<std::endl;
  std::cout<<" Num of events inside adaptive bins  :  "<<MinContAdaptBin<<std::endl;
  
//
  if ((SigmaProbSign==-1)&&
       (NSigma1L==0.0 ||
        NSigma2L==0.0 ||
        NSigma1R==0.0 ||
        NSigma2R==0.0)){
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"Error reading sideband limits: at least one NSigma[]==0 found !!!!"<<std::endl;
     std::cout<<"====> EXIT from Main!!!"<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     exit(0);
  }    


//



  char TXTNSigma1L[10]="";
  char TXTNSigma2L[10]="";
  char TXTNSigma1R[10]="";
  char TXTNSigma2R[10]="";
  
  std::stringstream sss;
  std::string ss;
  std::string dot1=".";
  std::string dot2="dot";

  sss<<NSigma1L;
  ss=sss.str();
  cout<<ss<<endl;
  strcpy(FMTNSigma1L,ss.c_str());
  replaceAll( ss,  dot1, dot2);
  strcpy(TXTNSigma1L,ss.c_str());
  sss.str("");
  sss.clear();
  sss<<NSigma1R;
  ss=sss.str();
  cout<<ss<<endl;
  strcpy(FMTNSigma1R,ss.c_str());
  replaceAll( ss,  dot1, dot2);
  strcpy(TXTNSigma1R,ss.c_str());
  sss.str("");
  sss.clear();
  sss<<NSigma2L;
  ss=sss.str();
  cout<<ss<<endl;
  strcpy(FMTNSigma2L,ss.c_str());
  replaceAll( ss,  dot1, dot2);
  strcpy(TXTNSigma2L,ss.c_str());
  sss.str("");
  sss.clear();
  sss<<NSigma2R;
  ss=sss.str();
  cout<<ss<<endl;
  strcpy(FMTNSigma2R,ss.c_str());
  replaceAll( ss,  dot1, dot2);
  strcpy(TXTNSigma2R,ss.c_str());
   
//   cout<<TXTNSigma1L<<endl;
//   cout<<FMTNSigma1L<<endl;
//   cout<<TXTNSigma1R<<endl;
//   cout<<FMTNSigma1R<<endl;
//   cout<<TXTNSigma2L<<endl;
//   cout<<FMTNSigma2L<<endl;
//   cout<<TXTNSigma2R<<endl;
//   cout<<FMTNSigma2R<<endl;

  std::cout<<"--------------------------------------------\n"<<endl;
  std::cout<<" Setting selection for q^2 bin: "<<*argv[1]<<" ==> "<<Q2Min<<"<q^2<"<<Q2Max<<std::endl;
  std::cout<<"--------------------------------------------\n"<<endl;
  sprintf(OutFileName,"testGoofitSB3DB0-%d%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.root",RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(ListParName,"ListParValues-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.txt"   ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(ListPloName,"ListParValues-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.plo"   ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PDFNameFitSB3D,"B0-FitSB3D-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.pdf",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
  sprintf(PNGNameFitSB3D,"B0-FitSB3D-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.png",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
  sprintf(PNGNameMassQ2Hist,"B0-MassQ2-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png" ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PNGNameMassHist,"B0-MassTot-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png"  ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PNGNameMassCheck,"B0-MassCheck-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png"  ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PNGNameFitSB3DMass,"B0-FitSB3D-Mass-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.png",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
  sprintf(PNGNameFitSB3DProjX,"B0-FitSB3D-ProjX-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.png",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
  sprintf(PNGNameFitSB3DProjY,"B0-FitSB3D-ProjY-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.png",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
  sprintf(PNGNameFitSB3DProjZ,"B0-FitSB3D-ProjZ-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s-Adapt-%d.png",RunEra,ProjectTXT,  Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT,MinContAdaptBin); 
//
  sprintf(PNGNameProjXYHist,"B0-SB-ProjXY-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png"  ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PNGNameProjZYHist,"B0-SB-ProjZY-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png"  ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
  sprintf(PNGNameProjZXHist,"B0-SB-ProjZX-%d-%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.png"  ,RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 
//  sprintf(fitMassFileName,"save-w-%d-bin-%d.root",Q2Bin);
  sprintf(OutSaveFileName,"savesb-%d%s-Q2Bin-%d-Bins-%d-%d-%d-BernDeg-%d-%d-%d-WSBL-%s-%s-WSBR-%s-%s%s%s%s.root",RunEra,ProjectTXT, Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,maxDegree1,maxDegree2,maxDegree3,TXTNSigma1L,TXTNSigma2L,TXTNSigma1R,TXTNSigma2R,TaggedVarTXT,SigmaMethodTXT,FoldedTXT); 

  sprintf(OutFileNameInputHisto,"testGoofitSB3DB0-%d-InputHisto-Q2Bin-%d-Bins-%d-%d-%d-masspectrum%s.root",RunEra,Q2Bin,xCosLHBin,xCosKHBin,xPhiHBin,FoldedTXT); 



  
  BinWCosThetaL=(XMaxCosThetaL-XMinCosThetaL)/double(xCosLHBin);
  BinWCosThetaK=(XMaxCosThetaK-XMinCosThetaK)/double(xCosKHBin);
  BinWPhi      =(XMaxPhi-XMinPhi)/double(xPhiHBin);

//  TApplication tapp("TApp",&argc, argv);
  GooFit::Application app("testGoofit3DB0-[RunEra] fit example", argc, argv);
  app_ptr = &app;
//  app.require_subcommand();

//  app.add_flag("--minuit1", minuit1, "Use Minuit 1 instead of Minuit 2");
 
  TStopwatch TimeWatch;
  TimeWatch.Start();

  FitSBModel(); 
//  app.Run() ;
  cout<<"esco..." <<endl;
  TimeWatch.Stop();
  TimeWatch.Print();
  
//  GOOFIT_PARSE(app);
  return 0 ;
}


void FitSBModel(){


  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

   gROOT->Reset();
   gROOT->SetStyle("Plain");
   gROOT->ForceStyle();
   gStyle->SetOptStat(000000);
   gStyle->SetOptFit(000000);
   gStyle->SetPadBorderMode(0);

   TCanvas* c1 = new TCanvas("c1","Mass",200,10,900,780);
   TCanvas* c2 = new TCanvas("c2","Fit Mass Spectrum",200,10,900,780);
   TCanvas* cc = new TCanvas("cc","Fit Mass Spectrum Check",200,10,900,780);
//    TCanvas* c5 = new TCanvas("c5","                      ",200,10,900,780);
   TCanvas* c6 = new TCanvas("c6","Sideband",200,10,900,900);
   TCanvas* cmass = new TCanvas("cmas","Mass",200,10,750,800);
   TCanvas* cprojX = new TCanvas("cprojX","Angular Projections",200,10,900,780);
   TCanvas* cprojY = new TCanvas("cprojY","Angular Projections",200,10,900,780);
   TCanvas* cprojZ = new TCanvas("cprojZ","Angular Projections",200,10,900,780);
   TCanvas* cxy = new TCanvas("cxy","Sideband",200,10,750,750);
   TCanvas* cyz = new TCanvas("cyz","Sideband",200,10,750,780);
   TCanvas* cxz = new TCanvas("czy","Sideband",200,10,750,780);
   csignstudy = new TCanvas("csignstudy","Mass Signal study",200,10,900,450);
   c6->Divide(2,2);  
   csignstudy->Divide(2,1);
//    TCanvas* c7 = new TCanvas("c7","Efficiencies" ,200,10,900,780);
//    TCanvas* c8 = new TCanvas("c8","Effi-Reco Closure Test" ,200,10,900,780);
//   c2->Divide(2,2);  
//    c5->Divide(2,2);  
//    c7->Divide(2,2);  
//    c8->Divide(2,2);  
//    TPad* pad2 = (TPad*)c2->GetPad(0);
//    pad2->SetLeftMargin(0.15); 
//    pad2->SetRightMargin(0.15); 
//    TPad* pad6 = (TPad*)c6->GetPad(0);
//    pad6->SetLeftMargin(0.15); 
//    pad6->SetRightMargin(0.15); 

   //TPad* pad1 = (TPad*)c1->GetPad(0);
//   TPad* pad2 = (TPad*)c2->GetPad(0);
   //pad1->SetLeftMargin(0.15); 
//   pad2->SetLeftMargin(0.15); 



//  gSystem->Exec(Form("mv %s %s.tmp",OutFileName,OutFileName));
//  OutFileInputHisto = TFile::Open(OutFileNameInputHisto,"READ");
  if (!TFile::Open(OutFileNameInputHisto,"READ"))
  {
    cout<<"File:"<<OutFileNameInputHisto<<" not found!!! create..."<<endl;
    CreateInputHistoFile();
    OutFileInputHisto = TFile::Open(OutFileNameInputHisto,"READ");
  }else{
   OutFileInputHisto = TFile::Open(OutFileNameInputHisto,"READ");
   cout<<"File:"<<OutFileNameInputHisto<<" FOUND !!!"<<endl;
  } 
  gSystem->Exec(Form("mv %s %s.tmp",OutFileName,OutFileName));
  OutFile = TFile::Open(OutFileName,"RECREATE");
  TTree *RecoB0TreeOut     = (TTree*)OutFileInputHisto->Get(OutputRecoB0TreeName);
   if(!RecoB0TreeOut ){
     cout<<"TTree Reco Data: "<< OutputRecoB0TreeName <<" not found!!! Suggestion: remove this file e try again..."<<endl;
     exit(1);
   }else{
     cout<<"TTree Reco Data: "<< OutputRecoB0TreeName <<" OK FOUND!!!"<<endl;
   }  
  HxMass    = (TH1D*)OutFileInputHisto->Get("HxMass");
  if(!HxMass ){
    cout<<"HxMass Histo: not found!!! Exit..."<<endl;
    exit(1);
  }else{
    cout<<"HxMass Histo: OK FOUND!!! Entries: "<<HxMass->GetEntries()<<endl;
  } 
//    HxMassQ2    = (TH1D*)OutFileInputHisto->Get("HxMassQ2");
//   if(!HxMassQ2 ){
//     cout<<"HxMassQ2 Histo: not found!!! Exit..."<<endl;
//     exit(1);
//   }else{
//     cout<<"HxMassQ2 Histo: OK FOUND!!! Entries: "<<HxMassQ2->GetEntries()<<endl;
//   } 
 

  
//   HxReco    = (TH3D*)OutFileInputHisto->Get("HxReco");
//   if(!HxReco ){
//     cout<<"HxReco Histo: not found!!! Exit..."<<endl;
//     exit(1);
//   }else{
//     cout<<"HxReco Histo: OK FOUND!!! Entries: "<<HxReco->GetEntries()<<endl;
//     if(HxReco->GetNbinsX()!=xCosLHBin){cout<<"Error HxReco NBinsX = "<<HxReco->GetNbinsX()<<" != xCosLHBin = "<<xCosLHBin<<endl;exit(1);}
//     if(HxReco->GetNbinsY()!=xCosKHBin){cout<<"Error HxReco NBinsY = "<<HxReco->GetNbinsY()<<" != xCosKHBin = "<<xCosKHBin<<endl;exit(1);}
//     if(HxReco->GetNbinsZ()!=xPhiHBin ){cout<<"Error HxReco NBinsZ = "<<HxReco->GetNbinsZ()<<" != xPhiHBin  = "<<xPhiHBin <<endl;exit(1);}
//   }  
  cout<<"======================="<<endl;
  cout<<"xCosLHBin = "<<xCosLHBin<<endl;
  cout<<"xCosKHBin = "<<xCosKHBin<<endl;
  cout<<"xPhiHBin  = "<<xPhiHBin <<endl; 
  cout<<"======================="<<endl;
//
  TH3D* HxReco = new   TH3D( "HxReco"    , "B^{0} Reco correct tagged",  xCosLHBin, XMinCosThetaL, XMaxCosThetaL,
 									 xCosKHBin, XMinCosThetaK, XMaxCosThetaK,
									 xPhiHBin , XMinPhi, XMaxPhi );


  std::vector<GooFit::Observable> dataVec;
  GooFit::Observable xCosL_x("xCosL_x"  ,XMinCosThetaL, XMaxCosThetaL)  ;
  GooFit::Observable xCosK_y("xCosK_y"  ,XMinCosThetaK, XMaxCosThetaK)  ;
  GooFit::Observable xPhiK_z("xPhiK_z"  ,XMinPhi, XMaxPhi)  ;
//   GooFit::Observable xBinWidth1("xBinWidth1", 0.,100.);
//   GooFit::Observable yBinWidth1("yBinWidth1", 0.,100.);
//   GooFit::Observable zBinWidth1("zBinWidth1", 0.,100.);
  dataVec.push_back(xCosL_x);
  dataVec.push_back(xCosK_y);
  dataVec.push_back(xPhiK_z);
  UnbinnedDataSet* dataReco = new GooFit::UnbinnedDataSet(dataVec);
  std::vector<GooFit::Observable> plotVec;
  double cos_theta_l	;
  double cos_theta_k	;  
  double phi_kst_mumu	;
  double xtagged_mass	;
  double xmumuMass	;
  double xmumuMassE     ;
//  double mmk1	        ;
//  double mmk2           ;
  RecoB0TreeOut->SetBranchAddress("cos_theta_l"   ,&cos_theta_l );
  RecoB0TreeOut->SetBranchAddress("cos_theta_k"   ,&cos_theta_k );
  RecoB0TreeOut->SetBranchAddress("phi_kst_mumu"  ,&phi_kst_mumu);
  RecoB0TreeOut->SetBranchAddress("tagged_mass"   ,&xtagged_mass );
  RecoB0TreeOut->SetBranchAddress("mumuMass"      ,&xmumuMass );
  RecoB0TreeOut->SetBranchAddress("mumuMassE"     ,&xmumuMassE );
//  RecoB0TreeOut->SetBranchAddress("mmk1"          ,&mmk1 );
//  RecoB0TreeOut->SetBranchAddress("mmk2"          ,&mmk2 );
  int nentries = (int)RecoB0TreeOut->GetEntries();
//  int nentries = 0;
  cout<<"nentries: "<< nentries<<endl;
//  nentries = nentries/2.;
//  cout<<"PADUL!!!!! half nentries: "<< nentries<<endl;
  std::vector<GooFit::Observable> dataMassSBVec;
  dataMassSBVec.push_back(xMass);
  UnbinnedDataSet* dataMassSB = new GooFit::UnbinnedDataSet(dataMassSBVec);
  tagged_mass = new RooRealVar("tagged_mass" , "#mu^{+}#mu^{-}K#pi mass", XMinSign, XMaxSign, "GeV");
  mumuMass    = new RooRealVar("mumuMass"    , "mumuMass" , 0, 6);
  mumuMassE   = new RooRealVar("mumuMassE"   , "mumuMassE", 0, 10000);
  RooDataSet *fulldata   = new RooDataSet("fulldata", "fulldataset",  RooArgSet(*tagged_mass,*mumuMass,*mumuMassE));
  for (Int_t i=0;i<nentries;i++) {
         RecoB0TreeOut->GetEntry(i);
//  	 if(mmk2>3.6&&mmk2<4.2&&mmk1>4.7&&mmk1<4.9&&Q2Bin==4)continue;
         if( (xtagged_mass>XMinSign&&xtagged_mass<XMaxSign) ){
	   HxMassQ2->Fill(xtagged_mass);
 	   xMass.setValue(xtagged_mass);
 	   dataMassSB->addEvent();
	   tagged_mass->setVal(xtagged_mass);
	   mumuMass->setVal(xmumuMass);
	   mumuMassE->setVal(xmumuMassE);
	   fulldata->add(RooArgSet(*tagged_mass,*mumuMass,*mumuMassE));
	 } 
  }

  if(Q2Bin==4){  
    double B0Sigma_tmp = FitMassSpectrum(dataMassSB, cc, HxMassQ2,pdfHxMassQ2,sigHxMassQ2,bkgHxMassQ2, maxDegree);
    gSystem->Exec(Form("mv %s %s.tmp",PNGNameMassCheck,PNGNameMassCheck));
    cc->Print(PNGNameMassCheck);
  }  
//  std::cout<< "Setting B0Sigma = "<<B0Sigma<<" from the fit to the mass spectrum\n"<<std::endl; 
  B0Sigma = FitMassSpectrumRoofit(fulldata, c2, HxMassQ2,pdfHxMassQ2,sigHxMassQ2,bkgHxMassQ2, maxDegree);
//  exit(1);
  std::cout<< "Setting B0Sigma = "<<B0Sigma<<" from the fit to the mass spectrum\n"<<std::endl; 
//  c2->Write();
  gSystem->Exec(Form("mv %s %s.tmp",PNGNameMassQ2Hist,PNGNameMassQ2Hist));
  c2->Print(PNGNameMassQ2Hist);
//   XMinSBL = B0Mass - NSigma1L*B0Sigma;
//   XMaxSBL = B0Mass - NSigma2L*B0Sigma;
//   XMinSBR = B0Mass + NSigma1R*B0Sigma;
//   XMaxSBR = B0Mass + NSigma2R*B0Sigma;
  std::cout<<" XMinSBL  			 :     "<<XMinSBL<<std::endl;
  std::cout<<" XMaxSBL  			 :     "<<XMaxSBL<<std::endl;
  std::cout<<" XMinSBR  			 :     "<<XMinSBR<<std::endl;
  std::cout<<" XMaxSBR  			 :     "<<XMaxSBR<<std::endl;
  
  std::vector<double> CorreAdaptX;
  std::vector<double> CorreAdaptY;
  std::vector<double> CorreAdaptZ;
  for (Int_t i=0;i<nentries;i++) {
  	  RecoB0TreeOut->GetEntry(i);
	  if(cos_theta_l==-99) continue;
//    	  if(mmk2>3.6&&mmk2<4.2&&mmk1>4.7&&mmk1<4.9&&Q2Bin==4)continue;
// 	  if(cos_theta_l>XMaxCosThetaL) continue;
// 	  if(cos_theta_l<XMinCosThetaL) continue;
// 	  if(cos_theta_k>XMaxCosThetaK) continue;
// 	  if(cos_theta_k<XMinCosThetaK) continue;
// 	  if(phi_kst_mumu<XMinPhi     ) continue;
// 	  if(phi_kst_mumu>XMaxPhi     ) continue;
          if( (xtagged_mass>XMinSBL&&xtagged_mass<XMaxSBL)|| 
	      (xtagged_mass>XMinSBR&&xtagged_mass<XMaxSBR)){
	   HxMassQ2SB->Fill(xtagged_mass);
	   HxMassVsCosL->Fill(xtagged_mass,cos_theta_l);
	   HxMassVsCosK->Fill(xtagged_mass,cos_theta_k);
	   HxMassVsPhi ->Fill(xtagged_mass,phi_kst_mumu);
     	   xCosL_x.setValue(cos_theta_l);
     	   xCosK_y.setValue(cos_theta_k);
     	   xPhiK_z.setValue(phi_kst_mumu);
	   HxReco->Fill(cos_theta_l,cos_theta_k,phi_kst_mumu);
// 	   double xL = xCosL_x.getValue();
//            double yK = xCosK_y.getValue();
//            double zP = xPhiK_z.getValue();

           dataReco->addEvent();
//	   std::cout<<xL<<" "<<yK<<" "<<zP<<std::endl;
	   CorreAdaptX.push_back(cos_theta_l);
	   CorreAdaptY.push_back(cos_theta_k);
	   CorreAdaptZ.push_back(phi_kst_mumu);
         }
  }
  std::cout<<"Found SB entries = "<<HxReco->GetEntries()<<std::endl;
  if(HxReco->GetEntries()<10){
   std::cout<<"Error!! too few SB entries for a Fit: SB entries =  "<<HxReco->GetEntries()<<" EXIT!!!"<<std::endl;
   exit(0);
  }
  double xBinw =  HxReco->GetXaxis()->GetBinWidth(1) ;
  double yBinw =  HxReco->GetYaxis()->GetBinWidth(1) ;
  double zBinw =  HxReco->GetZaxis()->GetBinWidth(1) ;
  TH1D* HxRecoCosL = (TH1D*) HxReco->Project3D("x");
  TH1D* HxRecoCosK = (TH1D*) HxReco->Project3D("y");
  TH1D* HxRecoPhi  = (TH1D*) HxReco->Project3D("z");
// 
  TH2D* HxRecoCosLK =(TH2D*) HxReco->Project3D("xy");
  
//   plotVec.push_back(xCosL_x);
//   plotVec.push_back(xCosK_y);
//   plotVec.push_back(xPhiK_z);
//   UnbinnedDataSet* dataPlot = new GooFit::UnbinnedDataSet(plotVec);
//   
//   
  vector<GooFit::Observable> obsPoly;
  obsPoly.push_back(xCosL_x);
  obsPoly.push_back(xCosK_y);
  obsPoly.push_back(xPhiK_z);

//   vector<GooFit::Observable> obsPolyPlot;
//   obsPolyPlot.push_back(xCosL_x);
//   obsPolyPlot.push_back(xCosK_y);
//   obsPolyPlot.push_back(xPhiK_z);

  GooFit::Variable XMinCosL( "XMinCosL" , XMinCosThetaL);
  GooFit::Variable XMaxCosL( "XMaxCosL" , XMaxCosThetaL);
  GooFit::Variable XMinCosK( "XMinCosK" , XMinCosThetaK);
  GooFit::Variable XMaxCosK( "XMaxCosK" , XMaxCosThetaK);
  GooFit::Variable XMinPhiK( "XMinPhiK" , XMinPhi);
  GooFit::Variable XMaxPhiK( "XMaxPhiK" , XMaxPhi);
//  
  GooFit::Variable xBinWidth("xBinWidth", xBinw);
  GooFit::Variable yBinWidth("yBinWidth", yBinw);
  GooFit::Variable zBinWidth("zBinWidth", zBinw);
//  
  vector<GooFit::Variable> limits;
  limits.push_back(XMinCosL);
  limits.push_back(XMaxCosL);
  limits.push_back(XMinCosK);
  limits.push_back(XMaxCosK);
  limits.push_back(XMinPhiK);
  limits.push_back(XMaxPhiK);
  
  vector<GooFit::Variable> Binws;
  Binws.push_back(xBinWidth);
  Binws.push_back(yBinWidth);
  Binws.push_back(zBinWidth);



  int	numParameters = (maxDegree1+1)*(maxDegree2+1)*(maxDegree3+1);
  int icc=0;
  char ParCheck[numParameters][30];
  for(int i = 0; i <= maxDegree1 ; ++i) {
    for(int j = 0; j <= maxDegree2 ; ++j) {
     for(int k = 0; k <= maxDegree3 ; ++k) {
            
      	    sprintf(ParCheck[icc], "cosL=%d cosK=%d phi=%d", i,j,k); 
//	    cout<<Form("ParCheck(%d)= %s",icc,ParCheck[icc])<<endl;
	    icc++;
	    
     }
    }
  }
  vector<GooFit::Variable> coeffPoly;
//  int lim = limits.size();
  char varName[100];
  int NumCalls = 150000;
  
  int NumParamFree = 0;
  int fixParamTmp = -999;
  double parIni=0.;
  bool FoundParamNotZero = false;
  bool SearchFixParam = true;
  std::string line;
//  std::size_t sz;
//  double NParIni = 100.;
  std::cout<<"Try to open list of initial parameters :"<< ListParName <<std::endl;
  std::fstream *parListInput = new std::fstream(ListParName,std::ifstream::in);
  if(parListInput->is_open()){
     std::cout<<"List of initial parameters :"<< ListParName <<" FOUND!!!"<<std::endl;
     for (int i=0;i<numParameters;++i){
      	    sprintf(varName, "p%d", i);
            std::getline(*parListInput, line);
	    char* pEnd;
	    parIni =  strtod(line.c_str(), &pEnd);
//	    parIni =  stod(line, sz);
	    
// 	    *parListInput >> parIni;
//	    getline (parList,line);
     			if(fabs(parIni)>0.0009999) {
			 
			 if(parIni==1.00000) fixParamTmp = i;
//			 std::cout<<"parIni = "<<parIni<<std::endl;
//     			if(parIni!=0.0 && fabs(parIni)>0.00001) {
//     			 coeffPoly.emplace_back(varName, parIni,-100.0+parIni,100.0+parIni);
     			 coeffPoly.emplace_back(varName, parIni,0.0001,ParMin,ParMax);
			 NumParamFree++;
			 if(NumParamFree>1) FoundParamNotZero = true;
//     			 coeffPoly.emplace_back(varName, parIni,0.00001,0.,1000.);
//			  coeffPoly.emplace_back(varName, parIni,-1*NParIni*fabs(parIni),NParIni*fabs(parIni));
//			  coeffPoly.emplace_back(varName, parIni,fabs(parIni)/1000.,-1*NParIni*fabs(parIni),NParIni*fabs(parIni));
			 }else{
       			  coeffPoly.emplace_back(varName, 0.);
//       			  coeffPoly.emplace_back(varName, parIni,-1.0,1.0);
			 } 
			 if(i==fixParam && parIni!=1.00000){
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<Form("Error reading parameters file: fixparam=%d  %s=%f !=1.00 !!!",fixParam,varName,parIni)<<std::endl;
     std::cout<<"====> Try to search fixparam!!!"<<std::endl;
//     std::cout<<"====> EXIT from Fit!!!"<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     //exit(0);
     SearchFixParam = true;
			 }
			 if(SearchFixParam&&fixParamTmp>=0){
			  fixParam = fixParamTmp;
			  if(i==fixParam){
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<Form("Search FixParam Found: fixparam=%d  %s=%f =1.00 !!!",fixParam,varName,parIni)<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
			  }
			 }
     }
    if(!FoundParamNotZero){
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"Error reading parameters file: not found any free parameter different from 0.0 !!!!"<<std::endl;
     std::cout<<"====> EXIT from Fit!!!"<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     std::cout<<"====================================================================================="<<std::endl;
     exit(0);
    } 
    parListInput->close(); 
    sprintf(testo,"cp %s %s.tmp",ListParName,ListParName);
    gSystem->Exec(testo);
//     sprintf(testo,"cp %s %s.tmp",ListParNorm,ListParNorm);
//     gSystem->Exec(testo);
    parListInput->clear(); 
    sprintf(testo,"cp %s %s.tmp",ListPloName,ListPloName);
    gSystem->Exec(testo);
//     sprintf(testo,"cp %s %s.tmp",ListPloNorm,ListPloNorm);
//     gSystem->Exec(testo);
  }else{
     std::cout<<"First FIT: let's search the parameter to fix... "<<std::endl;
     FirstFit=true;
     TRandom3* trand_ini = new TRandom3(time(0));
     std::cout<<"List of initial parameters "<< ListParName <<" not found"<<std::endl;
     if(Q2Bin==4) {
      NumCalls = 1200000;
      boolHesse = false;
     }else{
      NumCalls = 200000;
      boolHesse = true;
     } 
     FitPrintLevel=0;
     for (int i=0;i<numParameters;++i){
            if(i==fixParam){
//            if(i==fixParam&&!AutoFixPar){
	     parIni=1.;
             std::cout<<"Warning !!!! Setting p"<<fixParam<<"=1 [because the normalization condition of PDF the (free num par)=(num par-1)"<<std::endl;
	    }else{ 
             parIni = trand_ini->Uniform(RndMin,RndMax);
	     NumParamFree++;
	    } 
     	    sprintf(varName, "p%d", i);
// 	    if(i>=24&&i<=47&&i!=44){
//      			 coeffPoly.emplace_back(varName, 0.0000,0.001,ParMin,ParMax);
// 			 
// 	    }else{
     			 coeffPoly.emplace_back(varName, parIni,0.001,ParMin,ParMax);
//	    }		 

     }
  }   
   
     int   initCoeffFit = limits.size();
//     GooFit::BernsteinPdf    *model=0;
//  if(integral){
         GooFit::FastBernsteinPdf    *model     =  new GooFit::FastBernsteinPdf("model",obsPoly,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3);
//  }else{
//        model     =  new GooFit::BernsteinPdf("model",obsPoly,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3);
//  } 

//================================================================================
//================================================================================
///FIT
//================================================================================
//================================================================================


  model->setData(dataReco);

//  GooFit::FitManager fitter(&model);//
//  int NumCalls = 1500000;

  if(SetMinuit2){
   GooFit::FitManagerMinuit2 fitter(model);
   fitter.setMaxCalls(NumCalls);
   fitter.setVerbosity(2);
   fitter.fit();
  }else{
      std::cout<<"Warning !!!! bSetting num call for MINUIT :"<<NumCalls  <<std::endl;
      GooFit::FitManagerMinuit1 fitter(model);
      fitter.setMaxCalls(NumCalls);
      fitter.useHesseBefore(false);
      fitter.useHesse(boolHesse);
      fitter.useMinos(false);
      cout<<"\n"<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"\n"<<endl;

      Minuit = fitter.getMinuitObject();
      Minuit->SetPrintLevel(FitPrintLevel);
//        Minuit->SetErrorDef(1.);
//     //  Minuit->SetErrorDef(0.5);
//      double arglist[2];
//      int err = 0;
//      arglist[0]= 120000; // maximum iterations
//      Minuit->Migrad();

// AutoFixPar
//
      if(AutoFixPar&&FirstFit){
      
        int SBEntriesDataReco=dataReco->getNumEvents();

//        CutSignificance=3.;  
        //CutSignificance=(sqrt(SBEntriesDataReco/50.));  
//        CutSignificance=round(sqrt(SBEntriesDataReco/50.));  
//        CutSignificance=round(sqrt(dataReco->getNumEvents()/50));  
        std::cout<<"----------------------------------"<<std::endl ;
        std::cout<<Form("--> Setting Significance CUT = %f SBEvents=%d [%f] <--",CutSignificance,SBEntriesDataReco,sqrt(dataReco->getNumEvents()/50.))<<std::endl ;
        std::cout<<"----------------------------------"<<std::endl ;
//
        std::cout<<"=================================="<<std::endl ;
        std::cout<<"=================================="<<std::endl ;
        std::cout<<"==> Begin First FIT   <=="<<std::endl ;
        std::cout<<"=================================="<<std::endl ;
        std::cout<<"=================================="<<std::endl ;
//        double CutParValue0 = 0.000000001;
//        double CutParValue = 0.01;
	double CutParValue     = 0.01;
//	double CutParValue     = 0.000000001;
//	double CutParValueNorm = 0.01;
//	double CutParValueLast = 0.12;
        //double CutParValue = 0.01;
        //double CutParValueLast = 0.20;
//        double CutParValue = 0.09999;
//	double SigmaCut1 =1.00;
//	double SigmaCut2 =2;
	
//	if(NFactGen==1) CutSignificance=round(sqrt(dataReco->getNumEvents()/200)*CutSignificance);  
//	double SigmaCut2 =2.5*round(sqrt(NFactGen));
	double SigmaCut2 =CutSignificance*round(sqrt(NFactGen));
//	double SigmaCut2 =CutSignificance;
	if (SigmaCut2>5) SigmaCut2=5;
	
	
//	double SigmaCut25 =CutSignificance;
	double SigmaCut25 =CutSignificance*round(sqrt(NFactGen));
	if (SigmaCut25>5) SigmaCut25=5;
	
//	double SigmaCut2 =2.00*(sqrt(HxReco->GetEntries()/257));
	int err=1;
        double XStep=0.0001;
	bool IlFix= false;
	int IlFirstFix = initCoeffFit+fixParam;
	if (fixParam<=numParameters){
	 IlFix= true;
 	 std::cout<<"===================================================================="<<std::endl ;
	 std::cout<<Form("Parameter for NORMALIZATION IS SET fix to  ==> p(%d)=1!!!!",fixParam)<<std::endl;
 	 std::cout<<"===================================================================="<<std::endl ;
	}  
	std::cout<<Form("=================== [Zero FIT Loop Start] ===================\n")<<std::endl; 
//=====================================================================
//=====================================================================
//  	Minuit->mnrset(1);
//   	 for (int i=0;i<numParameters;++i){
// 	       if (IlFix&&i==fixParam) continue;
// 	        Minuit->Release(i+initCoeffFit);
//  	        Minuit->mnparm(i+initCoeffFit, varName,coeffPoly[i].getValue() ,XStep,ParMin,ParMax,err );
// 	 }      
  	if(IlFix) Minuit->FixParameter(IlFirstFix);
//  	for (int i=0;i<numParameters;++i){
// 	   if(coeffPoly[i].getValue()==0.){
//      	    sprintf(varName, "p%d", i);
//  	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep, -1.,1.,err);
//  	    Minuit->FixParameter(i+initCoeffFit);
// 	   } 
//         }
   	fitter.fit();
//First loop 
//        NumCalls=150000;	
//        fitter.setMaxCalls(NumCalls);
	bool FitAgain=true;
	std::cout<<Form("=================== [First FIT Loop Start] ===================\n")<<std::endl; 
	for (int iLoop=0;iLoop<numParameters;++iLoop){
//        double min_val =999;
//	int il_val =-999;
	 FitAgain=false;
 	 for (int i=0;i<numParameters;++i){
	   if(coeffPoly[i].getError()==0.) continue;
 	   sprintf(varName, "p%d", i);
	   double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
//           double coeff_val = coeffPoly[i].getValue();
// 	   if(fabs(coeffPoly[i].getError())!=0.&& (coeff_sign<1.&&coeffPoly[i].getValue()>1. || fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getValue()<1.) ){
// 	   if(coeff_val<min_val){
// 	    min_val=coeff_val;
// 	    il_val=i;
// 	    FitAgain=true;
// 	    std::cout<<Form("VALMINIMUM = %f IPar=%d",min_val,il_val)<<std::endl;
// 	   } 
	   if(fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getError()!=0.) {
	    std::cout<<Form("==>[First FIT Loop] Force setting p(%d)=0  [value=%3.10f<%f] [significance=%3.10f]",i,coeffPoly[i].getValue(),CutParValue,coeff_sign)<<std::endl;
	    FitAgain=true;
	    coeffPoly[i] =0.00;
 	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep, -1.,1.,err);
 	    Minuit->FixParameter(i+initCoeffFit);
// 	   }else if(fabs(coeffPoly[i].getValue())<0.0001&&coeffPoly[i].getError()!=0.&&iLoop==0) {
// 	    std::cout<<Form("==>[First FIT Loop] Force setting p(%d)=0  [value=%f<%f] [significance=%f]",i,coeffPoly[i].getValue(),CutParValue,coeff_sign)<<std::endl;
// 	    FitAgain=true;
// 	    coeffPoly[i] =0.00;
//  	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep, -1.,1.,err);
//  	    Minuit->FixParameter(i+initCoeffFit);
 	   }else{
  	    Minuit->mnparm(i+initCoeffFit, varName, coeffPoly[i].getValue(),XStep,ParMin,ParMax,err);
	   }
 	 } 
// 	 if (il_val>=0&&min_val<CutParValue&&coeffPoly[il_val].getError()!=0.000){
// 	  std::cout<<Form("==>[First Loop=%d] Force setting p(%d)=0  [value=%3.12f<%f] \
// 	  [minimum value=%3.12f]",iLoop,il_val,coeffPoly[il_val].getValue(),CutParValue,min_val)<<std::endl;
// 	  coeffPoly[il_val] =0.00;
//  	  sprintf(varName, "p%d", il_val);
//  	  Minuit->mnparm(il_val+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
//  	  Minuit->FixParameter(il_val+initCoeffFit);
// 	 }else{
// 	  FitAgain=false;
// 	 }
	 Minuit->mnrset(1);
 	 fitter.fit();
  	 if(!FitAgain) break;
 	 std::cout<<"=================================="<<std::endl ;
 	 std::cout<<Form("==> Begin FIT %d",iLoop)<<std::endl ;
 	 std::cout<<"=================================="<<std::endl ;
//	 Minuit->mnrset(1);
	}
// Exit from loop, search the param with greater significance
//        ParMax=1.;
        if(Q2Bin==4){
          fitter.setMaxCalls(NumCalls);
          fitter.useHesse(true);
	}  
	if(!IlFix){
 	 std::cout<<"=================================="<<std::endl ;
 	 std::cout<<Form("==> EXIT FROM LOOP, search the param")<<std::endl ;
	 std::cout<<Form("with greater significance")<<std::endl ;
 	 std::cout<<"=================================="<<std::endl ;
         Minuit->Release(IlFirstFix);
 	 for(int i=0;i<numParameters;++i) {
	  if(coeffPoly[i].getError()==0.) continue;
	  double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
 	   std::cout<<"=================================="<<std::endl ;
 	   std::cout<<Form("Par Value = %f Err = %f Signif = %f",coeffPoly[i].getValue(),coeffPoly[i].getError(),coeff_sign)<<std::endl ;
 	   std::cout<<"=================================="<<std::endl ;
//  	  if(fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getError()!=0.){
// 	   coeffPoly[i] =0.00;
// 	   std::cout<<Form("==>[search max significance] Force setting p(%d)=0 [value=%f<%f] significance=%f",i,coeffPoly[i].getValue(),CutParValue,coeff_sign)<<std::endl;
// 	  } 
//	  if(fabs(coeffPoly[i].getValue())<coeffPoly[i].getError()) coeffPoly[i] =0.00;
//ok 	  if(fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getError()!=0.) coeffPoly[i] =0.00;
// 	  if(fabs(coeffPoly[i].getError())!=0.&& (coeff_sign<1.&&coeffPoly[i].getValue()>1. || fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getValue()<1.) ) coeffPoly[i] =0.00;
  	  if(fabs(coeffPoly[i].getValue())>1.&&coeffPoly[i].getError()>0. ){
 	   double xCoeffNormTmp = fabs(coeffPoly[i].getValue()/coeffPoly[i].getError());
 	   if (xCoeffNormTmp>xCoeffNorm) {
 	    xCoeffNorm=xCoeffNormTmp;
 	    xCoeffIndex=i;
 	   }
 	  }
	 }
	  if(xCoeffIndex<0&&fixParam>=numParameters){
 	   std::cout<<"=================================="<<std::endl ;
 	   std::cout<<"Parameter for NORMALIZATION not found!!! Exit..."<<std::endl ;
 	   std::cout<<"=================================="<<std::endl ;
	   exit(1);
	  }else if(xCoeffIndex<0&&fixParam<numParameters){
	   xCoeffIndex=fixParam;
 	   std::cout<<"=================================="<<std::endl ;
	   std::cout<<Form("Parameter for NORMALIZATION remain fix to  ==> p(%d)=1!!!!",xCoeffIndex)<<std::endl;
 	   std::cout<<"=================================="<<std::endl ;
	  }else{
 	   std::cout<<"=================================="<<std::endl ;
	   std::cout<<Form("Parameter for NORMALIZATION is set by loop to ==> p(%d)=1!!!!",xCoeffIndex)<<std::endl;
 	   std::cout<<"=================================="<<std::endl ;
	  }
//       TRandom3* trand_ini2 = new TRandom3();
 	 for(int i=0;i<numParameters;++i) {
	  if(coeffPoly[i].getError()==0.) continue;
 	  sprintf(varName, "p%d", i);
	  if(coeffPoly[i].getValue()<=0.0){
 	     Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep, -1.,1.,err);
 	     Minuit->FixParameter(i+initCoeffFit);
	  }
	  if(coeffPoly[i].getValue()>0.0&&i!=xCoeffIndex){
	    double parRenorm = coeffPoly[i].getValue()/coeffPoly[xCoeffIndex].getValue();
 	    Minuit->mnparm(i+initCoeffFit, varName,parRenorm,XStep, ParMin,ParMax,err);
	  }
//	    double parIni2 = trand_ini2->Uniform(RndMin,RndMax);
//	 Minuit->mnparm(i+initCoeffFit, varName,parIni2,0.001, ParMin,ParMax,err);
	 }
  	 sprintf(varName, "p%d",xCoeffIndex );
 	 Minuit->mnparm(xCoeffIndex+initCoeffFit, varName,1.,XStep, -1.,1.,err);
 	 Minuit->FixParameter(xCoeffIndex+initCoeffFit);
	 Minuit->mnrset(1);
 	 fitter.fit();
	}else{
	   xCoeffIndex=fixParam;
 	   std::cout<<"=================================="<<std::endl ;
	   std::cout<<Form("Parameter for NORMALIZATION REMAIN SET to ==> p(%d)=1!!!!",xCoeffIndex)<<std::endl;
 	   std::cout<<"=================================="<<std::endl ;
	}
 	   std::cout<<"=================================="<<std::endl ;
	   std::cout<<"         DUMP SIGNIFICANCES"<<std::endl;
 	   std::cout<<"=================================="<<std::endl ;
 	for (int i=0;i<numParameters;++i){
 	  if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==0.000) continue;
 	  double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
	  if(coeff_sign>1){
	  std::cout<<Form("==>CHECKSIGN Ok significance p(%d)=0 [value=%f] [cut significance %f>1 (%f)] \
	  ",i,coeffPoly[i].getValue(),coeff_sign,SigmaCut2)<<std::endl;
	  }else{
	  std::cout<<Form("==>CHECKSIGN CUT significance p(%d)=0 [value=%f] [cut significance %f<1 (%f)] \
	  ",i,coeffPoly[i].getValue(),coeff_sign,SigmaCut2)<<std::endl;
	  }
	} 
// start a loop to fix the model...	
//        ParMax=1;

        if (Q2Bin==4) {
	  fitter.useHesse(true);
	  std::cout<<Form("Q2Bin=4 => Set fitter.useHesse(true)")<<std::endl;
	 } 
        double min_sign =999;
	int il_sign =-999;
	for (int iLoop=0;iLoop<numParameters;++iLoop){
//	 cout<<"=========================="<<endl;
//	 cout<<Form("==> FIX Loop N.%d",iLoop)<<endl;
	 NumParamFree=1.;
         min_sign =999;
	 il_sign =-999;
	 FitAgain=false;
 	 for (int i=0;i<numParameters;++i){
 	   if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==0.000) continue;
 	   if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==1.000) continue;
// 	   if(fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getError()!=0.) {
 	   double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
	   if(coeff_sign<min_sign){
	    min_sign=coeff_sign;
	    il_sign=i;
	    FitAgain=true;
	    std::cout<<Form("MINIMUM = %f IPar=%d loop=%d",min_sign,il_sign,iLoop)<<std::endl;
	   } 
// 	 if(fabs(coeffPoly[i].getError())!=0.&&  fabs(coeffPoly[i].getValue())<CutParValue&&coeff_sign<1 ) {
//ok 	 if(fabs(coeffPoly[i].getError())!=0.&& (coeff_sign<1.&&coeffPoly[i].getValue()>1. || fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getValue()<1.) ) {
// 	 if(fabs(coeffPoly[i].getError())!=0.&& (coeff_sign<1.&&coeffPoly[i].getValue()>1. || fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getValue()<1.) ) {
//  	 if(fabs(coeffPoly[i].getError())!=0.&&  fabs(coeffPoly[i].getValue())<CutParValue ) {
// 	  std::cout<<Form("==>[Loop=%d to fix model]  Force setting p(%d)=0 [value=%f<%f] significance=%f",iLoop,i,coeffPoly[i].getValue(),CutParValue,coeff_sign)<<std::endl;
// 	    FitAgain=true;
// 	    coeffPoly[i] =0.00;
//  	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
//  	    Minuit->FixParameter(i+initCoeffFit);
// 	 }
// 	 if((fabs(coeffPoly[i].getValue())<fabs(coeffPoly[i].getError())||fabs(coeffPoly[i].getValue())<CutParValue)&&iLoop>1) {
//ref 	 if(fabs(coeffPoly[i].getValue())<fabs(coeffPoly[i].getError())&&iLoop>1) {
//04062020
//  	   if(coeff_sign<SigmaCut1&&iLoop==iFitLoop) {
// 	    std::cout<<Form("==>[Loop=%d to fix model] Force setting p(%d)=0  [value=%f] \
// 	    [significance=%f<%f]",iLoop,i,coeffPoly[i].getValue(),coeff_sign,SigmaCut1)<<std::endl;
// 	    FitAgain=true;
// 	    coeffPoly[i] =0.00;
//  	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
//  	    Minuit->FixParameter(i+initCoeffFit);
// //	   }else{
// //	    Minuit->mnparm(i+initCoeffFit, varName, coeffPoly[i].getValue(),0.001,ParMin,ParMax,err);
// 	   }
	   if(coeffPoly[i].getValue()>0.){
	    NumParamFree++;
	   } ;
	 }// end loop params 
 	 std::cout<<"=================================="<<std::endl ;
 	 std::cout<<Form("==> Begin FIT Step %d Fixing par%d=1",iLoop,xCoeffIndex)<<std::endl ;
 	 std::cout<<"=================================="<<std::endl ;
//	 if (il_sign>=0&&min_sign<SigmaCut2){
	 if (il_sign>=0&&min_sign<SigmaCut2&&coeffPoly[il_sign].getError()!=0.000){
	  std::cout<<Form("==>[Loop=%d to fix model] Force setting p(%d)=0 [value=%f] [minimum significance=%f<%f] \
	  ",iLoop,il_sign,coeffPoly[il_sign].getValue(),min_sign,SigmaCut2)<<std::endl;
	  coeffPoly[il_sign] =0.00;
 	  sprintf(varName, "p%d", il_sign);
 	  Minuit->mnparm(il_sign+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
 	  Minuit->FixParameter(il_sign+initCoeffFit);
	 }else{
	  FitAgain=false;
          Minuit->SetPrintLevel(2);
	 }
//  	 for (int i=0;i<numParameters;++i){
// 	  if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==0.000) continue;
// 	  if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==1.000) continue;
// 	  if(i==il_sign) continue;
// 	  if(i==xCoeffIndex) continue;
//  	  sprintf(varName, "p%d", i);
//   	  Minuit->mnparm(i+initCoeffFit, varName, 0.1,XStep,ParMin,ParMax,err);
// 	 } 
	 Minuit->mnrset(1);
 	 fitter.fit();
  	 if(!FitAgain){
//  	  NumParamFree=1.;
//    	  for (int i=0;i<numParameters;++i){
//  	   if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==0.000) continue;
//  	   if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==1.000) continue;
// 	 
//  	   double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
// 	   if(coeffPoly[i].getValue()<CutParValueLast&&coeff_sign<SigmaCut25) {
// 	    std::cout<<Form("==>[last fit] Force setting p(%d)=0 [value=%f<%f] \
// 	    [significance=%f<%f]",i,coeffPoly[i].getValue(),CutParValueLast,coeff_sign,SigmaCut25)<<std::endl;
// 	    coeffPoly[i] =0.00;
// 	    Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
// 	    Minuit->FixParameter(i+initCoeffFit);
// 	   } 
// 	   if(coeffPoly[i].getValue()>0.){
// 	    NumParamFree++;
//  	   } ;
// 	  } 
//  	  Minuit->mnrset(1);
//  	  fitter.fit();
	  if(!IlFix){
 	   xCoeffNorm =0.0;
 	   int xCoeffIndexTmp=-1;
 	   for(int i=0;i<numParameters;++i) {
 	    if((coeffPoly[i].getValue())>1. && fabs(coeffPoly[i].getError()>0.) ){
// 	    if((coeffPoly[i].getValue()+coeffPoly[i].getError())>1. && fabs(coeffPoly[i].getError()>0. && i<xCoeffIndex) ){
 	     double xCoeffNormTmp = coeffPoly[i].getValue();
//	     double xCoeffNormTmp = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
 	     if (xCoeffNormTmp>xCoeffNorm) {
 	      xCoeffNorm=xCoeffNormTmp;
 	      xCoeffIndexTmp=i;
 	     }
 	    }
 	   }
	   if( xCoeffIndexTmp>0 ){
	    xCoeffIndex=xCoeffIndexTmp;
 	    std::cout<<"=============================================================="<<std::endl;
 	    std::cout<<"=============================================================="<<std::endl;
	    std::cout<<Form("WARNING: Normalization could be fixed better if p(%d)=1",xCoeffIndex)<<std::endl ;
	    std::cout<<Form("WARNING: Try to fit fixing p(%d)=1",xCoeffIndex)<<std::endl ;
 	    std::cout<<"=============================================================="<<std::endl;
 	    std::cout<<"=============================================================="<<std::endl;
 	    for(int i=0;i<numParameters;++i) {
	     if(coeffPoly[i].getValue()>0.000) {
 	      Minuit->Release(i+initCoeffFit);
 	      double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
	      double parRenorm = coeffPoly[i].getValue()/xCoeffNorm;
 	      sprintf(varName, "p%d", i);
 	       Minuit->mnparm(i+initCoeffFit, varName,parRenorm,XStep, ParMin,ParMax,err);
 	     }
	    }
	    Minuit->FixParameter(xCoeffIndex+initCoeffFit);
	    Minuit->mnrset(1);
 	    fitter.fit();
	    FitAgain=true;
	   }
 	  }
	  if(!FitAgain){
 	   std::cout<<Form("==> BREAK: EXIT FIT Step %d",iLoop)<<std::endl ;
	   break;
	  } 
	 } 
	}
        std::cout<<"Count NumParamFree "<<NumParamFree<<std::endl;
	if(NumParamFree<=1){
         std::cout<<"====> Error!!! Exit... "<<std::endl;
	 exit(0);
	}
/* 	if(!IlFix){
 	 xCoeffNorm =0.0;
 	 int xCoeffIndexTmp=-1;
 	 for(int i=0;i<numParameters;++i) {
 	  if(fabs(coeffPoly[i].getValue())>1. && fabs(coeffPoly[i].getError()>0.) ){
 	   double xCoeffNormTmp = coeffPoly[i].getValue();
//	   double xCoeffNormTmp = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
 	   if (xCoeffNormTmp>xCoeffNorm) {
 	    xCoeffNorm=xCoeffNormTmp;
 	    xCoeffIndexTmp=i;
 	   }
 	  }
 	 }
	 if( xCoeffIndexTmp>0 ){
	  xCoeffIndex=xCoeffIndexTmp;
 	  std::cout<<"=============================================================="<<std::endl;
 	  std::cout<<"=============================================================="<<std::endl;
	  std::cout<<Form("WARNING: Normalization could be fixed better if p(%d)=1",xCoeffIndex)<<std::endl ;
	  std::cout<<Form("WARNING: Try to fit fixing p(%d)=1",xCoeffIndex)<<std::endl ;
 	  std::cout<<"=============================================================="<<std::endl;
 	  std::cout<<"=============================================================="<<std::endl;
 	  for(int i=0;i<numParameters;++i) {
	   if(coeffPoly[i].getValue()>0.000) {
            Minuit->Release(i+initCoeffFit);
  	    double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
	    double parRenorm = coeffPoly[i].getValue()/xCoeffNorm;
            sprintf(varName, "p%d", i);
// 	    if (parRenorm>CutParValueNorm){
   	     Minuit->mnparm(i+initCoeffFit, varName,parRenorm,XStep, ParMin,ParMax,err);
// 	    }else{
// 	     std::cout<<Form("==>[try last renormalisation] Force setting p(%d)=0 [significance=%f]",i,coeff_sign)<<std::endl;
// 	     Minuit->mnparm(i+initCoeffFit, varName,0.000,0.001, ParMin,ParMax,err);
//             Minuit->FixParameter(i+initCoeffFit);
// 	    }
   	   }
	  } 
	  Minuit->FixParameter(xCoeffIndex+initCoeffFit);
	  Minuit->mnrset(1);
 	  fitter.fit();
	 } 
        }
 *///        Minuit->mnrset(1);
/*        std::cout<<"=========================="<<std::endl;
       std::cout<<"=       LAST FIT!!!      ="<<std::endl;
       std::cout<<"=========================="<<std::endl;
       for (int iLoop=0;iLoop<numParameters;++iLoop){
 	NumParamFree=1.;
	FitAgain=false;
   	for (int i=0;i<numParameters;++i){
 	 if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==0.000) continue;
 	 if(coeffPoly[i].getError()==0.000&&coeffPoly[i].getValue()==1.000) continue;
	
 	 double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
 	 double valmenosigma  = coeffPoly[i].getValue()-1.*fabs(coeffPoly[i].getError());
	 std::cout<<Form("==> coeffPoly[%d] valmenosigma=%f coeff_sign=%f\n",i,valmenosigma,coeff_sign);
//	 if((coeffPoly[i].getValue()-3.*coeffPoly[i].getError())<CutParValueLast&&coeff_sign<SigmaCut25) {
// 	 if(valmenosigma<CutParValueLast&&coeff_sign<SigmaCut25) {
//           sprintf(varName, "p%d", i);
// 	  std::cout<<Form("==>[LAST FIT Loop=%d] Force setting p(%d)=0 [value=%f<%f] \
// 	  [significance=%f<%f]",iLoop,i,coeffPoly[i].getValue(),CutParValueLast,coeff_sign,SigmaCut25)<<std::endl;
// 	  coeffPoly[i] =0.00;
// 	  Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
// 	  Minuit->FixParameter(i+initCoeffFit);
// 	  FitAgain=true;
// 	 } 
	 if (coeff_sign<SigmaCut25&&coeffPoly[i].getError()!=0.000){
          sprintf(varName, "p%d", i);
	  std::cout<<Form("==>[LAST FIT Loop=%d] Force setting p(%d)=0 [value=%f] [minimum significance=%f<%f] \
	  ",iLoop,i,coeffPoly[i].getValue(),coeff_sign,SigmaCut2)<<std::endl;
	  coeffPoly[i] =0.00;
	  Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
	  Minuit->FixParameter(i+initCoeffFit);
	  FitAgain=true;
         }
	 if(coeffPoly[i].getValue()>0.){
	  NumParamFree++;
 	 } ;
	} 
 	Minuit->mnrset(1);
 	fitter.fit();
        if(!FitAgain){
 	  std::cout<<Form("==> BREAK: EXIT LAST FIT Step %d",iLoop)<<std::endl ;
	  break;
	}  
       }	
 */
// 	for (int iLoop=0;iLoop<=iFitLoop+1;++iLoop){
// 	 cout<<"=========================="<<endl;
// 	 cout<<Form("==> LAST Loop N.%d",iLoop)<<endl;
//  	 for (int i=0;i<numParameters;++i){
//  	  if(coeffPoly[i].getValue()==0.&&coeffPoly[i].getError()==0.) continue;
//  	  sprintf(varName, "p%d", i);
// //	  if(fabs(coeffPoly[i].getValue())<CutParValue&&coeffPoly[i].getError()!=0.) {
//  	  double coeff_sign = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
// 	   
// 	  Minuit->mnparm(i+initCoeffFit, varName, coeffPoly[i].getValue(),0.001,ParMin,ParMax,err);
// 	  
// //  	  if(coeff_sign<SigmaCut2&&iLoop>iFitLoop ) {
// // //	  if(coeff_sign<round(sqrt(NFactGen))&&iLoop==1) {
// // 	   std::cout<<Form("==>[last fit] Force setting p(%d)=0 [value=%f]\
// // 	   [significance=%f<%f]",i,coeffPoly[i].getValue(),coeff_sign,SigmaCut2)<<std::endl;
// // 	   coeffPoly[i] =0.00;
// //  	   Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
// //  	   Minuit->FixParameter(i+initCoeffFit);
// /* // 	  }
// */ 	  if(coeff_sign<SigmaCut2&&iLoop>=iFitLoop ) {
// 	   std::cout<<Form("==>[last fit] Force setting p(%d)=0 [value=%f]\
// 	   [significance=%f<%f]",i,coeffPoly[i].getValue(),coeff_sign,SigmaCut2)<<std::endl;
// 	   coeffPoly[i] =0.00;
//  	   Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
//  	   Minuit->FixParameter(i+initCoeffFit);
// 	  }
// //	  if(coeffPoly[i].getValue()+SigmaCut1*coeffPoly[i].getError()<CutParValueLast) {
// 	  if(coeffPoly[i].getValue()<CutParValueLast) {
// 	   std::cout<<Form("==>[last fit] Force setting p(%d)=0 [value=%f<%f] significance=%f",i,coeffPoly[i].getValue(),CutParValueLast,coeff_sign)<<std::endl;
// 	   coeffPoly[i] =0.00;
// 	   Minuit->mnparm(i+initCoeffFit, varName, 0.,XStep,-1.,1.,err);
// 	   Minuit->FixParameter(i+initCoeffFit);
// 	  } 
//  	 }// end loop on params 
// 	 
// 	 Minuit->FixParameter(xCoeffIndex+initCoeffFit);
//  	 fitter.fit();
// 	}// end loop on fits
// 	 
	NumParamFree=0; 
 	for (int i=0;i<numParameters;++i){
	 if(coeffPoly[i].getValue()>0.){
	  NumParamFree++;
	 } ;
        } ;
        std::cout<<"After fit - Count NumParamFree "<<NumParamFree<<std::endl;
	if(NumParamFree<=1){
         std::cout<<"====> Error!!! Exit... "<<std::endl;
	 exit(0);
	} 
  	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<Form("REMIND: In This Fit Normalization parameter is ==> p(%d)=1!!!!",xCoeffIndex)<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
      }else{
//
//  Manual Fit
//
  	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<Form("                      START MANUAL FIT")<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
 	std::cout<<"=============================================================="<<std::endl;
        Minuit->FixParameter(initCoeffFit+fixParam);
        std::cout<<"Warning !!!! Fixing p"<<fixParam<<"=1"<<std::endl;
//          Minuit->FixParameter(19);
        for (int i=0;i<numParameters;++i){
          if (coeffPoly[i].getValue()==0.0){
            Minuit->FixParameter(i+initCoeffFit);
          }
         }
         std::fstream *parListFix = new std::fstream(FitStraName,std::ifstream::in);
         if(parListFix->is_open()){
          parListFix->close();
          std::cout<<"==========================================================================="<<std::endl ;
          std::cout<<"Namelist: "<<FitStraName<<" to fix parameters exist! Opening..."<<std::endl ;
          std::cout<<"==========================================================================="<<std::endl ;
          char*argm[]={FitStraName};
          std::map<std::string, std::string> mapfix = ReadNamelist(1,argm );
          for (int i=0;i<numParameters;++i){
            if( strcmp(mapfix[coeffPoly[i].getName()].c_str(),"fix")>=0){
             Minuit->FixParameter(i+initCoeffFit);
             std::cout<<"==========================================================================="<<std::endl ;
             std::cout<<"Fixing Parameter -> "<<coeffPoly[i].getName()<<std::endl;
             std::cout<<"==========================================================================="<<std::endl ;
            }
          }
         }else{
           std::cout<<"==========================================================================="<<std::endl ;
           std::cout<<"Warning ! Namelist: "<<FitStraName<<" to fix parameters in the fit doesn't exist. Proceed ahead..."<<std::endl ;
           std::cout<<"==========================================================================="<<std::endl ;
         }
//      Minuit->FixParameter(10);
        fitter.fit();
     }
//
// save covariance Matrix     	
// 
     Double_t matrix[NumParamFree-1][NumParamFree-1];
     Minuit->mnemat(&matrix[0][0],NumParamFree-1);
     covMatrix = new TMatrixD(NumParamFree-1,NumParamFree-1,&matrix[0][0]);
  }
//  covMatrix->Print("f=  %10.3e  ");
  std::vector<RooRealVar> parLis;
  RooArgList *coefLis = new RooArgList();
  for (int i=0;i<numParameters;++i){
   sprintf(varName, "p%03d_%d", i,RunEra);
   parLis.emplace_back(varName,varName, coeffPoly[i].getValue(),ParMin,ParMax);
  }	      
  for (int i=0;i<numParameters;++i){
   coefLis->add(parLis[i]);
  }	      
//  gROOT->ProcessLine(".L RooBernsteinSideband.cxx+");
  BernSideBand    = new RooBernsteinSideband(Form("BernSideBand_bin%d_%d",Q2Bin,RunEra),Form("BernSideBand_bin%d_%d",Q2Bin,RunEra),*ctL,*ctK,*phi,*coefLis,maxDegree1,maxDegree2,maxDegree3);

  RooWorkspace* wsb =  new RooWorkspace("wsb","workspace sideband");
//set to the fit range!!!
  RooRealVar *max_sbl=new RooRealVar(Form("max_sbl_bin%d_%d",Q2Bin,RunEra),Form("max_sbl_bin%d_%d",Q2Bin,RunEra),XMaxSBL);
  RooRealVar *min_sbr=new RooRealVar(Form("min_sbr_bin%d_%d",Q2Bin,RunEra),Form("min_sbr_bin%d_%d",Q2Bin,RunEra),XMinSBR);
  wsb->import(*covMatrix,Form("covMatrix_bin%d_%d",Q2Bin,RunEra));
  wsb->import(*BernSideBand);
  wsb->import(*bkg_mass_sb);
  wsb->import(*max_sbl);
  wsb->import(*min_sbr);
  wsb->writeToFile(OutSaveFileName);
  cout<<"save  workspace ==> wsb in "<<OutSaveFileName<<"\n"<<endl;
/*  double arglist[2]; 
  int err = 0;
   arglist[0]= 150000; // maximum iterations
  arglist[1]= 1.0; 
  for(int j=0;j<2;++j){
     std::vector<Variable> var; 
     double tmp_value, tmp_error;
     for(Variable &var : Minuit->getVaraibles()) {
     int index = var.getFitterIndex();
      Minuit->GetParameter(index, tmp_value, tmp_error);
      sprintf(varName, "p%d", index);
         if (index>5){
          if (tmp_value<=tmp_error && fabs(tmp_value)<0.001){
           Minuit->mnparm(index, varName, 0.,0.001, -500,500.,err);
           Minuit->FixParameter(index);
          }
          else{
           Minuit->mnparm(index, varName, fabs(tmp_value), 0.00001, 0.0,3000.,err);
          }
         }
//	std::cout<<"p("<<Index<<") = "<<" = "<<tmp_value<<std::endl ;
     }
       std::cout<<"============================================================\n"<<std::endl ;
       Minuit->mnexcm("SHOW PAR",arglist,1,err);
//       Minuit->mnexcm("MIGRAD",arglist,2,err);
         fitter.fit();
//       Minuit->Migrad();
    }   
  cout<<"                  ===*** End  Fit ***=== "<<endl;
  cout<<"                  ===*** End  Fit ***=== "<<endl;
  cout<<"                  ===*** End  Fit ***=== "<<endl;
  for(int i=0;i<numParameters;++i) {
    sprintf(varName, "p%d", i);
    double tmp_value = coeffPoly[i].getValue();
    double tmp_error = coeffPoly[i].getError();
    std::cout<<i<<" =>  "<<varName<<" = "<<tmp_value<<"+/-"<<tmp_error<<"\n"<<std::endl ;
  }
 */// 
 
  xCoeffNorm =0.0;
  xCoeffIndex=-1;
  double coeffy =0.0;
  double errory =0.0;
  std::fstream *parListOutput =  new std::fstream(ListParName,ios::out);
  std::fstream *parPlotOutput =  new std::fstream(ListPloName,ios::out);
  if(parListOutput->is_open() && parPlotOutput->is_open() ){
   std::cout<<"Open: "<<ListParName<<std::endl ;
   for(int i=0;i<numParameters;++i) {
    if( fabs(coeffPoly[i].getValue())>1. && fabs(coeffPoly[i].getError()>0.) ){
     double xCoeffNormTmp = coeffPoly[i].getValue()/fabs(coeffPoly[i].getError());
     if (xCoeffNormTmp>xCoeffNorm) {
      xCoeffNorm=xCoeffNormTmp;
      xCoeffIndex=i;
     }
    }
    if(fabs(coeffPoly[i].getValue())>fabs(coeffPoly[i].getError())){
     coeffy=  coeffPoly[i].getValue();
     errory=  coeffPoly[i].getError();
     std::cout<<Form("RESULTS==>  p(%d)=%f+/-%f rate=%f => [%s] ",i,coeffy,errory,coeffy/errory,ParCheck[i])<<std::endl;
    }else{
     coeffy=  0.0;
     errory=  0.0;
    } 
    
    *parListOutput <<std::scientific << std::setprecision(20)<< coeffy<<"+/-"<<errory<<std::endl;
    *parPlotOutput <<std::scientific << std::setprecision(20)<< coeffPoly[i].getValue()<<"+/-"<<coeffPoly[i].getError()<<std::endl;
   }
   
//
   parListOutput->close();
   parPlotOutput->close();
   std::cout<<"Close: "<<ListParName<<std::endl ;
   std::cout<<"Close: "<<ListPloName<<std::endl ;
  }else{
   if(!parListOutput->is_open()) std::cout<<"Error: can not open "<<ListParName<<std::endl ;
   if(!parPlotOutput->is_open()) std::cout<<"Error: can not open "<<ListPloName<<std::endl ;
   std::cout<<Form("Error!!!")<<std::endl ;
   exit(1);
  }
  if(xCoeffIndex>0){
   std::cout<<Form("WARNING: Normalization could be fixed better if p(%d)=1",xCoeffIndex)<<std::endl ;
  }
 
  OutFile->cd();
    
  
  TH3D* HSBFunc 	  = new TH3D( "HSBFunc"          , "HSBFunc",		NFact*xCosLHBin, XMinCosThetaL, XMaxCosThetaL,
 										NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK,
										NFact*xPhiHBin , XMinPhi, XMaxPhi );

//   TH2D* HSBFuncXY 	  = new TH2D( "HSBFuncXY"        , "HSBFuncXY",		NFact*xCosLHBin, XMinCosThetaL, XMaxCosThetaL,
//  										NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK);
// 
// //   TH2D* HSBFuncZY 	  = new TH2D( "HSBFuncZY"        , "HSBFuncZY",		NFact*xPhiHBin , XMinPhi, XMaxPhi,
// //  										NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK );
// // 	
//   TH2D* HSBFuncZY 	  = new TH2D( "HSBFuncZY"        , "HSBFuncZY",		NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK,
//  										NFact*xPhiHBin , XMinPhi, XMaxPhi);
// 	
//   TH2D* HSBFuncZX 	  = new TH2D( "HSBFuncZX"        , "HSBFuncZX",		NFact*xPhiHBin , XMinPhi, XMaxPhi,
//  										NFact*xCosLHBin, XMinCosThetaL, XMaxCosThetaL );
// 	
	
//   TH3D* HSideBandRecoTest = new TH3D( "HSideBandRecoTest", "HSideBandRecoTest", NFact*xCosLHBin, XMinCosThetaL, XMaxCosThetaL,
//  										NFact*xCosKHBin, XMinCosThetaK, XMaxCosThetaK,
// 										NFact*xPhiHBin , XMinPhi, XMaxPhi );
//   totalParams=0;
  GooFit::Observable xReco_w("xReco_w"  ,0., 2000000.)  ;
  GooFit::Observable xGene_w("xGene_w"  ,0., 2000000.)  ;
  GooFit::Observable BWidthX("BWidthX"	,0., 2.*XMaxCosThetaL);
  GooFit::Observable BWidthY("BWidthY"	,0., 2.*XMaxCosThetaK);
  GooFit::Observable BWidthZ("BWidthZ"	,0., 2.*XMaxPhi      );
  vector<GooFit::Observable> obsPolyTest;
  obsPolyTest.push_back(xCosL_x);
  obsPolyTest.push_back(xCosK_y);
  obsPolyTest.push_back(xPhiK_z);
  obsPolyTest.push_back(BWidthX);
  obsPolyTest.push_back(BWidthY);
  obsPolyTest.push_back(BWidthZ);
  vector<GooFit::Observable> dataTest;
  dataTest.push_back(xCosL_x);
  dataTest.push_back(xCosK_y);
  dataTest.push_back(xPhiK_z);
  dataTest.push_back(BWidthX);
  dataTest.push_back(BWidthY);
  dataTest.push_back(BWidthZ);
  UnbinnedDataSet* dataPlotTest = new GooFit::UnbinnedDataSet(dataTest);

  std::vector<double> DataBinContent;

  double xBinwPlot = HSBFunc->GetXaxis()->GetBinWidth(1) ;
  double yBinwPlot = HSBFunc->GetYaxis()->GetBinWidth(1) ;
  double zBinwPlot = HSBFunc->GetZaxis()->GetBinWidth(1) ;
  for(int i = 0; i < xCosLHBin*NFact ; ++i) {
   double xi = XMinCosThetaL + xBinwPlot/2.+ i*xBinwPlot;
   for(int j = 0 ; j < xCosKHBin*NFact ; ++j) {
    double yj = XMinCosThetaK + yBinwPlot/2.+ j*yBinwPlot;
    for(int k = 0 ; k < xPhiHBin*NFact  ; ++k) {
    double zk = XMinPhi + zBinwPlot/2.+ k*zBinwPlot;
//    if (xi<=XMaxCosThetaL && yj<=XMaxCosThetaK && zk<=XMaxPhi){
    xCosL_x.setValue(xi);
    xCosK_y.setValue(yj);
    xPhiK_z.setValue(zk);
   // cout <<xGene_w.getValue() <<endl;
//    xReco_w.setValue(1.);
//    xGene_w.setValue(2.);
    BWidthX.setValue(xBinwPlot);
    BWidthY.setValue(yBinwPlot);
    BWidthZ.setValue(zBinwPlot);
    
    dataPlotTest->addEvent();
    DataBinContent.push_back(HxReco->GetBinContent(i,j,k));
//    }else{
//    printf("(x,y,z)=%f %f %f (xmax,ymax,zmax) = %f %f %f\n",xi,yj,zk,XMaxCosThetaL,XMaxCosThetaK,XMaxPhi);
//    printf("(x,y,z)=%f %f %f\n",xCosL_x.getValue(),xCosK_y.getValue(),xPhiK_z.getValue());
//    }
    }
   }
  }

//      GooFit::Variable xBinWidthPlot("xBinWidth",xBinwPlot );
//      GooFit::Variable yBinWidthPlot("yBinWidth",yBinwPlot );
//      GooFit::Variable zBinWidthPlot("zBinWidth",zBinwPlot );
//      vector<GooFit::Variable> BinwsPlot;
//      BinwsPlot.push_back(xBinWidth);
//      BinwsPlot.push_back(yBinWidth);
//      BinwsPlot.push_back(zBinWidth);

  // totalParams=0;
//   GooFit::BernsteinPdf    *modelPlot=0;
//     GooFit::FastBernsteinPdf    *modelPlot=0;
//   if(integral){
//      GooFit::BernsteinTestPdf   *modelPlot     =  new GooFit::BernsteinTestPdf("modelPlot",obsPolyTest,coeffPoly,limits,BinwsPlot,maxDegree1,maxDegree2,maxDegree3);
  //}else{
//   GooFit::FastBernsteinPdf    *   modelPlot     =  new GooFit::FastBernsteinPdf("modelPlot",obsPoly,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3);
      GooFit::BernsteinTestPdf   *modelPlot     =  new GooFit::BernsteinTestPdf("modelPlot",obsPolyTest,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3,1);
//      GooFit::BernsteinTestPdf   *modelPlot     =  new GooFit::BernsteinTestPdf("modelPlot",obsPolyTest,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3,1);
  //} 


//  GooFit::BernsteinPdf    modelPlot("model",obsPolyPlot,coeffPoly,limits,Binws,maxDegree1,maxDegree2,maxDegree3);
//  GooFit::FastBernsteinPdf    modelPlot("modelPlot",obsPolyPlot,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3);
  
//  modelPlot.setData(dataReco);

//  double Vol3D = xBinwPlot*yBinwPlot*zBinwPlot;
  modelPlot->setData(dataPlotTest);
  vector<vector<double> > pdfVals_Model3D = modelPlot->getCompProbsAtDataPoints();
  double pdfVal = 0.0;
  double totalPdf = 0;
  double dataPlotEntries=dataPlotTest->getNumEvents();
  for (int i = 0; i < dataPlotEntries; ++i) {
    dataPlotTest->loadEvent(i);
    pdfVal = pdfVals_Model3D[0][i];
    totalPdf += pdfVal;
  }
  double Chi2 =0.;
  int iskip =0;
  int NDegreeofFreedomN =0;
//  int NDegreeofFreedom =-(NumParamFree+1);
  double SBEntries= HxReco->GetEntries();
//  cout <<"totalPdf = "<<totalPdf*Vol3D<<endl;
  cout <<"totalPdf = "<<totalPdf<<endl;
  cout <<"SBEntries= "<<SBEntries<<endl;
  cout <<"SBEntries= "<<dataReco->getNumEvents()<<"  (Check)"<<endl;
  cout <<"nentgrid = "<<dataPlotTest->getNumEvents()<<endl;
  double LLChi2 =0.;
  double ProbFuncTot =0;
//  int icount=0;
   for (int i = 0; i < dataPlotEntries; ++i) {
//  for (int i = 0; i < dataReco->getNumEvents(); ++i) {
    dataPlotTest->loadEvent(i);
    pdfVal = pdfVals_Model3D[0][i];
    if (pdfVal<0.0) std::cout<<"Warning!!!: SB Model "<<pdfVal <<"0 in CosL="<<xCosL_x.getValue()<<" CosK="<<xCosK_y.getValue()<<" Phi="<<xPhiK_z.getValue()<<std::endl;
//
//    HSBFunc->Fill(xCosL_x.getValue(),xCosK_y.getValue(),xPhiK_z.getValue(), SBEntries*pdfVal/totalPdf);
//
//---> perche' totalPdf e' 1...
//
    HSBFunc->Fill(xCosL_x.getValue(),xCosK_y.getValue(),xPhiK_z.getValue(), SBEntries*pdfVal);
//
//     HSBFuncXY->Fill(xCosL_x.getValue(),xCosK_y.getValue(), SBEntries*pdfVal/totalPdf);
//     HSBFuncZY->Fill(xPhiK_z.getValue(),xCosK_y.getValue(),xPhiK_z.getValue(), SBEntries*pdfVal/totalPdf);
// //   HSBFuncZY->Fill(xPhiK_z.getValue(),xCosK_y.getValue(), SBEntries*pdfVal/totalPdf);
//     HSBFuncZX->Fill(xPhiK_z.getValue(),xCosL_x.getValue(), SBEntries*pdfVal/totalPdf);
     if(pdfVal>0.) {
      double ProbFunc = SBEntries*pdfVal/totalPdf;
      double Chi2Temp = ( DataBinContent[i] - ProbFunc)*(DataBinContent[i] - ProbFunc)/ProbFunc;
      Chi2 = Chi2+( DataBinContent[i] - ProbFunc)*(DataBinContent[i] - ProbFunc)/ProbFunc;
      if(DataBinContent[i]!=0){
        LLChi2 = LLChi2+DataBinContent[i]*(log(DataBinContent[i])-log(ProbFunc));
      } else {
        iskip++;
      }	
//      if(DataBinContent[i]!=0) LLChi2 = LLChi2+DataBinContent[i]*(log(DataBinContent[i]/ProbFunc))+ProbFunc-DataBinContent[i];
      ProbFuncTot = ProbFunc+ProbFuncTot;
//       if(Chi2Temp>1.) {
//        icount++;
//        std::cout<<"======== "<<icount<<" ======================================================================\n"<<std::endl;
//        std::cout<<"==> Chi2Temp = "<<Chi2Temp<<" CosL="<<xCosL_x.getValue()<<" CosK"<<xCosK_y.getValue()<<" Phi="<<xPhiK_z.getValue()<<"\n"<<std::endl;
//        std::cout<<"==> ProbFunc ="<<ProbFunc<<" DataCont="<<DataBinContent[i]<<"\n"<<std::endl;
//       } 
      NDegreeofFreedomN++; 
     }
     //cout <<xGene_w.getValue() <<endl;
  }
  LLChi2=2*LLChi2;
  int NDegreeofFreedomR = NDegreeofFreedomN-NumParamFree;
  cout <<"Integrated Probability Function "<<ProbFuncTot <<endl;
  cout <<"In LLchi2 skipped cell = "<<iskip<<endl;
  double  PValueModelMin = TMath::Prob(Chi2, NDegreeofFreedomR);
  double  PValueModelMax = TMath::Prob(Chi2, NDegreeofFreedomN);
  double  PValueLLModelMin  = TMath::Prob(LLChi2, NDegreeofFreedomR);
  double  PValueLLModelMax  = TMath::Prob(LLChi2, NDegreeofFreedomN);
  cout <<"------------------------------------------------------- "<<endl;
  cout <<"Even Binning ("<<xCosLHBin*xCosKHBin*xPhiHBin <<" Bins) "<<endl;
  cout <<"------------------------------------------------------- "<<endl;
  cout <<"Chi2 Sideband			= "<<Chi2<<endl;
  cout <<"Chi2/NDOF			= "<<Chi2/NDegreeofFreedomN<<endl;
  cout <<"Chi2/NDOF_Reduced		= "<<Chi2/NDegreeofFreedomR<<endl;
  cout <<"P-Value		Min	= "<<PValueModelMin<<endl;
  cout <<"P-Value		Max	= "<<PValueModelMax<<endl;
  cout <<"LLChi2 Sideband		= "<<LLChi2<<endl;
  cout <<"LLChi2/NDOF			= "<<LLChi2/(NDegreeofFreedomN)<<endl;
  cout <<"LLChi2/NDOF_Reduced		= "<<LLChi2/(NDegreeofFreedomR)<<endl;
  cout <<"LL P-Value		Min	= "<<PValueLLModelMin<<endl;
  cout <<"LL P-Value		Max	= "<<PValueLLModelMax<<endl;
  cout <<"NDOF	(reduced)		= "<<NDegreeofFreedomR<<endl;
  cout <<"NDOF				= "<<NDegreeofFreedomN<<endl;
  cout <<"Num Free Param.		= "<<NumParamFree<<endl;
  cout <<"------------------------------------------------------- "<<endl;
  printf(Form("Q2Bin, Poly Degree(1,2,3),initial and final free params ==> %d & %d & %d & %d & %d & %d\\\\ \n",Q2Bin,maxDegree1,maxDegree2,maxDegree3,numParameters,NumParamFree));
  HSBFunc->Sumw2();

//======================================================================================
// Adaptive Binning GOF...
//======================================================================================  
TCanvas* ca = new TCanvas("ca","Adaptive Binning Histograms",200,200,800,800);
  ca->Divide(2,2);   
  
//
  int iCorreTagOrig = CorreAdaptX.size();
//   if( fmod(iCorreTag,MinContAdaptBin)==0){
  xAdaptNumBinC = int(iCorreTagOrig/MinContAdaptBin);
//   }else{
//    xAdaptNumBinC = int(iCorreTag/MinContAdaptBin)-1;
//   } 
//   std::cout<<Form("It will be added an extra bin with N. of entries = %d ?",iCorreTag-xAdaptNumBinC*MinContAdaptBin)<<std::endl;
  int iCorreTag=xAdaptNumBinC*MinContAdaptBin;
  std::cout<<"TKDTreeBinning Start   "<<std::endl;
  std::cout<<"TKDTreeBinning set iCorreTag	  = "<<iCorreTag<<std::endl;
  std::cout<<"TKDTreeBinning xAdaptNumBinC    = "<<xAdaptNumBinC<<std::endl;
  double *RecoAdaptC = new double[NDim*iCorreTag];
  for (int iC=0;iC<iCorreTag;iC++) { 
   RecoAdaptC[iC]	     =  CorreAdaptX[iC];
   RecoAdaptC[iC+  iCorreTag]=  CorreAdaptY[iC];
   RecoAdaptC[iC+2*iCorreTag]=  CorreAdaptZ[iC];
  } 
  RecoAdaptBinsC = new TKDTreeBinning(iCorreTag, NDim, RecoAdaptC, xAdaptNumBinC);
  int nbinsC =RecoAdaptBinsC->GetNBins();
  std::cout<<"TKDTreeBinning nbinsC    = "<<nbinsC<<std::endl;
  TH2Poly* h2polxyContC = new TH2Poly("h2polxyContC", "adapt. binning contents [CosL CosK]", RecoAdaptBinsC->GetDataMin(0), RecoAdaptBinsC->GetDataMax(0), RecoAdaptBinsC->GetDataMin(1), RecoAdaptBinsC->GetDataMax(1));
  TH2Poly* h2polxzContC = new TH2Poly("h2polxzContC", "adapt. binning contents [CosL Phi ]", RecoAdaptBinsC->GetDataMin(0), RecoAdaptBinsC->GetDataMax(0), RecoAdaptBinsC->GetDataMin(2), RecoAdaptBinsC->GetDataMax(2));
  TH2Poly* h2polxyDensC = new TH2Poly("h2polxyDensC", "adapt. binning density [CosL CosK]", RecoAdaptBinsC->GetDataMin(0), RecoAdaptBinsC->GetDataMax(0), RecoAdaptBinsC->GetDataMin(1), RecoAdaptBinsC->GetDataMax(1));
  TH2Poly* h2polxzDensC = new TH2Poly("h2polxzDensC", "adapt. binning density [CosL Phi ]", RecoAdaptBinsC->GetDataMin(0), RecoAdaptBinsC->GetDataMax(0), RecoAdaptBinsC->GetDataMin(2), RecoAdaptBinsC->GetDataMax(2));
  const double* binsMinEdgesC = RecoAdaptBinsC->GetBinsMinEdges();
  const double* binsMaxEdgesC = RecoAdaptBinsC->GetBinsMaxEdges();
  int edgeDim=0;       
  std::vector<double> DataAdaptBinContent;
  std::vector<double> Vol3DAdaptBin;
  const double* xyzvar;
  const double* xyzbinw;
//   GooFit::Observable xReco_w("xReco_w"  ,0., 2000000.)  ;
//   GooFit::Observable xGene_w("xGene_w"  ,0., 2000000.)  ;
//   GooFit::Observable BWidthX("BWidthX"	,0., 2.*XMaxCosThetaL);
//   GooFit::Observable BWidthY("BWidthY"	,0., 2.*XMaxCosThetaK);
//   GooFit::Observable BWidthZ("BWidthZ"	,0., 2.*XMaxPhi      );
  vector<GooFit::Observable> obsPolyAdapt;
  obsPolyAdapt.push_back(xCosL_x);
  obsPolyAdapt.push_back(xCosK_y);
  obsPolyAdapt.push_back(xPhiK_z);
// //   obsPolyAdapt.push_back(xReco_w);
// //   obsPolyAdapt.push_back(xGene_w);
  obsPolyAdapt.push_back(BWidthX);
  obsPolyAdapt.push_back(BWidthY);
  obsPolyAdapt.push_back(BWidthZ);
  UnbinnedDataSet* dataAdapt = new GooFit::UnbinnedDataSet(obsPolyAdapt);
  std::vector<int > AdaptExcludedEvents;
  for (int i = iCorreTag; i < iCorreTagOrig; ++i) {
  
   double point[3] = {CorreAdaptX[i],CorreAdaptY[i],CorreAdaptZ[i]};
   
   AdaptExcludedEvents.push_back(RecoAdaptBinsC->FindBin(point));
   cout<<Form("CosL=%f CosK=%f Phi=%f point = %d",CorreAdaptX[i],CorreAdaptY[i],CorreAdaptZ[i],RecoAdaptBinsC->FindBin(point))<<endl;
   
  }
  double vol =0;
  for (int i = 0; i < nbinsC; ++i) {
     edgeDim = i * NDim;
     h2polxyContC->AddBin(binsMinEdgesC[edgeDim], binsMinEdgesC[edgeDim + 1], binsMaxEdgesC[edgeDim], binsMaxEdgesC[edgeDim + 1]);
     h2polxzContC->AddBin(binsMinEdgesC[edgeDim], binsMinEdgesC[edgeDim + 2], binsMaxEdgesC[edgeDim], binsMaxEdgesC[edgeDim + 2]);
     h2polxyDensC->AddBin(binsMinEdgesC[edgeDim], binsMinEdgesC[edgeDim + 1], binsMaxEdgesC[edgeDim], binsMaxEdgesC[edgeDim + 1]);
     h2polxzDensC->AddBin(binsMinEdgesC[edgeDim], binsMinEdgesC[edgeDim + 2], binsMaxEdgesC[edgeDim], binsMaxEdgesC[edgeDim + 2]);
     xyzvar  = RecoAdaptBinsC->GetBinCenter(i);
     xyzbinw = RecoAdaptBinsC->GetBinWidth(i);
     xCosL_x.setValue(xyzvar[0]);
     xCosK_y.setValue(xyzvar[1]);
     xPhiK_z.setValue(xyzvar[2]);
//      xReco_w.setValue(1.);
//      xGene_w.setValue(2.);
     BWidthX.setValue(xyzbinw[0]);
     BWidthY.setValue(xyzbinw[1]);
     BWidthZ.setValue(xyzbinw[2]);
     vol += xyzbinw[0]*xyzbinw[1]*xyzbinw[2];
//      cout<<"================================"<<endl;
//      cout<<"NBIN    = "<<i<<endl;
//      cout<<"BWidthX = "<<xyzbinw[0]<<endl;
//      cout<<"BWidthY = "<<xyzbinw[1]<<endl;
//      cout<<"BWidthZ = "<<xyzbinw[2]<<endl;
//      cout<<"BWidthX = "<<binsMaxEdgesC[edgeDim]-binsMinEdgesC[edgeDim]<<endl;
//      cout<<"BWidthY = "<<binsMaxEdgesC[edgeDim + 1]-binsMinEdgesC[edgeDim + 1]<<endl;
//      cout<<"BWidthZ = "<<binsMaxEdgesC[edgeDim + 2]-binsMinEdgesC[edgeDim + 2]<<endl;
     dataAdapt->addEvent();
     DataAdaptBinContent.push_back(RecoAdaptBinsC->GetBinContent(i));
     Vol3DAdaptBin.push_back(xyzbinw[0]*xyzbinw[1]*xyzbinw[2]);
  }
  cout <<"Vol tot  [Adapt] = "<<vol<<endl;
  
//  
//  GooFit::FastBernsteinPdf   *modelAdapt     =  new GooFit::FastBernsteinPdf("modelAdapt",obsPolyAdapt,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3);
  GooFit::BernsteinTestPdf   *modelAdapt     =  new GooFit::BernsteinTestPdf("modelAdapt",obsPolyAdapt,coeffPoly,limits,maxDegree1,maxDegree2,maxDegree3,1);
//

  modelAdapt->setData(dataAdapt);
  vector<vector<double> > pdfVals_Model3DAdapt = modelAdapt->getCompProbsAtDataPoints();
  pdfVal = 0.0;
  totalPdf = 0;
  double dataAdaptEntries=dataAdapt->getNumEvents();
  for (int i = 0; i < dataAdaptEntries; ++i) {
    dataAdapt->loadEvent(i);
    pdfVal = pdfVals_Model3DAdapt[0][i];
    totalPdf += pdfVals_Model3DAdapt[0][i];
//    totalPdf += pdfVals_Model3DAdapt[0][i]*Vol3DAdaptBin[i];
  }
  cout <<"totalPdf [Adapt] = "<<totalPdf<<endl;
//  cout <<Form("dataAdaptEntries [%f]must be = nbinsC [%d] ",dataAdaptEntries,nbinsC)<<endl;
  double AdaptProbFuncTot =0;
//double   icount = 0;
  double lambda = 2./3.;
  double AdaptChi2 = 0;
  double AdaptLLChi2 = 0;
  double AdaptPowerDivergence = 0. ;
  
  int NDegreeofFreedomAdaptN = 0;  
//  int NDegreeofFreedomAdapt =-(NumParamFree);  
  for (int ii = 0; ii < nbinsC; ++ii){
       h2polxyContC->SetBinContent(ii+1, RecoAdaptBinsC->GetBinContent(ii));
       h2polxzContC->SetBinContent(ii+1, RecoAdaptBinsC->GetBinContent(ii));
       h2polxyDensC->SetBinContent(ii+1, RecoAdaptBinsC->GetBinDensity(ii));
       h2polxzDensC->SetBinContent(ii+1, RecoAdaptBinsC->GetBinDensity(ii));
       dataAdapt->loadEvent(ii);
       pdfVal = pdfVals_Model3DAdapt[0][ii];
       for (int iii = 0; iii < iCorreTagOrig-iCorreTag; ++iii) {
        if(AdaptExcludedEvents[iii]==ii){
	 DataAdaptBinContent[ii]++;
        }
       }	 	

       if( DataAdaptBinContent[ii]!=MinContAdaptBin) std::cout<<Form("DataAdaptBinContent[%d]=%f",ii,DataAdaptBinContent[ii])<<std::endl;
//       if(pdfVal!=0.&& DataAdaptBinContent[ii]==MinContAdaptBin) {
       if(pdfVal!=0.) {
//        double ProbFunc = pdfVal/totalPdf;
//        double ProbFunc = iCorreTag*pdfVal*Vol3DAdaptBin[ii]/totalPdf;
        double ProbFunc = SBEntries*pdfVal/totalPdf;
	AdaptPowerDivergence = AdaptPowerDivergence+ (DataAdaptBinContent[ii])*(pow(DataAdaptBinContent[ii]/ProbFunc,lambda)-1);

      double AdaptChi2Temp = ( DataAdaptBinContent[ii] - ProbFunc)*(DataAdaptBinContent[ii] - ProbFunc)/(ProbFunc);
//      AdaptLLChi2 = AdaptLLChi2+DataAdaptBinContent[ii]*(log(DataAdaptBinContent[ii])-log(ProbFunc));
      AdaptLLChi2 = AdaptLLChi2+DataAdaptBinContent[ii]*(log(DataAdaptBinContent[ii])-log(ProbFunc))+ProbFunc-DataAdaptBinContent[ii];
//      AdaptChi2 = AdaptChi2+ AdaptChi2Temp;
      AdaptChi2 = AdaptChi2+ ( DataAdaptBinContent[ii] - ProbFunc)*(DataAdaptBinContent[ii] - ProbFunc)/(ProbFunc);
      AdaptProbFuncTot = ProbFunc+AdaptProbFuncTot;
//        if(AdaptChi2Temp>1.) {
// 	icount++;
// 	std::cout<<"======== "<<icount<<" ======================================================================\n"<<std::endl;
// 	std::cout<<"==> AdaptChi2Temp = "<<AdaptChi2Temp<<" CosL="<<xCosL_x.getValue()<<" CosK"<<xCosK_y.getValue()<<" Phi="<<xPhiK_z.getValue()<<"\n"<<std::endl;
// 	std::cout<<"==> ProbFunc ="<<ProbFunc<<" DataAdaptCont="<<DataAdaptBinContent[ii]<<"\n"<<std::endl;
//        } 
      NDegreeofFreedomAdaptN++;
     }
   }
  AdaptPowerDivergence = 2.*AdaptPowerDivergence/(lambda+1)/lambda ;	
  cout <<"Integrated Probability Function [Adapt]"<<ProbFuncTot <<endl;
  int NDegreeofFreedomAdaptR = NDegreeofFreedomAdaptN-NumParamFree;
  AdaptLLChi2=2*AdaptLLChi2;
  PValueModelMin = TMath::Prob(AdaptChi2, NDegreeofFreedomAdaptR);
  PValueModelMax = TMath::Prob(AdaptChi2, NDegreeofFreedomAdaptN);
  PValueLLModelMin =  TMath::Prob(AdaptLLChi2, NDegreeofFreedomAdaptR);
  PValueLLModelMax =  TMath::Prob(AdaptLLChi2, NDegreeofFreedomAdaptN);
  double PValuePDModelMin =  TMath::Prob(AdaptPowerDivergence, NDegreeofFreedomAdaptR);
  double PValuePDModelMax =  TMath::Prob(AdaptPowerDivergence, NDegreeofFreedomAdaptN);
  cout <<"------------------------------------------------------- "<<endl;
  cout <<"Adaptive Binning ("<<xAdaptNumBinC <<" Bins)\n"          <<endl;
  cout <<"MinContAdaptBin = "<<MinContAdaptBin                     <<endl;
  cout <<"------------------------------------------------------- "<<endl;
  cout <<"Chi2 Sideband			[Adapt] = "<<AdaptChi2<<endl;
  cout <<"Chi2/NDOF			[Adapt] = "<<AdaptChi2/NDegreeofFreedomAdaptR<<endl;
  cout <<"Chi2/NDOF_Reduced		[Adapt] = "<<AdaptChi2/NDegreeofFreedomAdaptN<<endl;
  cout <<"P-Value			Min	[Adapt] = "<<PValueModelMin<<endl;
  cout <<"P-Value			Max	[Adapt] = "<<PValueModelMax<<endl;
  cout <<"LL Chi2 Sideband		[Adapt] = "<<AdaptLLChi2<<endl;
  cout <<"LL Chi2/NDOF			[Adapt] = "<<AdaptLLChi2/NDegreeofFreedomAdaptN<<endl;
  cout <<"LL Chi2/NDOF_Reduced		[Adapt] = "<<AdaptLLChi2/NDegreeofFreedomAdaptR<<endl;
  cout <<"LL P-Value		Min	[Adapt] = "<<PValueLLModelMin<<endl;
  cout <<"LL P-Value		Max	[Adapt] = "<<PValueLLModelMax<<endl;
  cout <<"PowerDivergence			[Adapt] = "<<AdaptPowerDivergence<<endl;
  cout <<"PowerDivergence/NDOF		[Adapt] = "<<AdaptPowerDivergence/NDegreeofFreedomAdaptN<<endl;
  cout <<"PowerDivergence/NDOF_Reduced	[Adapt] = "<<AdaptPowerDivergence/NDegreeofFreedomAdaptR<<endl;
  cout <<"PD P-Value		Min	[Adapt] = "<<PValuePDModelMin<<endl;
  cout <<"PD P-Value		Max	[Adapt] = "<<PValuePDModelMax<<endl;
  cout <<"NDOF	(reduced)		[Adapt] = "<<NDegreeofFreedomAdaptR<<endl;
  cout <<"NDOF				[Adapt] = "<<NDegreeofFreedomAdaptN<<endl;
  cout <<"Num Free Param.	   	   = "<<NumParamFree<<endl;
  cout <<"------------------------------------------------------- "<<endl;

  ca->cd(1);
//  h2polxyContC->Draw("lego");
  h2polxyContC->Draw("COLZ L");
  ca->Update();   
  ca->cd(2);
//  h2polxzContC->Draw("lego");
  h2polxzContC->Draw("COLZ L");
  ca->Update();   
  ca->cd(3);
  h2polxyDensC->Draw("COLZ L");
  ca->Update();   
  ca->cd(4);
  h2polxzDensC->Draw("COLZ L");
  ca->Update();   
 //    h2polxyC->Draw("LEGO");
//    ca->Update();   
//    ca->cd(4);
//    h2polxzC->Draw("LEGO");
//    ca->Update();   
//
// Adaptive Binning...End
//  



//  cout<<std::scientific << std::setprecision(40)<<"Norm PDF SideBand = "<<modelPlot->normalize()<<";"<<endl;
//==== closure test plots
//   modelPlot->setData(dataPlot);
//   vector<vector<double> > pdfVals_Model3D_Reco = modelPlot->getCompProbsAtDataPoints();
//   pdfVal = 0.0;
//   for (int i = 0; i < dataPlot->getNumEvents(); ++i) {
//     dataReco->loadEvent(i);
//     pdfVal = pdfVals_Model3D_Reco[0][i];
//     if (pdfVal<0.0) std::cout<<"Warning!!!: Effi Model Reco Test"<<pdfVal <<"0 in CosL="<<xCosL_x.getValue()<<" CosK="<<xCosK_y.getValue()<<" Phi="<<xPhiK_z.getValue()<<std::endl;
// //     double xL = xCosL_x.getValue();
// //     double yK = xCosK_y.getValue();
// //     double zP = xPhiK_z.getValue();
// //    HSBFunc->Fill(xL,yK,zP, pdfVal);
//      HSideBandRecoTest->Fill(xCosL_x.getValue(),xCosK_y.getValue(),xPhiK_z.getValue(), pdfVal*xGene_w.getValue());
// //     HSideBandCosLFunc->Fill(xL, pdfVal);
// //     HSideBandCosKFunc->Fill(yK, pdfVal);
// //     HSideBandPhiFunc ->Fill(zP, pdfVal);
//   }
//  HSideBandRecoTest->Sumw2();

  TH1D* HSBFuncX  = (TH1D*) HSBFunc->ProjectionX("HSBFuncX",1,HSBFunc->GetNbinsY(),1,HSBFunc->GetNbinsZ());HSBFuncX->SetTitle(Form("Cos#theta_{L} Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
  TH1D* HSBFuncY  = (TH1D*) HSBFunc->ProjectionY("HSBFuncY",1,HSBFunc->GetNbinsX(),1,HSBFunc->GetNbinsZ());HSBFuncY->SetTitle(Form("Cos#theta_{K} Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
  TH1D* HSBFuncZ  = (TH1D*) HSBFunc->ProjectionZ("HSBFuncZ",1,HSBFunc->GetNbinsX(),1,HSBFunc->GetNbinsY());HSBFuncZ->SetTitle(Form("#varphi Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
  TH2D* HSBFuncXY = (TH2D*) HSBFunc->Project3D("xy");HSBFuncXY->SetTitle(Form("2D Model Projection (Cos#theta_{l},Cos#theta_{k})    [q^{2} bin %d Run II %d]",Q2Bin,RunEra));
  TH2D* HSBFuncZY = (TH2D*) HSBFunc->Project3D("zy");HSBFuncZY->SetTitle(Form("2D Model Projection (#varphi,Cos#theta_{k})    [q^{2} bin %d Run II %d]",Q2Bin,RunEra));
  TH2D* HSBFuncZX = (TH2D*) HSBFunc->Project3D("zx");HSBFuncZX->SetTitle(Form("2D Model Projection (#varphi,Cos#theta_{l})   [q^{2} bin %d Run II %d]",Q2Bin,RunEra));
//  gStyle->SetOptStat(111111);
//  gStyle -> SetOptFit(111111);
  
   TH3D *HSideBand3D  = (TH3D*)HxReco->Clone(); HSideBand3D->SetName("HSideBand3D");HSideBand3D->Sumw2();
  
  
  
   TH1D* HSideBand3DX  = (TH1D*) HSideBand3D->ProjectionX("HSideBand3DX",1,HSideBand3D->GetNbinsY(),1,HSideBand3D->GetNbinsZ());HSideBand3DX->SetTitle(Form("Cos#theta_{L} Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
   TH1D* HSideBand3DY  = (TH1D*) HSideBand3D->ProjectionY("HSideBand3DY",1,HSideBand3D->GetNbinsX(),1,HSideBand3D->GetNbinsZ());HSideBand3DY->SetTitle(Form("Cos#theta_{K} Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
   TH1D* HSideBand3DZ  = (TH1D*) HSideBand3D->ProjectionZ("HSideBand3DZ",1,HSideBand3D->GetNbinsX(),1,HSideBand3D->GetNbinsY());HSideBand3DZ->SetTitle(Form("#varphi Projection [q^{2} bin %d run %d]",Q2Bin,RunEra));
   TH2D* HSideBand3DXY = (TH2D*) HSideBand3D->Project3D("xy");HSideBand3DXY->SetTitle(Form("2D Projection (Cos#theta_{l},Cos#theta_{k})    [q^{2} bin %d Run II %d]",Q2Bin,RunEra));
   TH2D* HSideBand3DZY = (TH2D*) HSideBand3D->Project3D("zy");HSideBand3DZY->SetTitle(Form("2D Projection (#varphi,Cos#theta_{k})    [q^{2} bin %d Run II %d]",Q2Bin,RunEra));
   TH2D* HSideBand3DZX = (TH2D*) HSideBand3D->Project3D("zx");HSideBand3DZX->SetTitle(Form("2D Projection (#varphi,Cos#theta_{l})    [q^{2} bin %d Run II %d]",Q2Bin,RunEra));


//   TH1D*HSideBand3DY_1=(TH1D*) HSideBand3DZY->ProjectionY("HSideBand3DY_1",1,HSideBand3DZY->GetNbinsZ());
//   TH1D*HSideBand3DZ_1=(TH1D*) HSideBand3DZY->ProjectionX("HSideBand3DZ_1",1,HSideBand3DZY->GetNbinsY());

//////////////////////////////////
//
// SideBand Plots
//
//////////////////////////////////
  gStyle->SetTitleBorderSize(0);
  gStyle->SetTitleFontSize(0.05) ;
  c6->cd(1);
//  HSideBand3DX->Draw("E1");
//  HSideBandX->Draw("E1");
  TH1D* HSBFuncX_ratio = (TH1D*)HSBFuncX->Clone();
  HSBFuncX_ratio->Rebin(NFact);
  HSBFuncX_ratio->SetLineWidth(2.);
  HSBFuncX_ratio->SetLineColor(kRed);
//  HSBFuncX->Scale(HSideBandX->Integral()/HSBFuncX->Integral()*NFact);
//  HSBFuncX->Scale(HSideBand3DX->Integral()/HSBFuncX->Integral());
//  HSBFuncX->Draw("same,HIST C");
  HSideBand3DX->SetMinimum(SetMinProj);
  RatioDataModel3DX = new TRatioPlot(HSideBand3DX,HSBFuncX_ratio);
  RatioDataModel3DX->SetGraphDrawOpt("L");
  RatioDataModel3DX->SetSeparationMargin(0.0);
  RatioDataModel3DX->SetH1DrawOpt("E1");
  RatioDataModel3DX->SetH2DrawOpt("HIST C");
  RatioDataModel3DX->Draw();
  RatioDataModel3DX->GetUpperPad()->cd();;
  HSBFuncX->Scale(NFact);
  HSBFuncX->Draw("same HIST C");
  RatioDataModel3DX->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioDataModel3DX->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  c6->Update();
  c6->cd(2);
//  HSideBandY->Draw("E1");
//  HSideBand3DY->Draw("E1");
  TH1D* HSBFuncY_ratio = (TH1D*)HSBFuncY->Clone();
  HSBFuncY_ratio->Rebin(NFact);
  HSBFuncY_ratio->SetLineWidth(2.);
  HSBFuncY_ratio->SetLineColor(kRed);
//  HSBFuncY->Scale(HSideBand3DY->Integral()/HSBFuncY->Integral());
//  HSBFuncY->Scale(HSideBandY->Integral()/HSBFuncY->Integral()*NFact);
//  HSBFuncY->Draw("same,HIST C");
  HSideBand3DY->SetMinimum(SetMinProj);
  RatioDataModel3DY = new TRatioPlot(HSideBand3DY,HSBFuncY_ratio);
  RatioDataModel3DY->SetGraphDrawOpt("L");
  RatioDataModel3DY->SetSeparationMargin(0.0);
  RatioDataModel3DY->SetH1DrawOpt("E1");
  RatioDataModel3DY->SetH2DrawOpt("HIST C");
  RatioDataModel3DY->Draw();
  RatioDataModel3DY->GetUpperPad()->cd();;
  HSBFuncY->Scale(NFact);
  HSBFuncY->Draw("same HIST C");
  RatioDataModel3DY->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioDataModel3DY->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  TLegend* leg_SBFunc = new TLegend(0.40,0.67,0.90,0.90);
  leg_SBFunc->SetTextSize(0.025) ;
  leg_SBFunc->SetTextAlign(13);
  leg_SBFunc->SetBorderSize(0.);
  leg_SBFunc->SetFillStyle(0);
//  leg_SBFunc->SetEntrySeparation(0.0001);
//  leg_SBFunc->SetNColumns(1);
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "%5.3f<#Chi^{2}_{/NDOF}<%5.3f", AdaptChi2/NDegreeofFreedomAdaptN,AdaptChi2/NDegreeofFreedomAdaptR),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "reduced NDOF=%d [NDOF=%d] ", NDegreeofFreedomAdaptR,NDegreeofFreedomAdaptN),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "%6.5f<P-Value Min<%6.5f ", PValueModelMin,PValueModelMax),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "Events x  adapt. bin=%d ",MinContAdaptBin),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "%5.3f<#Chi^{2}_{/NDOF}<%5.3f", AdaptPowerDivergence/NDegreeofFreedomAdaptN,AdaptPowerDivergence/NDegreeofFreedomAdaptR),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "reduced NDOF=%d [NDOF=%d] ", NDegreeofFreedomAdaptR,NDegreeofFreedomAdaptN),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "%6.5f<P-Value<%6.5f ", PValuePDModelMin,PValuePDModelMax),"");
//   leg_SBFunc->AddEntry(HSBFuncY ,Form( "num of Events x [adaptive] bin=%d ",MinContAdaptBin),"");
  leg_SBFunc->AddEntry(RatioDataModel3DY ,Form( "%5.3f<#Chi^{2}_{/NDOF}<%5.3f", AdaptPowerDivergence/NDegreeofFreedomAdaptN,AdaptPowerDivergence/NDegreeofFreedomAdaptR),"");
  leg_SBFunc->AddEntry(RatioDataModel3DY ,Form( "reduced NDOF=%d [NDOF=%d] ", NDegreeofFreedomAdaptR,NDegreeofFreedomAdaptN),"");
  leg_SBFunc->AddEntry(RatioDataModel3DY ,Form( "%6.5f<P-Value<%6.5f ", PValuePDModelMin,PValuePDModelMax),"");
  leg_SBFunc->AddEntry(RatioDataModel3DY ,Form( "num of Events x [adaptive] bin=%d ",MinContAdaptBin),"");
  leg_SBFunc->Draw();
  c6->Update();
  c6->cd(3);
//  HSideBandZ->Draw("E1");
//  HSideBand3DZ->Draw("E1");
  TH1D* HSBFuncZ_ratio = (TH1D*)HSBFuncZ->Clone();
  HSBFuncZ_ratio->Rebin(NFact);
  HSBFuncZ_ratio->SetLineWidth(2.);
  HSBFuncZ_ratio->SetLineColor(kRed);
//  HSBFuncZ->Scale(HSideBandZ->Integral()/HSBFuncZ->Integral()*NFact);
//  HSBFuncZ->Scale(HSideBand3DZ->Integral()/HSBFuncZ->Integral());
//  HSBFuncZ->Draw("same,HIST C");
  HSideBand3DZ->SetMinimum(SetMinProj);
  RatioDataModel3DZ = new TRatioPlot(HSideBand3DZ,HSBFuncZ_ratio);
  RatioDataModel3DZ->SetGraphDrawOpt("L");
  RatioDataModel3DZ->SetSeparationMargin(0.0);
  RatioDataModel3DZ->SetH1DrawOpt("E1");
  RatioDataModel3DZ->SetH2DrawOpt("HIST C");
  RatioDataModel3DZ->Draw();
  RatioDataModel3DZ->GetUpperPad()->cd();;
  HSBFuncZ->Scale(NFact);
  HSBFuncZ->Draw("same HIST C");
  RatioDataModel3DZ->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioDataModel3DZ->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  RatioDataModel3DZ->GetUpperRefXaxis()->SetLimits(-TMath::Pi(),TMath::Pi());
  RatioDataModel3DZ->GetLowerRefGraph()->GetXaxis()->SetLimits(-TMath::Pi(),TMath::Pi());
  c6->Update();

  c6->cd(4); 
  TH1D* pdfHxMassQ2Set = (TH1D*)pdfHxMassQ2->Clone();
  HxMassQ2   ->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  HxMassQ2SB ->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  pdfHxMassQ2->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  sigHxMassQ2->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  bkgHxMassQ2->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  pdfHxMassQ2Set->GetXaxis()->SetRangeUser(XLeftSet,XRightSet);
//   HxMassQ2   ->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//   HxMassQ2SB ->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//   pdfHxMassQ2->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//   sigHxMassQ2->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//   bkgHxMassQ2->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//  pdfHxMassQ2Set->GetXaxis()->SetLimits(HistMassL1,HistMassL2);
//  sigHxMassQ2->SetLineStyle(kDashed);
//  bkgHxMassQ2->SetLineStyle(kDashed);
  pdfHxMassQ2Set->SetFillColor(kBlue);
  pdfHxMassQ2Set->SetFillStyle(3013);
  pdfHxMassQ2Set->SetLineWidth(1.0);
//  
  TLegend* leg_signSB = new TLegend(0.25,0.75,0.90,0.90);
  leg_signSB->SetTextSize(0.025) ;
  leg_signSB->SetTextAlign(11);
  leg_signSB->SetBorderSize(0.);
  leg_signSB->SetFillStyle(0);
//  leg_signSB->AddEntry(HxMassQ2 ,Form( "in red:"),"");
  leg_signSB->AddEntry(HxMassQ2 ,Form( "#color[2]{sideband for q^{2} bin %d [%2.1f<q^{2}<%2.1f]}", Q2Bin, Q2Min,Q2Max),"");
  if(SigmaProbSign==0){
   leg_signSB->AddEntry(HxMassQ2 ,Form( "sb entries=%4.0f [-%s#sigma,-%s#sigma]&[%s#sigma,%s#sigma]",\\
    SBEntries,FMTNSigma1L,FMTNSigma2L,FMTNSigma1R,FMTNSigma2R),"");
   leg_signSB->AddEntry(HxMassQ2 ,Form( "bckg entries=%4.0f [-2#sigma,2#sigma]",NBckgInt2Sigma),"");
  }else{
   leg_signSB->AddEntry(HxMassQ2 ,Form( "sb entries=%4.0f",SBEntries),"");
   leg_signSB->AddEntry(HxMassQ2 ,Form( "bckg entries=%4.0f [95.5%% of signal]",NBckgInt2Sigma),"");
  }  
//  leg_signSB->AddEntry(HxMassQ2 ,Form( "bckg entries=%4.0f [-2#sigma,2#sigma]",NBckgInt2Sigma),"");
//  leg_signSB->AddEntry(HxMassQ2 ,Form( "#chi^{2}_{/NDOF}=%5.2f NDOF=%d", AdaptChi2/NDegreeofFreedomAdapt,NDegreeofFreedomAdapt),"pel");
// //  leg_signSB->SetHeader(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max));
//   leg_signSB->AddEntry(HxMassQ2 ,Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max),"pel");
//   leg_signSB->AddEntry(HxMassQ2SB ,Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max),"pel");
//  gStyle->SetTitleFontSize(0.09) ;
  HxMassQ2->SetMaximum(1.5 * HxMassQ2->GetMaximum());
  HxMassQ2->SetTitle(Form("B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum [q^{2} bin %d run %d]",Q2Bin,RunEra));
//  HxMassQ2->SetTitle(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum and sideband for q^{2} bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max));
  HxMassQ2->SetLineColor(kBlue);
  HxMassQ2->DrawCopy("E1,9");
  HxMassQ2->GetXaxis()->SetTitle("Mass (GeV/c^{2})");
//  HxMassQ2SB->SetTitleSize(20);
  HxMassQ2SB->SetLineColor(kOrange);
  HxMassQ2SB->SetTitle(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum  and sideband for q^{2} bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max));
  HxMassQ2SB->SetFillColor(kRed);
  HxMassQ2SB->DrawCopy("SAME,B,9");
  pdfHxMassQ2->DrawCopy("same,HIST C,9");
  sigHxMassQ2->DrawCopy("same,HIST C,9");
  bkgHxMassQ2->DrawCopy("same,HIST C,9");
  pdfHxMassQ2Set->DrawCopy("same,HIST C,9");
  leg_signSB->Draw();
  c6->Update();
  
//////////////////////////////////
//////////////////////////////////
  gStyle->SetHistLineStyle(1);
//  
// separate plot for ProjX 
//  
  cprojX->cd();
//  HSBFuncX->Scale(NFact);
//   HSideBand3DX->GetXaxis()->SetTitle("Cos#theta_{l}");
//   HSideBand3DX->Draw("E1");
// //  HSideBandX->Draw("E1");
  HSBFuncX->SetLineColor(kRed);
  HSBFuncX->SetLineWidth(2.);
// //  HSBFuncX->Scale(HSideBand3DX->Integral()/HSBFuncX->Integral()*NFact);
// //  HSBFuncX->Scale(HSideBand3DX->Integral()/HSBFuncX->Integral());
//  HSBFuncX->Draw("same,HIST C");
  TRatioPlot* RatioProjModel3DX = new TRatioPlot(HSideBand3DX,HSBFuncX_ratio);
  RatioProjModel3DX->SetGraphDrawOpt("L");
  RatioProjModel3DX->SetSeparationMargin(0.0);
  RatioProjModel3DX->SetH1DrawOpt("E1");
  RatioProjModel3DX->SetH2DrawOpt("HIST C");
  RatioProjModel3DX->Draw();
  RatioProjModel3DX->GetUpperPad()->cd();;
  HSBFuncX->Draw("same HIST C");
  RatioProjModel3DX->GetLowerRefYaxis()->SetTitle("Ratio");
  RatioProjModel3DX->GetUpperRefYaxis()->SetTitle(Form("Events/%2.2f",(XMaxCosThetaL-XMinCosThetaL)/xCosLHBin));  
  RatioProjModel3DX->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioProjModel3DX->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  cprojX->Update();

//  
// separate plot for ProjY 
//  
  cprojY->cd();
// //  HSBFuncY->Scale(NFact);
// //  HSideBandY->Draw("E1");
//   HSideBand3DY->GetXaxis()->SetTitle("Cos#theta_{k}");
//   HSideBand3DY->Draw("E1");
  HSBFuncY->SetLineColor(kRed);
  HSBFuncY->SetLineWidth(2.);
//   HSBFuncY->Draw("same,HIST C");
  TRatioPlot* RatioProjModel3DY = new TRatioPlot(HSideBand3DY,HSBFuncY_ratio);
  RatioProjModel3DY->SetGraphDrawOpt("L");
  RatioProjModel3DY->SetSeparationMargin(0.0);
  RatioProjModel3DY->SetH1DrawOpt("E1");
  RatioProjModel3DY->SetH2DrawOpt("HIST C");
  RatioProjModel3DY->Draw();
  RatioProjModel3DY->GetUpperPad()->cd();;
  HSBFuncY->Draw("same HIST C");
  RatioProjModel3DY->GetLowerRefYaxis()->SetTitle("Ratio");
  RatioProjModel3DY->GetUpperRefYaxis()->SetTitle(Form("Events/%2.2f",(XMaxCosThetaK-XMinCosThetaK)/xCosKHBin));  
  RatioProjModel3DY->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioProjModel3DY->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  TLegend* leg_SBProj = new TLegend(0.42,0.60,0.90,0.85);
  leg_SBProj->SetTextSize(0.025) ;
  leg_SBProj->SetTextAlign(13);
  leg_SBProj->SetBorderSize(0.);
  leg_SBProj->SetFillStyle(0);
  leg_SBProj->AddEntry(HSideBand3DY ,Form( "Data Projection"),"lep");
  leg_SBProj->AddEntry(HSBFuncY_ratio ,Form( "Fit  Projection"),"l");
  leg_SBProj->AddEntry(RatioProjModel3DY ,Form( "#chi^{2}_{/NDOF} #approx %3.2f", AdaptPowerDivergence/NDegreeofFreedomAdaptR),"");
  leg_SBProj->AddEntry(RatioProjModel3DY ,Form( "p-value #approx %3.2f ", PValuePDModelMin),"");
  leg_SBProj->Draw();
//   RatioProjModel3DY->GetUpperPad()->cd();;
//   HSBFuncY->Draw("same HIST C");
  cprojY->Update();
// separate plot for ProjZ 
  cprojZ->cd();
// //  HSBFuncZ->Scale(NFact);
// //  HSideBandZ->Draw("E1");
//   HSideBand3DZ->GetXaxis()->SetTitle("#varphi");
//   HSideBand3DZ->Draw("E1");
// //  HSBFuncZ->Scale(HSideBand3DZ->Integral()/HSBFuncZ->Integral()*NFact);
// //  HSBFuncZ->Scale(HSideBand3DZ->Integral()/HSBFuncZ->Integral());

  HSBFuncZ->SetLineColor(kRed);
  HSBFuncZ->SetLineWidth(2.);
  HSBFuncZ->Draw("same,HIST C");
  HSideBand3DZ->SetMinimum(SetMinProj);
  TRatioPlot* RatioProjModel3DZ = new TRatioPlot(HSideBand3DZ,HSBFuncZ_ratio);
  RatioProjModel3DZ->SetGraphDrawOpt("L");
  RatioProjModel3DZ->SetSeparationMargin(0.0);
  RatioProjModel3DZ->SetH1DrawOpt("E1");
  RatioProjModel3DZ->SetH2DrawOpt("HIST C");
  RatioProjModel3DZ->Draw();
  RatioProjModel3DZ->GetUpperPad()->cd();
  HSBFuncZ->SetTitle("");
  HSBFuncZ->Draw("same HIST C");
  RatioProjModel3DZ->GetLowerRefYaxis()->SetTitle("Ratio");
  RatioProjModel3DZ->GetUpperRefYaxis()->SetTitle(Form("Events/%2.2f",(XMaxPhi-XMinPhi)/xPhiHBin));  
  RatioProjModel3DZ->GetLowerRefGraph()->SetMinimum(SetMinRatio);
  RatioProjModel3DZ->GetLowerRefGraph()->SetMaximum(SetMaxRatio);
  RatioProjModel3DZ->GetUpperRefXaxis()->SetLimits(-TMath::Pi(),TMath::Pi());
  RatioProjModel3DZ->GetLowerRefGraph()->GetXaxis()->SetLimits(-TMath::Pi(),TMath::Pi());
  cprojZ->Update();

  gStyle->SetPalette(57);
  cxy->cd();
  gPad->SetTheta(40.);
  gPad->SetPhi(40.);
  //HSideBand3DXY->Rebin2D(2,2);
  HSideBand3DXY->Smooth(2);
  HSideBand3DXY->SetFillColor(38);
//  HSideBand3DXY->Draw("SURF2 0");
  HSideBand3DXY->GetXaxis()->SetLabelSize(0);
  HSideBand3DXY->GetYaxis()->SetLabelSize(0);
  HSideBand3DXY->GetZaxis()->SetLabelSize(0);
  HSideBand3DXY->GetXaxis()->SetTitle("Cos#theta_{l}");
  HSideBand3DXY->GetYaxis()->SetTitle("Cos#theta_{k}");
  HSideBand3DXY->Draw("LEGO2 0 fbbb");
//  auto cutg = new TCutG("cutg",5);
//  cxy->cd(2);
//  HSBFuncXY->Scale(HSideBand3DXY->Integral()/HSBFuncXY->Integral()*NFact);
  HSBFuncXY->SetLineColor(kRed);
  HSBFuncXY->GetXaxis()->SetTitle("Cos#theta_{l}");
  HSBFuncXY->GetYaxis()->SetTitle("Cos#theta_{k}");
  HSBFuncXY->Draw("SURF A same fbbb");
  
//
  cyz->cd();
  gPad->SetTheta(40.);
  gPad->SetPhi(40.);
//  HSideBand3DZY->Rebin2D(2,2);
  HSideBand3DZY->Smooth(2);
  HSideBand3DZY->GetXaxis()->SetLabelSize(0);
  HSideBand3DZY->GetYaxis()->SetLabelSize(0);
  HSideBand3DZY->GetZaxis()->SetLabelSize(0);
  HSideBand3DZY->GetYaxis()->SetTitle("#varphi");
  HSideBand3DZY->GetXaxis()->SetTitle("Cos#theta_{k}");
  HSideBand3DZY->Draw("LEGO2 0 fbbb");
//  cyz->cd(2);
//  HSBFuncZY->Scale(HSideBand3DZY->Integral()/HSBFuncZY->Integral()*NFact);
  HSBFuncZY->SetLineColor(kRed);
  HSBFuncZY->GetYaxis()->SetTitle("#varphi");
  HSBFuncZY->GetXaxis()->SetTitle("Cos#theta_{k}");
  HSBFuncZY->Draw("SURF A same fbbb");
//
  cxz->cd();
  gPad->SetTheta(40.);
  gPad->SetPhi(40.);
  //HSideBand3DZX->Rebin2D(2,2);
  HSideBand3DZX->Smooth(2);
  HSideBand3DZX->GetXaxis()->SetLabelSize(0);
  HSideBand3DZX->GetYaxis()->SetLabelSize(0);
  HSideBand3DZX->GetZaxis()->SetLabelSize(0);
  HSideBand3DZX->GetYaxis()->SetTitle("#varphi");
  HSideBand3DZX->GetXaxis()->SetTitle("Cos#theta_{l}");
  HSideBand3DZX->Draw("LEGO2 0 fbbb");
//  cxz->cd(2);
//  HSBFuncZX->Scale(HSideBand3DZX->Integral()/HSBFuncZX->Integral()*NFact);
  HSBFuncZX->SetLineColor(kRed);
  HSBFuncZX->GetYaxis()->SetTitle("#varphi");
  HSBFuncZX->GetXaxis()->SetTitle("Cos#theta_{l}");
  HSBFuncZX->Draw("SURF A same fbbb");

  
  
 
  
  
//
// Mass Spectrum
//
//   GooFit::Variable mean  ("mean"  ,5.2762,XStepMinuit, 5., 5.5);
//   GooFit::Variable sigma1("sigma1",0.0139,XStepMinuit, 0., 1.);
//   GooFit::Variable sigma2("sigma2",0.0228,XStepMinuit, 0., 1.);
//   GooFit::Variable sigma3("sigma3",0.0601,XStepMinuit, 0., 1.);
//////////////////////////////////
/// MASS ONLY                  ///
//////////////////////////////////
  cmass->cd();
  gStyle->SetTitleBorderSize(0);
  TH1D* HxMassQ2Clone  =(TH1D*)  HxMassQ2->Clone("");
  TH1D* HxMassQ2SBClone=(TH1D*)  HxMassQ2SB->Clone("");
  TH1D* pdfHxMassQ2Clone=(TH1D*) pdfHxMassQ2->Clone("");
  TH1D* sigHxMassQ2Clone=(TH1D*) sigHxMassQ2->Clone("");
  TH1D* bkgHxMassQ2Clone=(TH1D*) bkgHxMassQ2->Clone("");
  HxMassQ2Clone   ->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  HxMassQ2SBClone ->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  pdfHxMassQ2Clone->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  sigHxMassQ2Clone->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  bkgHxMassQ2Clone->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  HxMassQ2Clone   ->GetXaxis()->SetRangeUser(HistMassL1,HistMassL2);
  HxMassQ2SBClone ->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  pdfHxMassQ2Clone->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  sigHxMassQ2Clone->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
  bkgHxMassQ2Clone->GetXaxis()->SetRangeUser(XMinSBL,XMaxSBR);
//  pdfHxMassQ2->SetLineStyle(2);
//  sigHxMassQ2Clone->SetLineStyle(kDashed);
//  bkgHxMassQ2Clone->SetLineStyle(kDashed);
  TLegend* leg_signSBMass = new TLegend(0.53,0.65,0.65,0.88);
  leg_signSBMass->SetTextSize(0.022) ;
  leg_signSBMass->SetTextAlign(13);
  leg_signSBMass->SetBorderSize(0.);
  leg_signSBMass->SetFillStyle(0);
  leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "#color[2]{q^{2} bin  %d [%2.1f<q^{2}<%2.1f GeV^{2}/c^{4}]:}", Q2Bin, Q2Min,Q2Max),"");
  leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "Mass [data]"),"lep");
  leg_signSBMass->AddEntry(pdfHxMassQ2Clone ,Form( "fit model"),"l");
  leg_signSBMass->AddEntry(sigHxMassQ2Clone ,Form( "signal model"),"l");
  leg_signSBMass->AddEntry(bkgHxMassQ2Clone ,Form( "bckg model"),"l");
  if(SigmaProbSign==0){
   leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "sb events = %4.0f [-%s#sigma,-%1s#sigma]&[%s#sigma,%s#sigma]", \\
   SBEntries,NSigma1L,NSigma2L,NSigma1R,NSigma2R,FMTNSigma1L,FMTNSigma2L,FMTNSigma1R,FMTNSigma2R),"");
   leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "bckg events =%4.0f [-2#sigma,2#sigma]",NBckgInt2Sigma),"");
  }else{
   leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "sb entries=%4.0f",SBEntries),"");
   leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "bckg events =%4.0f [95.5%% of signal]",NBckgInt2Sigma),"");
  } 
//  leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "#chi^{2}_{/NDOF}=%5.2f NDOF=%d", AdaptChi2/NDegreeofFreedomAdapt,NDegreeofFreedomAdapt),"pel");
// //  leg_signSBMass->SetHeader(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2CloneBin, Q2CloneMin,Q2CloneMax));
//   leg_signSBMass->AddEntry(HxMassQ2Clone ,Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2CloneBin, Q2CloneMin,Q2CloneMax),"pel");
//   leg_signSBMass->AddEntry(HxMassQ2CloneSB ,Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2CloneBin, Q2CloneMin,Q2CloneMax),"pel");
//  gStyle->SetTitleFontSize(0.09) ;
  HxMassQ2Clone->SetTitle(Form("B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass - q^{2} bin %d Run II %d [%2.1f<q^{2}<%2.1f Gev^{2}/c^{4}]",Q2Bin, RunEra, Q2Min,Q2Max));
//  HxMassQ2Clone->SetTitle(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum and sideband for q^{2} bin %d [%2.1f<q^{2}<%2.1f] ", Q2CloneBin, Q2CloneMin,Q2CloneMax));
  HxMassQ2Clone->SetLineColor(kBlue);
  HxMassQ2Clone->GetXaxis()->SetTitle("Mass  (GeV/c^{2})");
  HxMassQ2Clone->GetYaxis()->SetTitleOffset(1.4);
  HxMassQ2Clone->GetYaxis()->SetTitle(Form("Events/(%4.4f GeV/c^{2})",(XMaxSign-XMinSign)/xMassHBin2));
  HxMassQ2Clone->DrawClone("E1");
//  HxMassQ2CloneSB->SetTitleSize(20);
  HxMassQ2SBClone->SetLineColor(kRed);
  HxMassQ2SBClone->SetTitle(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum and sideband for q^{2} bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max));
  HxMassQ2SBClone->SetFillColor(kRed);
  HxMassQ2SBClone->DrawClone("SAME,B");
  pdfHxMassQ2Clone->DrawClone("same,HIST C");
//  SetHistLineStyle(2);
  sigHxMassQ2Clone->DrawClone("same,HIST C");
  bkgHxMassQ2Clone->DrawClone("same,HIST C");
  leg_signSBMass->Draw();
  cmass->Update();
  gStyle->SetHistLineStyle(1);
//////////////////////////////////
//////////////////////////////////
//////////////////////////////////

  OutFile->cd();
  covMatrix->Write();

  c1->cd();
//     TLegend* leg_sign = new TLegend(0.30,0.70,0.90,0.90);
//     leg_sign->SetTextSize(0.025) ;
//     leg_sign->SetTextAlign(31);
//     leg_sign->SetBorderSize(0.);
//     leg_sign->SetFillStyle(0);
//     leg_sign->SetHeader(Form( "B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum for bin %d [%2.1f<q^{2}<%2.1f] ", Q2Bin, Q2Min,Q2Max));
//     leg_sign->AddEntry(HxMass ,"","");
//     if(signalYield->getError()!=0){
//       leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f  #pm %5.0f",signalYield->getValue(),signalYield->getError()),"");
//     }else{
//       leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f Fixed",signalYield->getValue()),"");
//     }
//     if(bckgYield->getError()!=0){
//       leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =    %5.0f  #pm  %5.0f",bckgYield->getValue(),bckgYield->getError()),"");
//     }else{
//       leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =    %5.0f  Fixed",bckgYield->getValue()),"");
//     }
//     
//     if(mean.getError()!=0){
//      leg_sign->AddEntry(&HxMass ,Form( "M_{B^{0}} =   %5.5f  #pm %5.5f",mean.getValue(),mean.getError()),"");
//     }else{
//      leg_sign->AddEntry(&HxMass ,Form( "M_{B^{0}} =   %5.5f Fixed",mean.getValue()),"");
//      }
//     if(sigma1.getError()!=0){
//      leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{1}_{B^{0}} =   %5.5f  #pm %5.5f",sigma1.getValue(),sigma1.getError()),"");
//     }else{
//      leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{1}_{B^{0}} =   %5.5f Fixed",sigma1.getValue()),"");
//     }
//     if(sigma2.getError()!=0){
//      leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f  #pm %5.5f",sigma2.getValue(),sigma2.getError()),"");
//     }else{
//      leg_sign->AddEntry(&HxMass ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f Fixed",sigma2.getValue()),"");
//    }
  gStyle->SetTitleBorderSize(0);
//  gStyle->SetTitleFontSize(0.08) ;
  HxMass->SetFillStyle(0);
  HxMass->SetTitle("B_{0} #rightarrow K^{*0}#mu^{+}#mu^{-} Mass Spectrum");
  HxMass->GetXaxis()->SetTitle("Mass (GeV/c^{2})");
  HxMass->SetMarkerStyle(8);
  HxMass->SetMarkerSize(MarkerSizeSet);
  HxMass->Draw("E1");
//   HxMassQ2->SetLineColor(kBlue);
//   HxMassQ2->Draw("same"); 

  HxMass->Write();
  HxMassQ2SB->Write();
  HxMassQ2->Write();
  HxMassVsCosL->Write();
  HxMassVsCosK->Write();
  HxMassVsPhi->Write();
  HxReco->Write();
  HSideBand3D->Write();
  HSBFunc->Write();
  HSBFuncX->Write();
  HSBFuncY->Write();
  HSBFuncZ->Write();
  HSBFuncXY->Write();
  HSBFuncZY->Write();
  HSBFuncZX->Write();
  HSideBand3DX->Write();
  HSideBand3DY->Write();
  HSideBand3DZ->Write();
//  HSideBand3DY_1->Write();
//  HSideBand3DZ_1->Write();
  HSideBand3DXY->Write();
  HSideBand3DZY->Write();
  HSideBand3DZX->Write();
  pdfHxMassQ2->Write();
  sigHxMassQ2->Write();
  bkgHxMassQ2->Write();
  
//   TH3D *H3Div = (TH3D *)HxReco->Clone(); 
//   H3Div->SetName("H3RecoDivGen");
//   H3Div->Divide(HxGene);
//
   cmass->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitSB3DMass,PNGNameFitSB3DMass));
   cmass->Print(PNGNameFitSB3DMass);
   c1->Write();
   c6->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitSB3D,PNGNameFitSB3D));
   c6->Print(PNGNameFitSB3D);
   gSystem->Exec(Form("mv %s %s.tmp",PDFNameFitSB3D,PDFNameFitSB3D));
   c6->Print(PDFNameFitSB3D);
   cxy->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameProjXYHist,PNGNameProjXYHist));
   cxy->Print(PNGNameProjXYHist);
   cyz->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameProjZYHist,PNGNameProjZYHist));
   cyz->Print(PNGNameProjZYHist);
   cxz->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameProjZXHist,PNGNameProjZXHist));
   cxz->Print(PNGNameProjZXHist);
//   
   cprojX->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitSB3DProjX,PNGNameFitSB3DProjX));
   cprojX->Print(PNGNameFitSB3DProjX);
//   
   cprojY->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitSB3DProjY,PNGNameFitSB3DProjY));
   cprojY->Print(PNGNameFitSB3DProjY);
//
   cprojZ->Write();
   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitSB3DProjZ,PNGNameFitSB3DProjZ));
   cprojZ->Print(PNGNameFitSB3DProjZ);
//
   h2polxyContC->Write();
   h2polxzContC->Write();
   h2polxyDensC->Write();
   h2polxzDensC->Write();
   ca->Write();
   csignstudy->Write();
   
//   c7->Write();
//   c8->Write();
//   gSystem->Exec(Form("mv %s %s.tmp",PDFNameMass,PDFNameMass));
//   c1->Print(PDFNameMass);
//   sprintf(testo,"mv %s %s.tmp",PDFNameFitEffi3D,PDFNameFitEffi3D);
//   gSystem->Exec(Form("mv %s %s.tmp",PDFNameFitEffi3D,PDFNameFitEffi3D));
//   c6->Print(PDFNameFitEffi3D);
//   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitEffi3D,PNGNameFitEffi3D));
//   c6->Print(PNGNameFitEffi3D);
//   gSystem->Exec(Form("mv %s %s.tmp",PDFNameFitClosure,PDFNameFitClosure));
//   c8->Print(PDFNameFitClosure);
//   gSystem->Exec(Form("mv %s %s.tmp",PNGNameFitClosure,PNGNameFitClosure));
//   c8->Print(PNGNameFitClosure);
  OutFile->Close();
  
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;
//=================================================================================  
//=================================================================================  
// START write par+norm txt files  
//=================================================================================  
//=================================================================================  
//   coeffy =0.0;
//   std::fstream *parListNormOutput =  new std::fstream(ListParNorm,ios::out);
//   std::fstream *parPlotNormOutput =  new std::fstream(ListPloNorm,ios::out);
//   if(parListNormOutput->is_open() && parPlotNormOutput->is_open() ){
//    std::cout<<"Open: "<<ListParNorm<<std::endl ;
//    std::cout<<"Open: "<<ListPloNorm<<std::endl ;
//    for(int i=0;i<numParameters;++i) {
//     if(fabs(coeffPoly[i].getValue())>fabs(coeffPoly[i].getError())){
//      coeffy=  coeffPoly[i].getValue();
//     }else{
//      coeffy=  0.0;
//     } 
//     *parListNormOutput <<std::scientific << std::setprecision(20)<< coeffy<<std::endl;
//     *parPlotNormOutput <<std::scientific << std::setprecision(20)<< coeffPoly[i].getValue()<<std::endl;
//    }
//
//    xCosL_x.setNumBins(SETNumBinsX);
//    xCosK_y.setNumBins(SETNumBinsY);
//    xPhiK_z.setNumBins(SETNumBinsZ);
//    cout<<"CosL integration NBins ="<<xCosL_x<<endl;
//    cout<<"CosK integration NBins ="<<xCosK_y<<endl;
//    cout<<"Phi  integration NBins ="<<xPhiK_z<<endl;
// //  
// // cout<<"hipHostFree(Norm)  = "<<(hipHostFree(model))<<endl;
//    cout<<"hipHostFree(modelPlot)  = "<<(hipHostFree(modelPlot))<<endl;
// //   cout<<"hipHostFree(model)      = "<<(hipHostFree(model))<<endl;
//    totalParams=0;
//    parListNormOutput->close();
//    parPlotNormOutput->close();
//    std::cout<<"Close: "<<ListParNorm<<std::endl ;
//    std::cout<<"Close: "<<ListPloNorm<<std::endl ;
//   }else{
//    if(!parListNormOutput->is_open()) std::cout<<"Error: can not open "<<ListParNorm<<std::endl ;
//    if(!parPlotNormOutput->is_open()) std::cout<<"Error: can not open "<<ListPloNorm<<std::endl ;
//    exit(1);
//   }
//=================================================================================  
//=================================================================================  
// END write par+norm txt files  
//=================================================================================  
//=================================================================================  
  std::cout<<"===================================================================="<<endl;
  std::cout<<"======== REMIND:  	NFact   ="<<NFact   <<"		=============="<<std::endl;
  std::cout<<"======== REMIND:  	NFactGen="<<NFactGen<<"		=============="<<std::endl;
  std::cout<<"====================================================================\n\n"<<endl;
  if(xCoeffIndex>0){
   std::cout<<"===================================================================="<<endl;
   std::cout<<Form("WARNING: Normalization could be fixed better if p(%d)=1",xCoeffIndex)<<std::endl ;
   std::cout<<"===================================================================="<<endl;
  }

  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;
}

//============================================================================================================================================================
//============================================================================================================================================================
//============================================================================================================================================================
//
//  CreateInputHistoFile();
//
//============================================================================================================================================================
//============================================================================================================================================================
//============================================================================================================================================================
void CreateInputHistoFile(){   

  if(Folded) printf("****************************** WARNING: Folded ******************************\n");
  
  TFile*OutFileNtupla = TFile::Open(OutFileNameInputHisto,"RECREATE");
  RecoB0TreeOut = new TTree(OutputRecoB0TreeName,OutputRecoB0TreeName) ;
  RecoB0TreeOut -> SetAutoSave(500000000);
  TCanvas* c2 = new TCanvas("c2","Fit Mass Spectrum",200,10,900,780);
  TCanvas* c3 = new TCanvas("c3","Reco Histograms",200,10,900,780);
  c3->Divide(2,2);  

  int nfile=0;
  TChain* RecoB0Tree = new TChain();  
  nfile = RecoB0Tree->Add(Form("%s/%s/%s",RecoDir,InputFileNameRecoB0,InputRecoB0TreeName));
  if( nfile==0 ||  !RecoB0Tree->GetFile() ){
    cout<<"Error:  no Reco files found!!!\n"<<endl;
    exit(1);
  }else{
    printf("Try to open %s/%s/%s\n",RecoDir,InputFileNameRecoB0,InputRecoB0TreeName);
    cout<<"Opening "<<nfile <<" Reco files found!!!\n"<<endl;
  }  
  if(!RecoB0Tree ){
    cout<<"TTree Reco Data: "<< InputRecoB0TreeName <<" not found!!!\n"<<endl;
    exit(1);
  }else{
    cout<<"TTree Reco Data: "<< InputRecoB0TreeName <<" OK FOUND!!!\n"<<endl;
  }  
  


  printf("(Mass Window     : xB0Mass>%8f && xB0Mass<%8f \n",XMinSign,XMaxSign);
//  printf("(SB Mass Windows : xB0Mass>%8f && xB0Mass<%8f || xB0Mass>%8f && xB0Mass<%8f)\n",XMinSBL,XMaxSBL,XMinSBR,XMaxSBR);




//   TH3D* HxReco = new   TH3D( "HxReco"    , "B^{0} Reco correct tagged",  xCosLHBin, XMinCosThetaL, XMaxCosThetaL,
//  									 xCosKHBin, XMinCosThetaK, XMaxCosThetaK,
// 									 xPhiHBin , XMinPhi, XMaxPhi );




	
						
//  GooFit::Observable* xcTau  = new GooFit::Observable("xcTau",XMin, XMax); 
  xMass.setNumBins( xMassHBin );      // set  step for integrals
  std::cout<<"xMass.getNumBins() = "<<xMass.getNumBins()<<std::endl;
  std::cout<<"Hist Mass Bin = "<<xMassHBin<<std::endl;
 

//======================================================================
//======================================================================
//======================================================================
//
//			      RECONSTRUCTED EVENTS
//
//======================================================================
//======================================================================
//======================================================================
  double  tagged_mass	 ;
  double  cos_theta_l	 ;
  double  cos_theta_k	 ;
  double  phi_kst_mumu   ;
  double  mumuMass	 ;
  double  mumuMassE	 ;
  double  recQ2 	 ;
  double  tagB0          ;
  double  mmk1		 ;
  double  mmk2		 ;
  double  bMass		 ;
  double  bBarMass	 ;
  double  dR_mum_trkm    ;
  double  dR_mup_trkp    ;
//   bool    passB0Psi_lmnr ;
//   bool    passB0Psi_jpsi ;
//   bool    passB0Psi_psip ;
  int    passB0Psi_lmnr ;
  int    passB0Psi_jpsi ;
  int    passB0Psi_psip ;
  int    xcut=       -99;
//  bool    xcut= false;
  bool    passB0Psi      ;
//  
  bool    XCut= false    ;
  double  kaonPt	=0;
  double  pionPt	=0;
  double  mmpiMass	=0;
  double  mmkMass	=0;
  double  wt_mass	=0;
  double  wt_kstarmass  =0;
//
  RecoB0Tree->SetBranchAddress("tagged_mass"   ,&tagged_mass);
  RecoB0Tree->SetBranchAddress("cos_theta_l"   ,&cos_theta_l);
  RecoB0Tree->SetBranchAddress("cos_theta_k"   ,&cos_theta_k);
  RecoB0Tree->SetBranchAddress("phi_kst_mumu"  ,&phi_kst_mumu);
  RecoB0Tree->SetBranchAddress("mumuMass"      ,&mumuMass);
  RecoB0Tree->SetBranchAddress("mumuMassE"     ,&mumuMassE);
  RecoB0Tree->SetBranchAddress("tagB0"         ,&tagB0);
  RecoB0Tree->SetBranchAddress("mmk1"          ,&mmk1);
  RecoB0Tree->SetBranchAddress("mmk2"          ,&mmk2);
  RecoB0Tree->SetBranchAddress("bMass"         ,&bMass);
  RecoB0Tree->SetBranchAddress("bBarMass"      ,&bBarMass);
  RecoB0Tree->SetBranchAddress("dR_mum_trkm",  &dR_mum_trkm);
  RecoB0Tree->SetBranchAddress("dR_mup_trkp",  &dR_mup_trkp);
  RecoB0Tree->SetBranchAddress("passB0Psi_lmnr",&passB0Psi_lmnr);
  RecoB0Tree->SetBranchAddress("passB0Psi_jpsi",&passB0Psi_jpsi);
  RecoB0Tree->SetBranchAddress("passB0Psi_psip",&passB0Psi_psip);
  RecoB0Tree->SetBranchAddress("xcut"          ,&xcut);
  RecoB0Tree->SetBranchAddress("kaonPt"	       ,&kaonPt        );
  RecoB0Tree->SetBranchAddress("pionPt"	       ,&pionPt        );
  RecoB0Tree->SetBranchAddress("mmpiMass"      ,&mmpiMass      );
  RecoB0Tree->SetBranchAddress("mmkMass"       ,&mmkMass       );
  RecoB0Tree->SetBranchAddress("wt_mass"       ,&wt_mass       );
  RecoB0Tree->SetBranchAddress("wt_kstarmass"  ,&wt_kstarmass  );
  
  RecoB0TreeOut->Branch("cos_theta_l"   ,&cos_theta_l    ,   "cos_theta_l/D"   );
  RecoB0TreeOut->Branch("cos_theta_k"   ,&cos_theta_k    ,   "cos_theta_k/D"   );
  RecoB0TreeOut->Branch("phi_kst_mumu"  ,&phi_kst_mumu   ,   "phi_kst_mumu/D"  );
  RecoB0TreeOut->Branch("tagged_mass"   ,&tagged_mass    ,   "tagged_mass/D"   );
  RecoB0TreeOut->Branch("mumuMass"      ,&mumuMass       ,   "mumuMass/D"      );
  RecoB0TreeOut->Branch("mumuMassE"     ,&mumuMassE      ,   "mumuMassE/D"     );
  RecoB0TreeOut->Branch("mmk1"          ,&mmk1           ,   "mmk1/D"          );
  RecoB0TreeOut->Branch("mmk2"          ,&mmk2           ,   "mmk2/D"     );
  RecoB0TreeOut->Branch("kaonPt"      	,&kaonPt 	 ,   "kaonPt/D"        );
  RecoB0TreeOut->Branch("pionPt"      	,&pionPt 	 ,   "pionPt/D"        );
  RecoB0TreeOut->Branch("mmpiMass"    	,&mmpiMass       ,   "mmpiMass/D"      );
  RecoB0TreeOut->Branch("mmkMass"     	,&mmkMass        ,   "mmkMass/D"       );
  RecoB0TreeOut->Branch("wt_mass"     	,&wt_mass        ,   "wt_mass/D"       );
  RecoB0TreeOut->Branch("wt_kstarmass"	,&wt_kstarmass   ,   "wt_kstarmass/D"  );
//s
  double x0Cut=-0.4;
  double y0Cut= 0.3;
  double x1Cut= 0.6;
  double y1Cut=-0.1;
//  
  double x_0Cut=3;
  double y_0Cut=3.8;
  double x_1Cut=3.6;
  double y_1Cut=4.8;
  
  double CutX1=3.2;
  double CutX2=3.6;
  double CutY1=4.7;
  double CutY2=4.9;
//
//  double nSigma_psiRej =3.;
//
  std::vector<GooFit::Observable> dataMassVec;
  dataMassVec.push_back(xMass);
  UnbinnedDataSet* dataMass = new GooFit::UnbinnedDataSet(dataMassVec);
//  
//  
  int nentries = (int)RecoB0Tree->GetEntries();
  
   for (Int_t i=0;i<nentries;i++) { 
    RecoB0Tree->GetEntry(i);
    recQ2         =  mumuMass*mumuMass  ;
    if(Q2Bin==4){
     XCut=(((BpMass-wt_mass)-y0Cut)/(y1Cut-y0Cut))<(((wt_kstarmass-KstarMass)-x0Cut)/(x1Cut-x0Cut))&&kaonPt>pionPt&&(wt_kstarmass-KstarMass)>0
     	 &&(mmpiMass>CutX1&&mmpiMass<CutX2)&&(mmkMass>CutY1&&mmkMass<CutY2)&&((mmkMass-y_0Cut)/(y_1Cut-y_0Cut))>((mmpiMass-x_0Cut)/(x_1Cut-x_0Cut));
     if (XCut&&xcut!=1 || !XCut&&xcut!=0){
       std::cout<<"==>> Error checking XCut!!!! <<=="<<xMassHBin<<std::endl;
     }
//     passB0Psi =(passB0Psi_jpsi==1)&&(!xcut);
     passB0Psi =(passB0Psi_jpsi==1)&&(xcut==0);
    }else if(Q2Bin==6){  
     passB0Psi = (passB0Psi_psip==1);
    }else{  
     passB0Psi = (passB0Psi_lmnr==1);
    }  
//    double theBMass = tagged_mass;
//    double theBMass = tagB0*bMass+(1.-tagB0)*bBarMass;
//    double deltaB0M = theBMass-B0Mass;
//    double deltaJpsiM = mumuMass - JPsiMass;
//    double deltaPsiPM = mumuMass - PsiPMass;
//     if (!(recQ2> 8.68&&recQ2<10.09)&&
//         !(recQ2>12.86&&recQ2<14.18)){
	
//      if     ( fabs(mumuMass - JPsiMass) > nSigma_psiRej*mumuMassE && fabs(mumuMass - PsiPMass) > nSigma_psiRej*mumuMassE &&  \
//            (( mumuMass < JPsiMass && !( fabs(deltaB0M - deltaJpsiM) < 0.18 || fabs(deltaB0M - deltaPsiPM) < 0.0) ) || \
//             ( mumuMass > PsiPMass  && !( fabs(deltaB0M - deltaJpsiM) < 0.0  || fabs(deltaB0M - deltaPsiPM) < 0.09) ) || \
//             ( mumuMass > JPsiMass && mumuMass < PsiPMass && !( fabs(deltaB0M - deltaJpsiM) < 0.08 || fabs(deltaB0M - deltaPsiPM) < 0.08 )))){
//       if (! ( (mmk1 > 5.158 && mmk1 < 5.398) || (mmk2 > 5.158 && mmk2 < 5.398)) ){
//uffaaa

//        if ( fabs(mumuMass - JPsiMass) > nSigma_psiRej*mumuMassE && fabs(mumuMass - PsiPMass) > nSigma_psiRej*mumuMassE &&  \
//               (( mumuMass < JPsiMass && !( fabs(deltaB0M - deltaJpsiM) < 0.18 || fabs(deltaB0M - deltaPsiPM) < 0.0) ) || \
//                ( mumuMass > PsiPMass  && !( fabs(deltaB0M - deltaJpsiM) < 0.0  || fabs(deltaB0M - deltaPsiPM) < 0.08) ) || \
//                ( mumuMass > JPsiMass && mumuMass < PsiPMass && !( fabs(deltaB0M - deltaJpsiM) < 0.08 || fabs(deltaB0M - deltaPsiPM) < 0.09
//    	    )))){

//  if ( fabs(mumuMass - JPsiMass) > nSigma_psiRej*mumuMassE && fabs(mumuMass - PsiPMass) > nSigma_psiRej*mumuMassE &&  \
//         (( mumuMass < JPsiMass && !( fabs(deltaB0M - deltaJpsiM) < 0.16 || fabs(deltaB0M - deltaPsiPM) < 0.06) ) || \
//          ( mumuMass > PsiPMass && !( fabs(deltaB0M - deltaJpsiM) < 0.06 || fabs(deltaB0M - deltaPsiPM) < 0.03) ) || \
//          ( mumuMass > JPsiMass && mumuMass < PsiPMass && !( fabs(deltaB0M - deltaJpsiM) < 0.06 || fabs(deltaB0M - deltaPsiPM) < 0.06
//	    )))){
       if(passB0Psi){
        if(tagged_mass>=XMinSign&&tagged_mass<=XMaxSign){
//	if(dR_mum_trkm>0.0001&&
//	   dR_mup_trkp>0.0001
//	  ){
//
         xMass.setValue(tagged_mass);
         HxMass  ->Fill(tagged_mass);
         dataMass->addEvent();
         if(recQ2>Q2Min&&recQ2<Q2Max){
     	  if(cos_theta_l>=XMinCosThetaL&&cos_theta_l<=XMaxCosThetaL&&cos_theta_k>=XMinCosThetaK&&cos_theta_k<=XMaxCosThetaK){
     	       RecoB0TreeOut->Fill();
//
     	  }
     	 }
        }
       }
//      } 
//     }
//     }
    }
//   }
  
   
   
//  char TXT[200];
//  sprintf(TXT,"Mass Reco   Entries = %7f",HxMassQ2->GetEntries());
  cout<<"***********************************"<<endl;
  cout<<"***** RECONSTRUCTED EVENTS ********\n"<<endl;
  cout<<"***********************************\n"<<endl;
  cout<<"RecoB0Tree   Entries      = "<<nentries<<endl;
//  cout<<TXT<<endl;
  cout<<"\n***********************************"<<endl;
  cout<<"***********************************"<<endl;
//
  B0Sigma = FitMassSpectrum(dataMass, c2, HxMass,pdfHxMass,sigHxMass,bkgHxMass,5);
//  
  
  
 


  
//   TH1D* HxRecoX  = (TH1D*) HxReco->ProjectionX("HxRecoX",1,HxReco->GetNbinsY(),1,HxReco->GetNbinsZ());HxRecoX->SetTitle("HxReco Projection Cos#theta_{L}");
//   TH1D* HxRecoY  = (TH1D*) HxReco->ProjectionY("HxRecoY",1,HxReco->GetNbinsX(),1,HxReco->GetNbinsZ());HxRecoY->SetTitle("HxReco Projection Cos#theta_{K}");
//   TH1D* HxRecoZ  = (TH1D*) HxReco->ProjectionZ("HxRecoZ",1,HxReco->GetNbinsX(),1,HxReco->GetNbinsY());HxRecoZ->SetTitle("HxReco Projection #phi");
  
  OutFileNtupla->cd();
  
  c2->Write();
  gSystem->Exec(Form("mv %s %s.tmp",PNGNameMassHist,PNGNameMassHist));
//   HxReco->Write();
//   HxRecoX->Write();
//   HxRecoY->Write();
//   HxRecoZ->Write();
  HxMass->Write();
  pdfHxMass->Write();
  sigHxMass->Write();
  bkgHxMass->Write();
  HxMassQ2->Write();
  RecoB0TreeOut->Write();
  OutFileNtupla->Close();
  
//   sprintf(testo,"mv %s %s.tmp",PDFNameRecoHisto,PDFNameRecoHisto);
//   gSystem->Exec(testo);
//   c3->cd(1);HxRecoX->Draw();
//   c3->cd(2);HxRecoY->Draw();
//   c3->cd(3);HxRecoZ->Draw();
//   c3->Print(PDFNameRecoHisto);
  cout<<"**********************************************************************\n"<<endl;
  cout<<"save  HxReco  in "<<OutFileNameInputHisto<<"\n"<<endl;
  cout<<"**********************************************************************\n"<<endl;
}
//==========================================================================================
//
//       FitMassSpectrumRoofit
//
//==========================================================================================
double FitMassSpectrumRoofit(RooDataSet* data, TCanvas* c2, TH1D* masHist, TH1D* pdfHist, TH1D*sigHist, TH1D* bkgHist, int MaxDegreeBckg){
// //
//   if(MaxDegreeBckg<=0) {
//    cout<<"**********************************************************************\n"<<endl;
//    cout<<"Error!! MaxDegree <=0 in   FitMassSpectrumRoofit		       *\n"<<endl;
//    cout<<"**********************************************************************\n"<<endl;
//   }
// //
//     int Q2BinTMP=Q2Bin;
//      if(Q2Bin==6) {
//       Q2Bin=4;
//     }; 

    TFile* fitMassFile = new TFile( fitMassFileName, "READ" );
    if ( !fitMassFile || !fitMassFile ->IsOpen() ) {
      cout<<Form("File not found: %s\n",fitMassFileName)<<endl;
      exit(1);
    }
     RooWorkspace* w = (RooWorkspace*)fitMassFile->Get("w");
    if ( !w || w->IsZombie() ) {
     cout<<Form("Workspace not found in file:%s\n",fitMassFileName)<<endl;
     exit(1);
    } else {
     cout<<Form("Workspace Found!!! In file : %s\n",fitMassFileName)<<endl;
    }
//
    if(!(w->loadSnapshot(Form("reference_fit_RT_%d",Q2Bin)))){
      cout<<Form("Snapshot %s Workspace not found!!!\n",Form("reference_fit_RT_%d",Q2Bin))<<endl;
      exit(1);
    }else{
      w->loadSnapshot(Form("reference_fit_RT_%d",Q2Bin));
//      w->cd(Form("reference_fit_RT_%d",Q2Bin));
      cout<<Form("Snapshot %s Workspace found...\n",Form("reference_fit_RT_%d",Q2Bin))<<endl;
      cout<<"=========================================================================="<<endl;
      cout<<"=========================================================================="<<endl;
      cout<<Form(" DUMP WORKSPACE IN reference_fit_RT_%d",Q2Bin)<<endl;
      cout<<"=========================================================================="<<endl;
      cout<<Form("Snapshot %s Workspace found...\n",Form("reference_fit_RT_%d",Q2Bin))<<endl;
      w->Print("V");
      cout<<"=========================================================================="<<endl;
      cout<<"=========================================================================="<<endl;
    };

    w->ls();
    RooRealVar* tagged_mass= w->var("tagged_mass");
    double tagged_mass_rangeValMin=tagged_mass->getMin();
    double tagged_mass_rangeValMax=tagged_mass->getMax();
    std::cout<<Form("From workspace read tagged_mass in range [%f-%f]\n",tagged_mass_rangeValMin,tagged_mass_rangeValMax) <<std::endl;
    if( (tagged_mass_rangeValMin!=tagged_mass_rangeMin) || (tagged_mass_rangeValMax!=tagged_mass_rangeMax) ){
      std::cout<<Form("Warning! Force setting tagged_mass in range [%f-%f]\n",tagged_mass_rangeMin,tagged_mass_rangeMax) <<std::endl;
     tagged_mass->setRange(tagged_mass_rangeMin,tagged_mass_rangeMax);
    } 
//    tagged_mass->setRange(XMinSign,XMaxSign);
//    if(Q2Bin==0)  tagged_mass_rangeMin = 4.9;
    tagged_mass->setRange("full",tagged_mass_rangeMin,tagged_mass_rangeMax);
    NumFittedData = data->sumEntries(Form("tagged_mass>%f&&tagged_mass<%f",tagged_mass_rangeMin,tagged_mass_rangeMax));
    std::cout<<Form("==>> Warning!!!  Fitting number of events = %f", NumFittedData)<<std::endl;
   
//    tagged_mass->setRange(XMinFull,XMaxFull);
    
//     RooRealVar *nsig_ref    = w->var("Yield");
//     RooRealVar *nbkg_ref    = w->var("nbkg");
// //
//     if(!nsig_ref){
//       cout<<"Yield from ref. fit  not found!!!\n"<<endl;
//       exit(1);
//     }else{
//       cout<<Form("Yield from ref. fit found = %f\n",nsig_ref->getVal())<<endl;
//     }
//     if(!nbkg_ref){
//       cout<<"BackYield form ref not found!!!\n"<<endl;
//       exit(1);
//     }else{
//       cout<<Form("BackYield  found = %f\n",nbkg_ref->getVal())<<endl;
//     }
   
     
    RooRealVar *yield_fromMC_RT    = w->var(Form("nRT_%d",Q2Bin));
    RooRealVar *yield_fromMC_WT    = w->var(Form("nWT_%d",Q2Bin));
//
    if(!yield_fromMC_RT){
      cout<<"yield_fromMC_RT  not found!!!\n"<<endl;
      exit(1);
    }else{
      cout<<Form("yield_fromMC_RT  found = %f\n",yield_fromMC_RT->getVal())<<endl;
    }
    if(!yield_fromMC_WT){
      cout<<"yield_fromMC_WT  not found!!!\n"<<endl;
      exit(1);
    }else{
      cout<<Form("yield_fromMC_WT  found = %f\n",yield_fromMC_WT->getVal())<<endl;
    }
    RooRealVar * fraction = new RooRealVar("fraction","fraction",0.,1.);
    fraction->setVal(yield_fromMC_RT->getVal()/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal()));
    fraction->setError(sqrt(
    pow(yield_fromMC_RT->getError()*yield_fromMC_WT->getVal()/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal())/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal()),2)+
    pow(yield_fromMC_WT->getError()*yield_fromMC_RT->getVal()/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal())/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal()),2)));
    cout<<Form("fraction mistagged = %f +/- %f\n",fraction->getVal(),fraction->getError())<<endl;

    RooRealVar * fractionWT = new RooRealVar("fractionWT","fractionWT",0.,1.);
    fractionWT->setVal(yield_fromMC_WT->getVal()/(yield_fromMC_RT->getVal()+yield_fromMC_WT->getVal()));
    fractionWT->setError(fM_sigmas);
//    
    RooRealVar    * mean_rt      = 0;
    RooAbsPdf     *theRTgauss    = 0;
    RooRealVar	  * sigma_rt1	 = 0;
    RooRealVar	  * sigma_rt2	 = 0;
    RooGaussian   * c_sigma_rt1  = 0;
    RooGaussian   * c_sigma_rt2  = 0;
//    RooGaussian   * c_mean_rt	 = 0;
    RooGaussian   * c_f1rt	 = 0;
    RooProdPdf	  * c_RTgauss	 = 0;
    RooRealVar	  * alpha_rt1	 = 0;
    RooRealVar	  * alpha_rt2	 = 0;
    RooRealVar	  * n_rt1	 = 0;
    RooRealVar	  * n_rt2	 = 0;
    RooRealVar	  * f1rt	 = 0;
    RooArgSet	  * c_vars	 = 0;
    RooGaussian   * c_alpha_rt1  = 0;
    RooGaussian   * c_alpha_rt2  = 0;
    RooGaussian   * c_n_rt1	 = 0;
    RooGaussian   * c_n_rt2	 = 0;
    RooRealVar    * deltaPeakVar = 0;
    RooGaussian   * c_deltaPeaks = 0;
    RooArgList    * c_pdfs       = 0;
    RooArgList    * c_pdfs_rt    = 0;
    RooArgList    * c_pdfs_wt    = 0;
    RooRealVar	  *mean_wt	 = 0;
    RooRealVar	  *sigma_wt	 = 0;
    RooRealVar	  *alpha_wt1	 = 0;
    RooRealVar	  *alpha_wt2	 = 0;
    RooRealVar	  *n_wt1	 = 0;
    RooRealVar	  *n_wt2	 = 0;
    RooFormulaVar *mWT_data      = 0;




   if(w->pdf(Form("gauscb_RT_%d",Q2Bin))){ 
       theRTgauss   = w->pdf(Form("gauscb_RT_%d",Q2Bin));
       cout<<Form("FitMassSpectrumRoofit: gauscb_RT%d",Q2Bin)<<endl;
       mean_rt     = w->var(Form("mean_{RT}^{%d}",Q2Bin));
       sigma_rt1   = w->var(Form("#sigma_{RT1}^{%d}",Q2Bin));
       alpha_rt1   = w->var(Form("#alpha_{RT1}^{%d}",Q2Bin));
       n_rt1	   = w->var(Form("n_{RT1}^{%d}",Q2Bin));
       f1rt	   = w->var(Form("f^{RT%d}",Q2Bin));
       sigma_rt2   = w->var(Form("#sigma_{RT2}^{%d}",Q2Bin));
       c_sigma_rt1 = _constrainVar(sigma_rt1,w);
       c_alpha_rt1 = _constrainVar(alpha_rt1,w);
       c_sigma_rt2 = _constrainVar(sigma_rt2,w);
       c_f1rt	   = _constrainVar(f1rt,w);
       c_n_rt1     = _constrainVar(n_rt1,w);
       c_pdfs	 = new RooArgList(              *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_n_rt1,*c_f1rt);
       c_pdfs_rt = new RooArgList(*theRTgauss,  *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_n_rt1,*c_f1rt);
       c_vars	 = new RooArgSet(                 *sigma_rt1,   *sigma_rt2,   *alpha_rt1,   *n_rt1,*f1rt);
    } 
    
    
    
// RT Double Gaussian   
    if(w->pdf(Form("doublegaus_RT%d",Q2Bin))){
       cout<<Form("FitMassSpectrumRoofit: doublegaus_RT%d",Q2Bin)<<endl;
       theRTgauss   = w->pdf(Form("doublegaus_RT%d",Q2Bin));
       mean_rt     = w->var(Form("mean^{RT%d}",Q2Bin));
       sigma_rt1   = w->var(Form("#sigma_{1}^{RT%d}",Q2Bin));
       sigma_rt2   = w->var(Form("#sigma_{2}^{RT%d}",Q2Bin));
       f1rt	   = w->var(Form("f^{RT%d}",Q2Bin));
       c_sigma_rt1 = _constrainVar(sigma_rt1,w);
       c_sigma_rt2 = _constrainVar(sigma_rt2,w);
//       c_mean_rt   = _constrainVar(mean_rt,w);
       c_f1rt	   = _constrainVar(f1rt,w);
 
       ////// creating constraints for the RT component
//       c_RTgauss = new RooProdPdf("c_RTgauss" , "c_RTgauss" , RooArgList(*theRTgauss,*c_sigma_rt1,*c_sigma_rt2,*c_mean_rt,*c_f1rt  ) );
//        c_pdfs    = new RooArgList(            *c_sigma_rt1,*c_sigma_rt2,*c_mean_rt,*c_f1rt);
//        c_pdfs_rt = new RooArgList(*theRTgauss,*c_sigma_rt1,*c_sigma_rt2,*c_mean_rt,*c_f1rt);
//        c_vars    = new RooArgSet(               *sigma_rt1,  *sigma_rt2,  *mean_rt ,*f1rt);
//        c_pdfs    = new RooArgList(            *c_sigma_rt1,*c_sigma_rt2,*c_mean_rt,*c_f1rt);
//        c_pdfs_rt = new RooArgList(*theRTgauss,*c_sigma_rt1,*c_sigma_rt2,*c_mean_rt,*c_f1rt);
//        c_vars    = new RooArgSet(               *sigma_rt1,  *sigma_rt2,  *mean_rt,*f1rt);
       c_pdfs    = new RooArgList(            *c_sigma_rt1,*c_sigma_rt2,*c_f1rt);
       c_pdfs_rt = new RooArgList(*theRTgauss,*c_sigma_rt1,*c_sigma_rt2,*c_f1rt);
       c_vars    = new RooArgSet(               *sigma_rt1,  *sigma_rt2,*f1rt);
   }
// RT Double CB   
    if( w->pdf(Form("doublecb_RT%d",Q2Bin))){ 
       cout<<Form("FitMassSpectrumRoofit: doublecb_RT%d",Q2Bin)<<endl;
       theRTgauss  = w->pdf(Form("doublecb_RT%d",Q2Bin));
       mean_rt     = w->var(Form("mean_{RT}^{%d}",Q2Bin));
       sigma_rt1   = w->var(Form("#sigma_{RT1}^{%d}",Q2Bin));
       alpha_rt1   = w->var(Form("#alpha_{RT1}^{%d}",Q2Bin));
       alpha_rt2   = w->var(Form("#alpha_{RT2}^{%d}",Q2Bin));
       n_rt1	   = w->var(Form("n_{RT1}^{%d}",Q2Bin));
       n_rt2	   = w->var(Form("n_{RT2}^{%d}",Q2Bin));
       f1rt	   = w->var(Form("f^{RT%d}",Q2Bin));
//      deltaPeaks  = RooFormulaVar("deltaPeaks%s"%ibin, "@0 - @1", RooArgList(mean_rt, mean_wt));  
       c_sigma_rt1 = _constrainVar(sigma_rt1,w);
       c_alpha_rt1 = _constrainVar(alpha_rt1,w);
       c_alpha_rt2 = _constrainVar(alpha_rt2,w);
       c_n_rt1     = _constrainVar(n_rt1,w);
       c_n_rt2     = _constrainVar(n_rt2,w);
       if (Q2Bin < 4){ 
           cout<<Form("FitMassSpectrumRoofit: fast doublecb_RT%d",Q2Bin)<<endl;
           c_pdfs    = new RooArgList(            *c_sigma_rt1, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2);
           c_pdfs_rt = new RooArgList(*theRTgauss,*c_sigma_rt1, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2);
           c_vars    = new RooArgSet(               *sigma_rt1,   *alpha_rt1,   *alpha_rt2,   *n_rt1,   *n_rt2);
       }else{
           cout<<Form("FitMassSpectrumRoofit: old doublecb_RT%d",Q2Bin)<<endl;
           sigma_rt2     = w->var(Form("#sigma_{RT2}^{%d}",Q2Bin));
           c_f1rt	 = _constrainVar(f1rt,w);
           c_sigma_rt2   = _constrainVar(sigma_rt2, w);
           c_pdfs    = new RooArgList(  	    *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2, *c_f1rt);
           c_pdfs_rt = new RooArgList(*theRTgauss , *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2, *c_f1rt);
           c_vars    = new RooArgSet(		      *sigma_rt1,   *sigma_rt2,   *alpha_rt1,	*alpha_rt2,   *n_rt1,	*n_rt2, *c_f1rt);
//         c_pdfs    = new RooArgList(              *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2);
//         c_pdfs_rt = new RooArgList(*theRTgauss , *c_sigma_rt1, *c_sigma_rt2, *c_alpha_rt1, *c_alpha_rt2, *c_n_rt1, *c_n_rt2);
//         c_vars    = new RooArgSet(                 *sigma_rt1,   *sigma_rt2,   *alpha_rt1,   *alpha_rt2,   *n_rt1,   *n_rt2);
       }
    } 

    if(!theRTgauss)  {
     cout<<"pdf theRTgauss not found!!!\n"<<endl;
     exit(1);
    } else{
     cout<<Form("pdf %s  found...\n",theRTgauss->GetName())<<endl;
    }
//
    c_RTgauss = new RooProdPdf("c_RTgauss" , "c_RTgauss" , *c_pdfs_rt );
//
    if(!(w->loadSnapshot(Form("reference_fit_WT_%d",Q2Bin)))){
      cout<<Form("Snapshot %s Workspace not found!!!\n",Form("reference_fit_WT_%d",Q2Bin))<<endl;
      exit(1);
    }else{
      w->loadSnapshot(Form("reference_fit_WT_%d",Q2Bin));
//      w->cd(Form("reference_fit_WT_%d",Q2Bin));
      
      cout<<"=========================================================================="<<endl;
      cout<<"=========================================================================="<<endl;
      cout<<Form(" DUMP WORKSPACE IN reference_fit_WT_%d",Q2Bin)<<endl;
      cout<<"=========================================================================="<<endl;
      cout<<Form("Snapshot %s Workspace found...\n",Form("reference_fit_WT_%d",Q2Bin))<<endl;
      w->Print("V");
      cout<<"=========================================================================="<<endl;
      cout<<"=========================================================================="<<endl;
    };
    
//    
    mean_wt	= !(w->var(Form("mean^{WT%d}",Q2Bin)))?(w->var(Form("mean_{WT}^{%d}",Q2Bin))):(w->var(Form("mean^{WT%d}",Q2Bin)));
    sigma_wt	= !(w->var(Form("#sigma_{CB}^{WT%d}",Q2Bin)))?(w->var(Form("#sigma_{WT1}^{%d}",Q2Bin))):(w->var(Form("#sigma_{CB}^{WT%d}",Q2Bin)));
    alpha_wt1	= !(w->var(Form("#alpha_{1}^{WT%d}",Q2Bin)))?(w->var(Form("#alpha_{WT1}^{%d}",Q2Bin))):(w->var(Form("#alpha_{1}^{WT%d}",Q2Bin)));
    alpha_wt2	= !(w->var(Form("#alpha_{2}^{WT%d}",Q2Bin)))?(w->var(Form("#alpha_{WT2}^{%d}",Q2Bin))):(w->var(Form("#alpha_{2}^{WT%d}",Q2Bin)));
    n_wt1	= !(w->var(Form("n_{1}^{WT%d}",Q2Bin)))?(w->var(Form("n_{WT1}^{%d}",Q2Bin))):(w->var(Form("n_{1}^{WT%d}",Q2Bin)));
    n_wt2	= !(w->var(Form("n_{2}^{WT%d}",Q2Bin)))?(w->var(Form("n_{WT2}^{%d}",Q2Bin))):(w->var(Form("n_{2}^{WT%d}",Q2Bin)));
//    
//  workaround
//    
//     if(Q2Bin==7){
//       n_wt2->setMax(150);
//     }  
      
// 

//  => now theWTgauss is renamed theWTgaussMC (from the MC fit). 
//     RooAbsPdf *theWTgaussMC = w->pdf(Form("doublecb_%d",Q2Bin));
//     if(!theWTgaussMC)  {
//      cout<<"pdf theWTgaussMC not found!!!\n"<<endl;
//      exit(1);
//     } else{
//      cout<<Form("pdf %s  found...\n",theWTgaussMC->GetName())<<endl;
//     }
//    RooGaussian* c_mean_wt     = _constrainVar(mean_wt, w);
    RooGaussian* c_sigma_wt    = _constrainVar(sigma_wt, w);
    RooGaussian* c_alpha_wt1   = _constrainVar(alpha_wt1, w);
    RooGaussian* c_alpha_wt2   = _constrainVar(alpha_wt2, w);
    RooGaussian* c_n_wt1       = _constrainVar(n_wt1, w);
    RooGaussian* c_n_wt2       = _constrainVar(n_wt2, w);
    double deltaPeakValue=mean_rt->getVal()-mean_wt->getVal();
    double deltaPeakError=sqrt(mean_rt->getError()*mean_rt->getError()+mean_wt->getError()*mean_wt->getError());

    deltaPeakVar = new RooRealVar (Form("deltaPeakVar%d",Q2Bin), Form("deltaPeakVar%d",Q2Bin), deltaPeakValue, 0., 0.2) ;
    c_deltaPeaks = new RooGaussian(Form("deltaPeaks%d",Q2Bin) , "c_deltaPeaks", *deltaPeakVar, RooConst( deltaPeakValue ), RooConst(deltaPeakError )); // value to be checked
    mWT_data = new  RooFormulaVar(Form("mWT_data%d",Q2Bin), "@0 + @1", RooArgList(*mean_rt, *deltaPeakVar));

//  => new theWTgauss with deltaPeaks constraint 
    RooDoubleCBFast* theWTgauss = new RooDoubleCBFast(Form("doublecb_%d",Q2Bin),Form("doublecb_%d",Q2Bin), *tagged_mass, *mWT_data, *sigma_wt, *alpha_wt1, *n_wt1, *alpha_wt2, *n_wt2);	
    c_vars->add(*deltaPeakVar);
    c_pdfs ->add(*c_deltaPeaks);
        
//   if(w->obj( "deltaPeaks")) {
//    if(w->obj( Form("deltaPeaks%d",Q2Bin))) {
//     deltaPeaks  = (RooFormulaVar *)w->obj(Form("deltaPeaks%d",Q2Bin));
//     c_deltaPeaks = new RooGaussian(Form("deltaPeaks%d",Q2Bin) , "c_deltaPeaks", *deltaPeaks, RooConst( deltaPeaks->getVal() ), RooConst( 0.0005 )); // value to be checked
// //    c_vars->add(*deltaPeaks);
// //    c_pdfs ->add(*c_deltaPeaks);
//    }else if(w->pdf(Form("doublecb_RT%d",Q2Bin))){
//     cout<<Form("FitMassSpectrumRoofit: deltaPeaks NOT FOUND!!!")<<endl;
// //    exit(0);
//    }	    

 //     RooGaussian* c_sigma_wt2   = _constrainVar(sigma_wt2, w);
//     RooGaussian* c_f3          = _constrainVar(f3wt, w);
    

    ////// creating constraints for the WT component
//    RooProdPdf* c_WTgauss  = new RooProdPdf("c_WTgauss" , "c_WTgauss"\
//    ,RooArgList(*theWTgauss,*c_alpha_wt1,*c_n_wt1,*c_sigma_wt,*c_mean_wt,*c_alpha_wt2,*c_n_wt2  ) );     
//    RooRealVar  frt("F_{RT}"			  , "frt"   , fraction->getVal() , 0, 1);
//    RooGaussian c_frt("c_frt"           	   , "c_frt" , frt,  RooFit::RooConst(fraction->getVal()) , RooFit::RooConst(fraction->getError()) );
    RooRealVar  frt(Form("f_{M}^{%d}",Q2Bin)			  , Form("f_{M}^{%d}",Q2Bin)   , fractionWT->getVal() , 0, 1);
    RooGaussian c_frt("c_frt"           	   , "c_frt" , frt,  RooFit::RooConst(fractionWT->getVal()) , RooFit::RooConst(fractionWT->getError()) );
//    RooAddPdf	signalFunction("sumgaus"	  , "rt+wt" , RooArgList(*theRTgauss,*theWTgauss), RooArgList(frt));
//     RooGaussian c_frt("c_frt"           	   , "c_frt" , frt,  (fraction) , (*fraction_s) );
//    RooGaussian c_frt("c_frt"           	   , "c_frt" , frt,  RooFit::RooConst(0.87628877977) , RooFit::RooConst(0.000523435458235) );
 
    c_pdfs_wt = new RooArgList(*theWTgauss);
    c_pdfs_wt->add(*c_sigma_wt);
//    c_pdfs_wt->add(*c_mean_wt);
    c_pdfs_wt->add(*c_deltaPeaks);
    c_pdfs_wt->add(*c_alpha_wt1);
    c_pdfs_wt->add(*c_alpha_wt2);
    c_pdfs_wt->add(*c_n_wt1);
    c_pdfs_wt->add(*c_n_wt2);


    RooProdPdf* c_WTgauss  = new RooProdPdf("c_WTgauss" , "c_WTgauss",*c_pdfs_wt);
    RooAddPdf	signalFunction("sumgaus"	  , "rt+wt" , RooArgList(*c_WTgauss,*c_RTgauss), RooArgList(frt));
//    RooAddPdf	signalFunction("sumgaus"	  , "rt+wt" , RooArgList(*c_RTgauss,*c_WTgauss), RooArgList(frt));
    RooProdPdf  c_signalFunction("c_signalFunction", "c_signalFunction", RooArgList(signalFunction, c_frt))   ;  
    c_pdfs->add(*c_sigma_wt);
//    c_pdfs->add(*c_mean_wt);
    c_pdfs->add(*c_deltaPeaks);
    c_pdfs->add(*c_alpha_wt1);
    c_pdfs->add(*c_alpha_wt2);
    c_pdfs->add(*c_n_wt1);
    c_pdfs->add(*c_n_wt2);
    c_pdfs->add(c_frt);
//    c_pdfs->add(signalFunction);

    c_vars->add(*sigma_wt);
    c_vars->add(*deltaPeakVar);
//    c_vars->add(*mean_wt);
    c_vars->add(*alpha_wt1);
    c_vars->add(*alpha_wt2);
    c_vars->add(*n_wt1);
    c_vars->add(*n_wt2);
    c_vars->add(frt);

////// now create background parametrization
    RooRealVar*  slope= new RooRealVar("slope"      , "slope"           ,    -6.,   -10, 10);
//    RooRealVar*  slope= new RooRealVar("slope"      , "slope"           ,    0.5,   -10, 10);
//    RooExponential bkg_exp("bkg_exp"    , "exponential"     ,  *slope,   *tagged_mass  );
//     RooRealVar     pol_c1("p1"          , "coeff x^0 term"  ,    0.5,   -10, 10);
//     RooRealVar     pol_c2("p2"          , "coeff x^1 term"  ,    0.5,   -10, 10);
//     RooRealVar     pol_c3("p3"          , "coeff x^2 term"  ,    0.5,   -10, 10);
//     RooRealVar     pol_c4("p4"          , "coeff x^3 term"  ,    0.5,   -10, 10);
// 
//     RooChebychev   bkg_exp("bkg_exp"    , "2nd order pol"   ,  *tagged_mass, RooArgList(pol_c1,pol_c2,pol_c3,pol_c4));

    double pol_bmax =1.;
    if(Q2Bin==6) pol_bmax =1.;
    RooRealVar*     pol_b0= new RooRealVar("pol_b0"          , "b0"  ,    pol_bmax  );
    RooRealVar*     pol_b1= new RooRealVar("pol_b1"          , "b1"  ,    0.1,  0., pol_bmax);
    RooRealVar*     pol_b2= new RooRealVar("pol_b2"          , "b2"  ,    0.1,  0., pol_bmax);
    RooRealVar*     pol_b3= new RooRealVar("pol_b3"          , "b3"  ,    0.0 );
    RooRealVar*     pol_b4= new RooRealVar("pol_b4"          , "b4"  ,    0.1 , 0., pol_bmax);
    if(Q2Bin!=4){
//   if(Q2Bin!=4&&Q2Bin!=6){
     bkg_exp = new RooExponential("bkg_exp"    , "exponential"     ,  *slope,   *tagged_mass  );
   }else{
//     bkg_exp = new RooExponential("bkg_exp"    , "exponential"     ,  *slope,   *tagged_mass  );
    pol_b0->setConstant(kTRUE);
////    pol_b4->setConstant(kTRUE);
    pol_b3->setConstant(kTRUE);
    bkg_exp = new RooBernstein("bkg_exp"    , "bernstein pol"  ,  *tagged_mass, RooArgList(*pol_b0,*pol_b1,*pol_b2,*pol_b3,*pol_b4));
   }
    
    int NCPU=1;
    if(NFactGen>1) NCPU=10;
    double yieldIni = NFactGen*1000;
    double backgIni = NFactGen*1000;
    double yieldMin = 0.;
    double backgMin = 0.;
    double yieldMax = NFactGen*1000000.;
    double backgMax = NFactGen*1000000.;
   if(Q2Bin==4) {
       yieldIni = yieldSignal;
       backgIni = yieldBckg;
       yieldMin = 100000.;
       backgMin = 100000.;
       yieldMax = 3000000.;
       backgMax = 1000000.;
       NCPU=60;
    }   
   if(Q2Bin==6) {
       yieldIni = 100000;
       backgIni = 60000;
       yieldMin = 0.;
       backgMin = 0.;
       yieldMax = 1000000.;
       backgMax = 1000000.;
    }   
    RooRealVar     nsig("Yield"         , "signal frac"    ,   yieldIni,     yieldMin, yieldMax  );
    RooRealVar     nbkg("nbkg"          , "bkg fraction"   ,   backgIni,     backgMin, backgMax  );

//    RooRealVar     nsig("Yield"         , "signal frac"    ,    4000,     0,   1000000);
//    RooRealVar     nbkg("nbkg"          , "bkg fraction"   ,    1000,     0,   550000);
// 
//    RooProdPdf  c_signalFunction("c_signalFunction", "c_signalFunction", RooArgList(signalFunction, c_frt))   ;  
//    RooProdPdf c_signalFunction("c_signalFunction", "c_signalFunction", *c_pdfs);
    RooAddPdf fitFunction("fitfunction" , "fit function"  ,  RooArgList(c_signalFunction, *bkg_exp), RooArgList(nsig, nbkg));
//
//    RooAddPdf fitFunction("fitfunction" , "fit function"  ,  RooArgList(signalFunction, bkg_pol), RooArgList(nsig, nbkg));
//     tagged_mass->setRange("fullRedefined",XMinSBL,XMaxSBR);
    RooFitResult* r = fitFunction.fitTo(*data, 
    		       RooFit::Extended(kTRUE), 
    		       RooFit::NumCPU(NCPU),
    		       RooFit::Save(), 
    		       RooFit::Range("full"), 
    		       RooFit::Verbose(kFALSE),
    		       RooFit::Constrain(*c_vars)
    		      );
     		      
   r->Print();	
   std::cout<<Form("Warning! Number of Fitted Data = %f, Yield=%f, nbkg=%f Yield+nbkg=%f",NumFittedData,nsig.getVal(),nbkg.getVal(),nsig.getVal()+nbkg.getVal())<<std::endl;
//   Q2Bin=Q2BinTMP;	      
//
//
// save a clone of bkg_exp 
//

    if(Q2Bin!=4){
//    if(Q2Bin!=4&&Q2Bin!=6){
     bkg_mass_sb = (RooExponential*)bkg_exp->clone(Form("bkg_mass_sb_bin%d_%d",Q2Bin,RunEra) );
     bkg_mass_sb->setNormRange("full");
    }else{ 
     bkg_mass_sb = (RooBernstein*)  bkg_exp->clone(Form("bkg_mass_sb_bin%d_%d",Q2Bin,RunEra) );
     bkg_mass_sb->setNormRange("full");
    } 
//   RooAbsBinning binning = (tagged_mass->getBinning("full")) ;
//     data->getRange(*tagged_mass,tagged_mass_rangeMin,tagged_mass_rangeMax);
//    std::cout<<Form("%f<[fit mass range]<%f",tagged_mass_rangeMin,tagged_mass_rangeMax)<<std::endl; 
//    tagged_mass_rangeMin=tagged_mass->getMin("full");
//    tagged_mass_rangeMax=tagged_mass->getMax("full");
//    std::cout<<Form("%f<[fit mass range]<%f",tagged_mass_rangeMin,tagged_mass_rangeMax)<<std::endl; 

//      cout<<"----"<<endl;
//       exit(1);
//      data->Print("V");
//      c2->cd();
//      RooPlot* frame = tagged_mass->frame( );
//      data->plotOn(frame, Binning(35), MarkerSize(.7));
//     fitFunction.plotOn(frame);
//     drawPdfComponents(fitFunction, frame, ROOT.kAzure, RooFit.NormRange("full"), RooFit.Range("full"), isData = True);
// 
//     fitFunction.paramOn(frame,  RooFit.Layout(0.62,0.86,0.88));
//     frame.Draw();
//     niceFrame(frame, '')
//     frame. addObject(_writeFitStatus(r))
// 
//     if not args.year=='test':  writeCMS(frame, args.year, [ q2binning[ibin], q2binning[ibin+1] ])
//     frame.Draw()
 //    c2->Print("test.pdf");
 
     double B0SigmaTemp=0.;
     c2->cd();
     TLegend* leg_sign = new TLegend(0.30,0.48,0.90,0.90);
     leg_sign->SetTextSize(0.025) ;
     leg_sign->SetTextAlign(31);
     leg_sign->SetBorderSize(0.);
     leg_sign->SetFillStyle(0);
     leg_sign->SetHeader("B^{0} mass spectrum  Fit Projection");
     if(nsig.getError()!=0){
       leg_sign->AddEntry(masHist ,Form( "Yield_{Sign} =     %5.0f  #pm %5.0f",nsig.getVal(),nsig.getError()),"");
     }else{
       leg_sign->AddEntry(masHist ,Form( "Yield_{Sign} =     %5.0f Fixed",nsig.getVal()),"");
     }
     if(nbkg.getError()!=0){
       leg_sign->AddEntry(masHist ,Form( "Yield_{Bckg} =     %5.0f  #pm  %5.0f",nbkg.getVal(),nbkg.getError()),"");
     }else{
       leg_sign->AddEntry(masHist ,Form( "Yield_{Bckg} =     %5.0f  Fixed",nbkg.getVal()),"");
     }
     if(mean_rt->getError()!=0){
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}}[RT] =   %5.5f  #pm %5.5f",mean_rt->getVal(),mean_rt->getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}}[RT] =   %5.5f Fixed",mean_rt->getVal()),"");
      }
//     if(mean_wt==0){exit(0);};
     if(mean_wt->getError()!=0){
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}}[WT] =   %5.5f  #pm %5.5f",mean_wt->getVal(),mean_wt->getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}}[WT] =   %5.5f Fixed",mean_wt->getVal()),"");
      }
     if(sigma_rt1->getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "#sigma_{1}^{RT%d}#scale[0.6]{1}_{B^{0}} =   %5.5f  #pm %5.5f",Q2Bin,sigma_rt1->getVal(),sigma_rt1->getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "#sigma_{1}^{RT%d}#scale[0.6]{1}_{B^{0}} =   %5.5f Fixed",Q2Bin,sigma_rt1->getVal()),"");
     }
     if(sigma_rt2!=0){
      if(sigma_rt2->getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "#sigma_{2}^{RT%d}#scale[0.6]{2}_{B^{0}} =   %5.5f  #pm %5.5f",Q2Bin,sigma_rt2->getVal(),sigma_rt2->getError()),"");
//   	}else{
//    	 leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f Fixed",sigma2.getVal()),"");
      }
     } 
     if(sigma_wt->getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "#sigma^{WT%d}#scale[0.6]{1}_{B^{0}} =   %5.5f  #pm %5.5f",Q2Bin,sigma_wt->getVal(),sigma_wt->getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "#sigma^{WT%d}#scale[0.6]{1}_{B^{0}} =   %5.5f Fixed",Q2Bin,sigma_wt->getVal()),"");
     }
//      if(sigma_wt2){
//       if(sigma_wt2->getError()!=XStepMinuit){
//       leg_sign->AddEntry(masHist ,Form( "#sigma_{2}^{WT%d}#scale[0.6]{2}_{B^{0}} =   %5.5f  #pm %5.5f",Q2Bin,sigma_wt2->getVal(),sigma_wt2->getError()),"");
// //   	}else{
// //    	 leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f Fixed",sigma2.getVal()),"");
//       }
//     } 
     double min_CBGaus_rt=0;
     double max_CBGaus_rt=0;
//      float x1zoom=5.1;
//      float x2zoom=5.4;
     float x1zoom=XMinSign;
     float x2zoom=XMaxSign;
//      RooPlot *rframe = tagged_mass->frame(Title("Signal models RT"));
//      RooPlot *wframe = tagged_mass->frame(Title("Signal models WT"));
     RooAbsPdf *gaussRT_study = 0;
     RooGaussian *gaussRT_study1 = 0;
     RooGaussian *gaussRT_study2 = 0;
     TF1 * Func_theRTgauss    = 0;
     TF1 * Func_gaussRT_study = 0;
     TF1 * Clone_gaussRT_study = 0;
//     RooRealVar  f3("f3","f3",0.);
     if(w->pdf(Form("doublecb_RT%d",Q2Bin))){
      if(Q2Bin<4){
       min_CBGaus_rt =  mean_rt->getVal()-alpha_rt1->getVal()*sigma_rt1->getVal();
       max_CBGaus_rt =  mean_rt->getVal()+alpha_rt2->getVal()*sigma_rt1->getVal();
       gaussRT_study = new RooGaussian("gaussRT_study","gauss RT study"    ,*tagged_mass,*mean_rt,*sigma_rt1);
      }else{
//       sigma_rt2_pos = new RooRealVar("sigma_rt2_pos","|(sigma_rt2)|",fabs(sigma_rt2->getVal()));
       min_CBGaus_rt =  mean_rt->getVal()-fabs(alpha_rt1->getVal())*sigma_rt1->getVal();
       max_CBGaus_rt =  mean_rt->getVal()+fabs(alpha_rt2->getVal())*sigma_rt2->getVal();
       gaussRT_study1 = new RooGaussian("gaussRT_study1","gauss RT study1"    ,*tagged_mass,*mean_rt,*sigma_rt1);
       gaussRT_study2 = new RooGaussian("gaussRT_study2","gauss RT study2"    ,*tagged_mass,*mean_rt,*sigma_rt2);
       gaussRT_study = new RooAddPdf("gaussRT_study","gauss RT study"    ,RooArgList(*gaussRT_study1,*gaussRT_study2),RooArgList(*f1rt));
      } 
      std::cout<<Form("RT Double CB Gaus = %f<mass<%f",min_CBGaus_rt,max_CBGaus_rt)<<std::endl;
      csignstudy->cd(1);
      gPad->SetLeftMargin(0.15);
      Func_theRTgauss	 = theRTgauss	->asTF( RooArgList(*tagged_mass) );
      Func_gaussRT_study = gaussRT_study->asTF( RooArgList(*tagged_mass) );
      Func_theRTgauss->SetTitle("RT Model");
//      rframe->GetYaxis()->SetTitleOffset(1.4);
//      theRTgauss->plotOn(rframe,LineColor(kRed));
//      gaussRT_study->plotOn(rframe,LineColor(kBlue));
//      theRTgauss->plotOn(rframe, Range(min_CBGaus_rt,max_CBGaus_rt,kFALSE),FillColor(kRed),DrawOption("F"),FillStyle(3013),VLines());
      TLegend* leg_rt = new TLegend(0.70,0.70,0.90,0.90);
      leg_rt->SetTextSize(0.025) ;
      leg_rt->SetTextAlign(31);
      leg_rt->SetBorderSize(0.);
      leg_rt->SetFillStyle(0);
      leg_rt->AddEntry(Func_theRTgauss ,Form("#color[2]{Double CB Model}"),"");
      leg_rt->AddEntry(Func_gaussRT_study,Form("#color[4]{(2)Gaussian  Model}"),"");
//      rframe->addObject(leg_rt);
//      rframe->Draw();
      Func_theRTgauss->SetLineColor(kRed);
      Func_gaussRT_study->SetLineColor(kBlue);
      Func_theRTgauss->SetLineWidth(1.);
      Func_gaussRT_study->SetLineWidth(1.);
//     Func_gaussRT_study->SetLineStyle(kDashed);
      Func_theRTgauss->SetRange(x1zoom,x2zoom);
      Func_gaussRT_study->SetRange(x1zoom,x2zoom);
      Func_theRTgauss->Draw();
      Func_gaussRT_study->Draw("SAME");
      Clone_gaussRT_study =  (TF1*)Func_gaussRT_study->Clone();
      Clone_gaussRT_study->SetRange(min_CBGaus_rt,max_CBGaus_rt);
      Clone_gaussRT_study->SetFillColor(kBlue);
      Clone_gaussRT_study->SetFillStyle(3013);
      Clone_gaussRT_study->SetLineWidth(1.);
      Clone_gaussRT_study->Draw("SAME FC");
      leg_rt->Draw("SAME");
     } 
 //
     double min_CBGaus_wt =  mean_wt->getVal()-alpha_wt1->getVal()*sigma_wt->getVal();
     double max_CBGaus_wt =  mean_wt->getVal()+alpha_wt2->getVal()*sigma_wt->getVal();
     RooGaussian *gaussWT_study = new RooGaussian("gaussWT_study","gauss WT study"    ,*tagged_mass,*mean_wt,*sigma_wt);
     std::cout<<Form("WT Double CB Gaus = %f<mass<%f",min_CBGaus_wt,max_CBGaus_wt)<<std::endl;
     csignstudy->cd(2);
     gPad->SetLeftMargin(0.15);
//     wframe->GetYaxis()->SetTitleOffset(1.4);
//     theWTgauss->plotOn(wframe,LineColor(kRed));
     TF1 * Func_theWTgauss    = theWTgauss   ->asTF( RooArgList(*tagged_mass) );
     TF1 * Func_gaussWT_study = gaussWT_study->asTF( RooArgList(*tagged_mass) );
      Func_theWTgauss->SetTitle("WT Model");
//     RooAbsReal* IntegtheWTgauss    = theWTgauss->createIntegral(*tagged_mass,*tagged_mass,"full");
//     double scal = sigma_wt->getVal()*sqrt(2*TMath::Pi())*theWTgauss->getVal(RooArgList(*mean_wt));
//     cout<<IntegtheWTgauss->getVal()<<" "<<f->GetMaximum()<<endl;exit(0);
//     double scal = sigma_wt->getVal()*sqrt(2*TMath::Pi())*theWTgauss->getVal(RooArgList(*mean_wt));
//     double scal = IntegtheWTgauss->getVal()/gaussWT_study->getVal(RooArgList(*mean_wt));
//     gaussWT_study->plotOn(wframe,LineColor(kBlue),Normalization(1/IntegtheWTgauss->getVal()));
//     theWTgauss->plotOn(wframe, Range(min_CBGaus_wt,max_CBGaus_wt,kFALSE),FillColor(kRed),DrawOption("F"),FillStyle(3013),VLines());
     TLegend* leg_wt = new TLegend(0.70,0.70,0.90,0.90);
     leg_wt->SetTextSize(0.025) ;
     leg_wt->SetTextAlign(31);
     leg_wt->SetBorderSize(0.);
     leg_wt->SetFillStyle(0);
     leg_wt->AddEntry(Func_theWTgauss ,Form("#color[2]{Double CB Model}"),"");
     leg_wt->AddEntry(Func_gaussWT_study ,Form("#color[4]{Gaussian  Model}"),"");
//     leg_wt->AddEntry(theWTgauss ,Form("#color[2]{Double CB Model}"),"");
//     leg_wt->AddEntry(gaussWT_study ,Form("#color[4]{Gaussian  Model}"),"");
//     wframe->addObject(leg_wt);
//     wframe->Draw();
     Func_theWTgauss->SetLineColor(kRed);
     Func_gaussWT_study->SetLineColor(kBlue);
     Func_theWTgauss->SetLineWidth(1.0);
     Func_gaussWT_study->SetLineWidth(1.0);
//     Func_gaussWT_study->SetLineStyle(kDashed);
     Func_theWTgauss->SetRange(x1zoom,x2zoom);
     Func_gaussWT_study->SetRange(x1zoom,x2zoom);
     Func_theWTgauss->Draw();
     Func_gaussWT_study->Draw("SAME");
     TF1* Clone_gaussWT_study =  (TF1*)Func_gaussWT_study->Clone();
     Clone_gaussWT_study->SetRange(min_CBGaus_wt,max_CBGaus_wt);
     Clone_gaussWT_study->SetFillColor(kBlue);
     Clone_gaussWT_study->SetFillStyle(3013);
     Clone_gaussWT_study->SetLineWidth(1.0);
//     Clone_gaussWT_study->GetXaxis()->SetRangeUser(5.1,5.4);
     Clone_gaussWT_study->Draw("SAME FC");
     leg_wt->Draw("SAME");
     
     char PNGSignStudy[300]="";sprintf(PNGSignStudy,Form("signal-mass-study-Q2Bin-%d.png",Q2Bin));
     gSystem->Exec(Form("mv %s %s.tmp",PNGSignStudy,PNGSignStudy));
     csignstudy->Print(PNGSignStudy);
     
     double B0SigmaRT=0;
     double Sigma1RT =sigma_rt1->getVal();
     if(sigma_rt2!=0){
      double Sigma2RT =sigma_rt2->getVal();
      double WG1=f1rt->getVal();
      leg_sign->AddEntry(masHist ,Form( "f^{RT%d} =   %5.5f  #pm %5.5f",Q2Bin,f1rt->getVal(),f1rt->getError()),"");
      B0SigmaRT = sqrt(Sigma1RT*Sigma1RT*WG1+(1.-WG1)*Sigma2RT*Sigma2RT);
     }else{
      B0SigmaRT = Sigma1RT;
     }

      double B0sigma_wt =sigma_wt->getVal();
//      if(sigma_wt2){
//       double Sigma2WT =sigma_wt2->getVal();
//       double WG1=f3wt->getVal();
//       leg_sign->AddEntry(masHist ,Form( "f^{WT%d} =   %5.5f  #pm %5.5f",Q2Bin,f3wt->getVal(),f3wt->getError()),"");
//       B0sigma_wt = sqrt(Sigma1WT*Sigma1WT*WG1+(1.-WG1)*Sigma2WT*Sigma2WT);
//      }else{
//       B0sigma_wt = Sigma1WT;
//      }
     
     
//     B0SigmaTemp = sqrt(B0SigmaRT*B0SigmaRT*fraction->getVal()+(1.-fraction->getVal())*B0sigma_wt*B0sigma_wt);
     B0SigmaTemp = sqrt(B0SigmaRT*B0SigmaRT*(1.-fractionWT->getVal())+fractionWT->getVal()*B0sigma_wt*B0sigma_wt);
     

     std::cout<<Form("B0SigmaRT = %f B0sigma_wt = %f B0SigmaTot = %f",B0SigmaRT,B0sigma_wt,B0SigmaTemp) <<std::endl;

//     tagged_mass->setRange("SignLeft" ,mean_rt->getVal(),XMaxSign);
//     tagged_mass->setRange("SignRight",XMinSign,mean_rt->getVal());
//      tagged_mass->setRange(5.1,5.4);
//
     TCanvas* ccdf_signal = new TCanvas("ccdf_signal","cdf",200,10,1200,600);
//     ccdf_signal->Divide(2,2);
     TCanvas* ccdf_rt = new TCanvas("ccdf_rt","cdf RT",200,10,1200,600);
     ccdf_rt->Divide(2,2);
     
     TCanvas* ccdf_signal_zoom = new TCanvas("ccdf_signal_zoom","cdf",200,10,1200,600);
     ccdf_signal_zoom->Divide(2,2);
     TCanvas* ccdf_rt_zoom = new TCanvas("ccdf_rt_zoom","cdf RT",200,10,1200,600);
     ccdf_rt_zoom->Divide(2,2);
     TF1 *  Func_signal = c_signalFunction.asTF( RooArgList(*tagged_mass) );
     RooAbsPdf* signalCdf = (RooAbsPdf*)c_signalFunction.createCdf(*tagged_mass);
     TF1 *  Func_signalCdf= signalCdf->asTF( RooArgList(*tagged_mass) );
//     RooAbsPdf* theRTgaussCdf = (RooAbsPdf*)theRTgauss->createCdf(*tagged_mass);
//     TF1 *  Func_theRTgaussCdf= theRTgaussCdf->asTF( RooArgList(*tagged_mass) );
     std::cout<<"Estimate %% probability for Full/RT Signal:"<<std::endl;
     double integTails= 0;
//     double XLeftLim  = 0;
//     double XRightLim = 0;
//      double SigmaEstL = 0;
//      double SigmaEstR = 0;
     Func_signal->SetLineColor(kRed);
     Func_signal->SetLineWidth(1.0);
//      for(double iSigma=1;iSigma<5;iSigma++){
//      
//        integTails= TMath::Erfc(iSigma/sqrt(2));//  Erfc(x) = (2/sqrt(pi)) Integral(exp(-t^2))dt between x and infinity; t=x/sqrt(2)
//        XLeftLim  = Func_signalCdf->GetX(integTails/2.);
//        XRightLim = Func_signalCdf->GetX(1-integTails/2.);
//        SigmaEstL = (fabs(mean_rt->getVal()-XLeftLim))/iSigma;
//        SigmaEstR = (fabs(mean_rt->getVal()-XRightLim))/iSigma;
//        std::cout<<Form("Full signal %f%% [%iSigma gauss sigma] in the Range %f<mass<%f average sigmaL=%f sigmaR=%f",1-integTails,int(iSigma),XLeftLim,XRightLim,SigmaEstL,SigmaEstR)<<std::endl;
//        ccdf_signal->cd(iSigma);
//        Func_signal->SetRange(4.8,5.6);
//        Func_signal->SetMaximum(1.);
//        Func_signal->Draw();
//        TF1* Clone_signal =  (TF1*) Func_signal->Clone();
//        Clone_signal->SetRange(XLeftLim,XRightLim);
//        Clone_signal->SetFillColor(kBlue);
//        Clone_signal->SetFillStyle(3013);
//        Clone_signal->SetLineWidth(1.);
//        Clone_signal->Draw("SAME FC");
// //
//        ccdf_signal_zoom->cd(iSigma);
//        Func_signal->SetRange(4.8,5.6);
//        Func_signal->SetMaximum(0.2);
//        Func_signal->Draw();
//        Clone_signal =  (TF1*) Func_signal->Clone();
//        Clone_signal->SetRange(XLeftLim,XRightLim);
//        Clone_signal->Draw("SAME FC");
//        XLeftLim  = Func_theRTgaussCdf->GetX(integTails/2.);
//        XRightLim = Func_theRTgaussCdf->GetX(1-integTails/2.);
//        SigmaEstL = (fabs(mean_rt->getVal()-XLeftLim))/iSigma;
//        SigmaEstR = (fabs(mean_rt->getVal()-XRightLim))/iSigma;
//        std::cout<<Form("RT   signal %f%% [%iSigma gauss sigma] in the Range %f<mass<%f average sigmaL=%f sigmaR=%f",1-integTails,int(iSigma),XLeftLim,XRightLim,SigmaEstL,SigmaEstR)<<std::endl;
//
//        ccdf_rt->cd(iSigma);
//        Func_theRTgauss->SetRange(5.0,5.6);
//        Func_theRTgauss->SetMaximum(1.);
//        Func_theRTgauss->Draw();
//        TF1* Clone_theRTgauss =  (TF1*) Func_theRTgauss->Clone();
//        Clone_theRTgauss->SetRange(XLeftLim,XRightLim);
//        Clone_theRTgauss->SetFillColor(kBlue);
//        Clone_theRTgauss->SetFillStyle(3013);
//        Clone_theRTgauss->SetLineWidth(1.);
//        Clone_theRTgauss->Draw("SAME FC");
//        ccdf_rt->Update();
//        //
//        ccdf_rt_zoom->cd(iSigma);
//        Func_theRTgauss->SetRange(5.0,5.6);
//        Func_theRTgauss->SetMaximum(0.3);
//        Func_theRTgauss->Draw();
//        Clone_theRTgauss =  (TF1*) Func_theRTgauss->Clone();
//        Clone_theRTgauss->SetRange(XLeftLim,XRightLim);
//        Clone_theRTgauss->Draw("SAME FC");
//        ccdf_rt_zoom->Update();
//     }
//      ccdf_signal  ->Print(Form("cdf_signal_Q2Bin_%d.png",Q2Bin));
//      ccdf_rt->Print(Form("cdf_rt_Q2Bin_%d.png",Q2Bin));
//      ccdf_signal_zoom  ->Print(Form("cdf_signal_zoom_Q2Bin_%d.png",Q2Bin));
//      ccdf_rt_zoom->Print(Form("cdf_rt_zoom_Q2Bin_%d.png",Q2Bin));
//     tagged_mass->setRange("testRange",XLeftLim,XRightLim);
//     RooAbsReal*  testRTRange= theRTgauss->createIntegral(*tagged_mass,NormSet(*tagged_mass),Range("testRange"));
//     std::cout<<Form(" %f%% in the Range %f<mass<%f",testRTRange->getVal(),XLeftLim,XRightLim)<<std::endl;
//      TCanvas* ccdf = new TCanvas("ccdf","cdf",200,10,750,800);
//      ccdf->cd();
//      Func_signalCdf->Draw();
//      ccdf->Print("cdf.png");
     


//      tagged_mass->setRange("3sigmaintegral",mean_rt->getVal()-3*B0SigmaTemp,mean_rt->getVal()+3*B0SigmaTemp);
//      RooAbsReal* BckgInt3Sigma = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"3sigmaintegral");
//      RooAbsReal* SignInt3Sigma = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"3sigmaintegral");
//      NBckgInt3Sigma = BckgInt3Sigma->getVal()*nbkg.getVal();
//      NSignInt3Sigma = SignInt3Sigma->getVal()*nsig.getVal();
//      std::cout<<Form("Bckg event +/- 3 sigma_w from mean RT = %f",NBckgInt3Sigma) <<std::endl;
//      std::cout<<Form("Sign event +/- 3 sigma_w from mean RT = %f",SignInt3Sigma) <<std::endl;
//      std::cout<<Form("Sign %% in +/- 3 sigma_w from mean RT = %f",SignInt3Sigma->getVal()) <<std::endl;
     integTails= TMath::Erfc(sqrt(2));
//      XLeftLim  = Func_theRTgaussCdf->GetX(integTails/2.);
//      XRightLim = Func_theRTgaussCdf->GetX(1-integTails/2.);
     XLeftSet  = Func_signalCdf->GetX(integTails/2.);
     XRightSet = Func_signalCdf->GetX(1-integTails/2.);
     tagged_mass->setRange("2sigmaRTintegral",XLeftSet,XRightSet);
     RooAbsReal* BckgInt2SigmaRT = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"2sigmaRTintegral");
     RooAbsReal* SignInt2SigmaRT = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"2sigmaRTintegral");
     NBckgInt2Sigma = BckgInt2SigmaRT->getVal()*nbkg.getVal();
     NSignInt2Sigma = SignInt2SigmaRT->getVal()*nsig.getVal();
     std::cout<<Form("Bckg event %f<mass<%f from mean RT = %f",XLeftSet,XRightSet,NBckgInt2Sigma) <<std::endl;
     std::cout<<Form("Sign event %f<mass<%f from mean RT = %f",XLeftSet,XRightSet,NSignInt2Sigma) <<std::endl;
     std::cout<<Form("Sign %% in %f<mass<%f from mean RT = %f",XLeftSet,XRightSet,SignInt2SigmaRT->getVal()) <<std::endl;
//==============================================================
//===
//===   MODIFICA della scelta delle SideBand
//===
//==============================================================
//
     if(SigmaProbSign==0){
      std::cout<<Form("FitMassSpectrumRoofit::WARNING: Limits from sigma gauss model")<<std::endl;
      XMinSBL = B0Mass - NSigma1L*B0SigmaTemp;
      XMaxSBL = B0Mass - NSigma2L*B0SigmaTemp;
//      
      XMinSBR = B0Mass + NSigma1R*B0SigmaTemp;
      XMaxSBR = B0Mass + NSigma2R*B0SigmaTemp;
//
     }else if(SigmaProbSign==-1){
      std::cout<<Form("FitMassSpectrumRoofit::WARNING: setting bare limits")<<std::endl;
      XMinSBL = NSigma1L;
      XMaxSBL = NSigma2L;
//      
      XMinSBR = NSigma1R;
      XMaxSBR = NSigma2R;
     }else if(SigmaProbSign==1){
      std::cout<<Form("FitMassSpectrumRoofit::WARNING: Limits from gauss sign prob")<<std::endl;
      integTails= TMath::Erfc(NSigma2L/sqrt(2));
      XMaxSBL	= Func_signalCdf->GetX(integTails/2.);//  Erfc(x) = (2/sqrt(pi)) Integral(exp(-t^2))dt between x and infinity; t=x/sqrt(2)
//
      std::cout<<Form("Sideband Left  Internal Limit&Prob [%f - %f]",XMaxSBL, integTails/2.)<<std::endl;
      
//       integTails= TMath::Erfc(fabs(NSigma2L-NSigma1L)/sqrt(2));
//       XMinSBL	= XMaxSBL-(fabs(mean_rt->getVal()-Func_signalCdf->GetX(integTails/2.)));
      XMinSBL   = NSigma1L;
      integTails= TMath::Erfc(NSigma1R/sqrt(2));
      XMinSBR	= Func_signalCdf->GetX(1-integTails/2.);
      std::cout<<Form("Sideband Right Internal Limit&Prob [%f - %f]",XMinSBR, integTails/2.)<<std::endl;
//       integTails= TMath::Erfc(fabs(NSigma2R-NSigma1R)/sqrt(2));
//       XMaxSBR	= XMinSBR+(fabs(mean_rt->getVal()-Func_signalCdf->GetX(1-integTails/2.)));
      XMaxSBR   = NSigma2R;
     }else{
      std::cout<<Form("Sideband Definition=>>SigmaProbSign: 	INVALID OPTION: %d !!! Exit...",SigmaProbSign)<<std::endl;
      exit(1);
     } 
//     
     std::cout<<Form("Sideband Left  [%f-%f]",XMinSBL, XMaxSBL)<<std::endl;
     std::cout<<Form("Sideband Right [%f-%f]",XMinSBR, XMaxSBR)<<std::endl;

     ccdf_signal ->cd();
     Func_signal->SetRange(XMinSign,XMaxSign);
//     Func_signal->SetNormalized(true);
     TF1* Clone_signal  =  (TF1*) Func_signal->Clone();
     TF1* Clone_signalL =  (TF1*) Func_signal->Clone();
     TF1* Clone_signalR =  (TF1*) Func_signal->Clone();
//     Clone_signal->SetNormalized(true);
//     double Func_signal_Integ = Clone_signal->Integral(5.0,5.6);
//     std::cout<<"Func_signal_Integ ="<<Func_signal_Integ<<std::endl;
//     Clone_signal->GetHistogram()->Scale(1/Func_signal_Integ);
     Clone_signal->SetMaximum(0.1);
     Clone_signal->Draw();
//     Clone_signalL->SetNormalized(true);
//     double Func_signal_IntegL = Clone_signal->Integral(XMinSBL,XMaxSBL);
     Clone_signalL->SetRange(XMinSBL,XMaxSBL);
//     std::cout<<"Func_signal_IntegL ="<<Func_signal_IntegL<<std::endl;
//     Clone_signalL->GetHistogram()->Scale(Func_signal_IntegL/Func_signal_Integ);
     Clone_signalL->SetFillColor(kBlue);
     Clone_signalL->SetFillStyle(3013);
     Clone_signalL->SetLineWidth(1.);
     Clone_signalL->Draw("SAME FC");
//     Clone_signalR->SetNormalized(true);
//     double Func_signal_IntegR = Clone_signal->Integral(XMinSBR,XMaxSBR);
     Clone_signalR->SetRange(XMinSBR,XMaxSBR);
//     std::cout<<"Func_signal_IntegR ="<<Func_signal_IntegR<<std::endl;
//     Clone_signalR->GetHistogram()->Scale(Func_signal_IntegR/Func_signal_Integ);
     Clone_signalR->SetFillColor(kBlue);
     Clone_signalR->SetFillStyle(3013);
     Clone_signalR->SetLineWidth(1.);
     Clone_signalR->Draw("SAME FC");
     ccdf_signal ->Print(Form("cdf_signal_Q2Bin_%d.png",Q2Bin));
//     std::cout<<Form("Func_signal_IntegL%=%f Func_signal_IntegR%=%f",Func_signal_IntegL/Func_signal_Integ,Func_signal_IntegR/Func_signal_Integ)<<std::endl;
//exit(0);
//     RooRealVar* tagged_massS = new RooRealVar("tagged_mass" , "#mu^{+}#mu^{-}K#pi mass", 5.0, 5.6, "GeV");
//     RooRealVar* tagged_massF = new RooRealVar("tagged_mass" , "#mu^{+}#mu^{-}K#pi mass", 4.9, 5.6, "GeV");
//     tagged_mass->setRange(XMinSign,XMaxSign);
//     tagged_mass->setRange("full",XMinSign,XMaxSign);
     tagged_mass->setRange( "SBLeft"  ,XMinSBL,XMaxSBL);
     tagged_mass->setRange( "SBRight" ,XMinSBR,XMaxSBR);
//      tagged_massS->setRange("SBLeftS" ,XMinSBL,XMaxSBL);
//      tagged_massS->setRange("SBRightS",XMinSBR,XMaxSBR);
//      tagged_massF->setRange("SBLeftF" ,XMinSBL,XMaxSBL);
//      tagged_massF->setRange("SBRightF",XMinSBR,XMaxSBR);
     
     double tagged_mass_rangeFitMin=tagged_mass->getMin("full");
     double tagged_mass_rangeFitMax=tagged_mass->getMax("full");
     tagged_mass_rangeValMin=tagged_mass->getMin();
     tagged_mass_rangeValMax=tagged_mass->getMax();
     std::cout<<Form("%f<[tagged_mass fit mass range]<%f",tagged_mass_rangeFitMin,tagged_mass_rangeFitMax)<<std::endl; 
     std::cout<<Form("%f<[tagged_mass val mass range]<%f",tagged_mass_rangeValMin,tagged_mass_rangeValMax)<<std::endl; 
     
//     RooExponential* bkgEXP = new RooExponential(bkg_exp);
  
//      RooAbsReal* BckgAll = bkg_exp->createIntegral(*tagged_mass);
//      RooAbsReal* BckgFull = bkg_exp->createIntegral(*tagged_massF);
//      RooAbsReal* BckgFits = bkg_exp->createIntegral(*tagged_massS);
//      std::cout<<Form("Integrals All=%f Fits=%f and Full=%f",BckgAll->getVal(),BckgFits->getVal(),BckgFull->getVal())<<std::endl; 
     
     RooAbsReal* BckgSBL = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"SBLeft");
//     RooAbsReal* BckgSBL = bkg_exp->createIntegral(*tagged_mass,NormSet(*tagged_mass),Range("SBLeft"));
     RooAbsReal* BckgSBR = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"SBRight");
     double BckgEventsSBL = BckgSBL->getVal()*nbkg.getVal();
     double BckgEventsSBR = BckgSBR->getVal()*nbkg.getVal();
//     

     RooAbsReal* SignSBL = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"SBLeft");
     RooAbsReal* SignSBR = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"SBRight");
     double SignEventsSBL = SignSBL->getVal()*nsig.getVal();
     double SignEventsSBR = SignSBR->getVal()*nsig.getVal();
//     
     RooAbsReal* SignSBL_wt = c_WTgauss->createIntegral(*tagged_mass,*tagged_mass,"SBLeft");
     RooAbsReal* SignSBR_wt = c_WTgauss->createIntegral(*tagged_mass,*tagged_mass,"SBRight");
     double SignEventsSBL_wt = SignSBL_wt->getVal()*nsig.getVal()*(fractionWT->getVal());
     double SignEventsSBR_wt = SignSBR_wt->getVal()*nsig.getVal()*(fractionWT->getVal());
//      double SignEventsSBL_wt = SignSBL_wt->getVal()*nsig.getVal()*(1-fraction->getVal());
//      double SignEventsSBR_wt = SignSBR_wt->getVal()*nsig.getVal()*(1-fraction->getVal());
//     
     RooAbsReal* SignSBL_rt = c_RTgauss->createIntegral(*tagged_mass,*tagged_mass,"SBLeft");
     RooAbsReal* SignSBR_rt = c_RTgauss->createIntegral(*tagged_mass,*tagged_mass,"SBRight");
     double SignEventsSBL_rt = SignSBL_rt->getVal()*nsig.getVal()*(1.-fractionWT->getVal());
     double SignEventsSBR_rt = SignSBR_rt->getVal()*nsig.getVal()*(1.-fractionWT->getVal());
//     
     RooAbsReal* ModelSBL = fitFunction.createIntegral(*tagged_mass,*tagged_mass,"SBLeft");
     RooAbsReal* ModelSBR = fitFunction.createIntegral(*tagged_mass,*tagged_mass,"SBRight");
     double ModelEventsSBL = ModelSBL->getVal()*(nsig.getVal()+nbkg.getVal());
     double ModelEventsSBR = ModelSBR->getVal()*(nsig.getVal()+nbkg.getVal());
     double RealEventsSBL  = data->sumEntries(Form("tagged_mass>%f&&tagged_mass<%f",XMinSBL,XMaxSBL)) ;
     double RealEventsSBR  = data->sumEntries(Form("tagged_mass>%f&&tagged_mass<%f",XMinSBR,XMaxSBR)) ;
//
     RooAbsReal* SignalFull  = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"full");
//     RooAbsReal* BckgFull    = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"full");
     RooAbsReal* ModelFull   = fitFunction.createIntegral(*tagged_mass,*tagged_mass,"full");

//     
     std::cout<<Form("Signal Norm Check = %f",SignalFull->getVal())  <<std::endl;
     std::cout<<Form("Estimated Bckg	events SB Left = %f",BckgEventsSBL)  <<std::endl;
     std::cout<<Form("Estimated Bckg	events SB Right= %f",BckgEventsSBR)  <<std::endl;
     std::cout<<Form("Estimated Bckg	events SB Total= %f",BckgEventsSBL+BckgEventsSBR)  <<std::endl;
     std::cout<<Form("========> real	events SB Left = %f",RealEventsSBL ) <<std::endl;
     std::cout<<Form("========> real	events SB Right= %f",RealEventsSBR ) <<std::endl;
     std::cout<<Form("========> real	events SB Total= %f",RealEventsSBL+RealEventsSBR ) <<std::endl;
     std::cout<<Form("========> real	all events inside [full] range Total= %f",data->sumEntries(Form("tagged_mass>%f&&tagged_mass<%f",tagged_mass_rangeMin,tagged_mass_rangeMax)) ) <<std::endl;
     std::cout<<Form("========> Fit	all events inside [full] range Total= %f",ModelFull->getVal()*(nsig.getVal()+nbkg.getVal())) <<std::endl;
     std::cout<<Form("========> real	all events Total= %f",data->sumEntries(Form("tagged_mass>%f&&tagged_mass<%f",XMinSign,XMaxSign)) ) <<std::endl;
     std::cout<<Form("========> Fit	all events Total= %f",(nsig.getVal()+nbkg.getVal())) <<std::endl;
//
     std::cout<<Form("Estimated Sign	events SB Left = %f",SignEventsSBL)  <<std::endl;
     std::cout<<Form("Estimated Sign	events SB Right= %f",SignEventsSBR)  <<std::endl;
     std::cout<<Form("Estimated Sign WT events SB Left = %f",SignEventsSBL_wt)  <<std::endl;
     std::cout<<Form("Estimated Sign WT	events SB Right= %f",SignEventsSBR_wt)  <<std::endl;
     std::cout<<Form("Estimated Sign RT events SB Left = %f",SignEventsSBL_rt)  <<std::endl;
     std::cout<<Form("Estimated Sign RT	events SB Right= %f",SignEventsSBR_rt)  <<std::endl;
     std::cout<<Form("Estimated Model	events SB Left = %f",ModelEventsSBL) <<std::endl;
     std::cout<<Form("Estimated Model	events SB Right= %f",ModelEventsSBR) <<std::endl;
     std::cout<<Form("events signal in SB [all]/ signal Tot  = %f",(SignEventsSBL+SignEventsSBR)/ nsig.getVal())<<std::endl;
     std::cout<<Form("events signal in SB [RT] / signal Tot  = %f",(SignEventsSBL_rt+SignEventsSBR_rt)/ nsig.getVal())<<std::endl;
     std::cout<<Form("events signal in SB [WT] / signal Tot  = %f",(SignEventsSBL_wt+SignEventsSBR_wt)/ nsig.getVal())<<std::endl;
     std::cout<<Form("events signal in SB [all]/ SB     Tot  = %f",(SignEventsSBL+SignEventsSBR)/ (RealEventsSBL+RealEventsSBR))<<std::endl;
     std::cout<<Form("events signal in SB [RT] / SB     Tot  = %f",(SignEventsSBL_rt+SignEventsSBR_rt)/ (RealEventsSBL+RealEventsSBR))<<std::endl;
     std::cout<<Form("events signal in SB [WT] / SB     Tot  = %f",(SignEventsSBL_wt+SignEventsSBR_wt)/ (RealEventsSBL+RealEventsSBR))<<std::endl;
     std::cout<<Form("events signal in SB Left / signal Tot  = %f",(SignEventsSBL)/ nsig.getVal())<<std::endl;
     std::cout<<Form("events signal in SB Right/ signal Tot  = %f",(SignEventsSBR)/ nsig.getVal())<<std::endl;
     std::cout<<Form("events signal in SB Left / SB     Tot  = %f",(SignEventsSBL)/ (RealEventsSBL+RealEventsSBR))<<std::endl;
     std::cout<<Form("events signal in SB Right/ SB     Tot  = %f",(SignEventsSBR)/ (RealEventsSBL+RealEventsSBR))<<std::endl;
     std::cout<<Form("events signal in SB Left / SB     Left = %f",(SignEventsSBL)/ (RealEventsSBL))<<std::endl;
     std::cout<<Form("events signal in SB Right/ SB     Right= %f",(SignEventsSBR)/ (RealEventsSBR))<<std::endl;
     std::cout<<"\n"<<std::endl;
     std::cout<<Form("%d &  %3.2f-%3.2f & %3.2f-%3.2f & %5.3f & %5.3f & %5.3f & %5.3f & %5.3f & %5.3f & %5.3f & %3.2f\\\\ \n",Q2Bin,XMinSBL,XMaxSBL,XMinSBR,XMaxSBR,\
     (SignEventsSBL)/nsig.getVal(),(SignEventsSBR)/nsig.getVal(), (SignEventsSBL)/ (RealEventsSBL),\
     (SignEventsSBR)/ (RealEventsSBR),\
     (SignEventsSBL_rt+SignEventsSBR_rt)/(RealEventsSBL+RealEventsSBR),\
     (SignEventsSBL_wt+SignEventsSBR_wt)/(RealEventsSBL+RealEventsSBR),\
     (SignEventsSBL+SignEventsSBR)/(RealEventsSBL+RealEventsSBR),\
     (RealEventsSBL+RealEventsSBR)/NBckgInt2Sigma) <<std::endl;
//     
     
     double xbinw = pdfHist->GetXaxis()->GetBinWidth(1);
     cout<<"Binw pdfHist ="<<xbinw<<endl;
//      for (int i = 1; pdfHist->GetNbinsX(); ++i) {
//         double xmass = xbinw/2.+(i-1)*xbinw;
// //      const RooArgSet * dataLoad = data->get (i);
// //      double xmass = dataLoad->getRealValue(tagged_mass->GetName());
// 	pdfHist->Fill(xmass, fitFunction.evaluate() );
// 	sigHist->Fill(xmass, c_signalFunction.evaluate());
// 	bkgHist->Fill(xmass, c_frt.evaluate());
//      }
     double NStepMass  = pdfHist->GetNbinsX();
     double NBINFactor = NStepMass/masHist->GetNbinsX();
     fitFunction.fillHistogram(pdfHist,*tagged_mass);
     c_signalFunction.fillHistogram(sigHist,*tagged_mass);
     bkg_exp->fillHistogram(bkgHist,*tagged_mass);
     tagged_mass->setRange(XMinSign,XMaxSign);
     RooAbsReal* BckgFullW   = bkg_exp->createIntegral(*tagged_mass,*tagged_mass);
     RooAbsReal* SignalFullW = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass);
     RooAbsReal* ModelFullW  = fitFunction.createIntegral(*tagged_mass,*tagged_mass);
     
//     tagged_mass->setRange("full1",5.0,5.6);
     RooAbsReal* BckgFullS   = bkg_exp->createIntegral(*tagged_mass,*tagged_mass,"full");
     RooAbsReal* SignalFullS = c_signalFunction.createIntegral(*tagged_mass,*tagged_mass,"full");
     RooAbsReal* ModelFullS  = fitFunction.createIntegral(*tagged_mass,*tagged_mass,"full");
     
     double scaleModelW  = ModelFullW->getVal()/ModelFullS->getVal();
     double scaleSignalW = SignalFullW->getVal()/SignalFullS->getVal();
     double scaleBckgW   = BckgFullW->getVal()/BckgFullS->getVal();

     int MinBinRangeFit = pdfHist->GetXaxis()->FindBin(tagged_mass_rangeMin);
     int MaxBinRangeFit = pdfHist->GetXaxis()->FindBin(tagged_mass_rangeMax);
     std::cout<<Form("pdfHist (=model of mass spectrum) MinBinRangeFit = %d MaxBinRangeFit = %d NumBins = %d",MinBinRangeFit,MaxBinRangeFit,pdfHist->GetNbinsX()) <<std::endl;
//      pdfHist->Scale(NBINFactor*(nsig.getVal()+nbkg.getVal())*scaleModelW);
//      sigHist->Scale(NBINFactor*(nsig.getVal())*scaleSignalW);
//      bkgHist->Scale(NBINFactor*(nbkg.getVal())*scaleBckgW);
     pdfHist->Scale(NBINFactor*(nsig.getVal()+nbkg.getVal()));
     sigHist->Scale(NBINFactor*(nsig.getVal()));
     bkgHist->Scale(NBINFactor*(nbkg.getVal()));
//     cout<< Form("Integ Model (4.9-5.6) =%f  Integ Model (5.0-5.6) =%f scaleModelW=%f", ModelFullW->getVal(),ModelFullS->getVal(),scaleModelW)<<endl;
//     cout<< Form("Integ Signal(4.9-5.6) =%f  Integ Signal(5.0-5.6) =%f scaleSignalW=%f", SignalFullW->getVal(),SignalFullS->getVal(),scaleSignalW)<<endl;
//     cout<< Form("Integ Bckg  (4.9-5.6) =%f  Integ Bckg  (5.0-5.6) =%f scaleBckgW=%f", BckgFullW->getVal(),BckgFullS->getVal(),scaleBckgW)<<endl;
     cout<< Form("Integ Model (%f-%f) =%f  Integ Model (%f-%f) =%f scaleModelW=%f" ,XMinSign,XMaxSign,ModelFullW->getVal() ,tagged_mass_rangeMin,tagged_mass_rangeMax,ModelFullS->getVal() ,scaleModelW)<<endl;
     cout<< Form("Integ Signal(%f-%f) =%f  Integ Signal(%f-%f) =%f scaleSignalW=%f",XMinSign,XMaxSign,SignalFullW->getVal(),tagged_mass_rangeMin,tagged_mass_rangeMax,SignalFullS->getVal(),scaleSignalW)<<endl;
     cout<< Form("Integ Bckg  (%f-%f) =%f  Integ Bckg  (%f-%f) =%f scaleBckgW=%f"  ,XMinSign,XMaxSign,BckgFullW->getVal()  ,tagged_mass_rangeMin,tagged_mass_rangeMax,BckgFullS->getVal(),scaleBckgW)<<endl;
//exit(1);
   //   int MinBinSBL = bkgHist->GetXaxis()->FindBin(mean_rt->getVal()-3*B0SigmaTemp);
//      int MaxBinSBL = bkgHist->GetXaxis()->FindBin(mean_rt->getVal()+3*B0SigmaTemp);
//      double BckgIntSBL= nbkg.getVal()*bkgHist->Integral(MinBinSBL,MaxBinSBL)/bkgHist->Integral(MinBinRangeFit,MaxBinRangeFit);
//      std::cout<<Form("pdfHist (=model of mass spectrum) MinBinSBL = %d MaxBinSBL = %d Integ = %f",MinBinSBL,MaxBinSBL,BckgIntSBL) <<std::endl;
//  
     masHist->GetXaxis()->SetTitle("Mass (GeV/c^{2})");
     masHist->SetMarkerStyle(8);
     masHist->SetMarkerSize(MarkerSizeSet);
     masHist->SetTitle("");
     masHist->Draw("E1");
//     masHist.Draw("p");
     pdfHist->GetXaxis()->SetRangeUser(tagged_mass_rangeMin,tagged_mass_rangeMax);
     sigHist->GetXaxis()->SetRangeUser(tagged_mass_rangeMin,tagged_mass_rangeMax);
     bkgHist->GetXaxis()->SetRangeUser(tagged_mass_rangeMin,tagged_mass_rangeMax);
     pdfHist->SetLineWidth(PlotLineWidth);
     pdfHist->SetFillColor(0);
     pdfHist->SetLineColor(kBlue);
     pdfHist->Draw("same,HIST C");
     sigHist->SetLineWidth(PlotLineWidth);
     sigHist->SetLineColor(kMagenta);
     sigHist->SetLineStyle(kDashed);
     sigHist->SetFillColor(0);
     sigHist->Draw("same,HIST C");
     bkgHist->SetLineWidth(PlotLineWidth);
     bkgHist->SetLineColor(kRed);
     bkgHist->SetLineStyle(kDashed);
     bkgHist->SetFillColor(0);
     bkgHist->Draw("same,HIST C");
     leg_sign->Draw("same");
  
  fitMassFile->Close();
  return B0SigmaTemp;

}
//=========================================================================================

RooGaussian* _constrainVar(RooRealVar *var,RooWorkspace *w){
    
//    float constr[2] = *_getFittedVar(var.GetName(), w);
    RooRealVar c_val(Form("c_val_%s", var->GetName()),Form("c_val_%s", var->GetName()),var->getVal());
    RooRealVar c_err(Form("c_err_%s", var->GetName()),Form("c_err_%s", var->GetName()),var->getError());
    RooGaussian* gauss_constr =
                            new RooGaussian(   Form("c_%s", var->GetName()) , 
                                Form("c_%s", var->GetName()) , 
                                *var         ,  
                                RooConst( var->getVal() ), 
                                RooConst( var->getError() ) 
                                ) ;
    std::cout<< Form("constraining var %s: %f with uncertainty %f - limits [%f , %f]",var->GetName(),c_val.getVal(),c_err.getVal(),var->getMin(),var->getMax())<<std::endl;  
    if(Q2Bin==3&&RunEra==2016){
    double checkMax = var->getVal()+ 7*var->getError();			      
    double checkMin = var->getVal()- 7*var->getError();			      
    std::cout<< Form("Warning in _constrainVar: limits for %s, from [%f,%f] ==> [%f,%f]\n",var->GetName(),var->getMin(),var->getMax(),checkMin,checkMax);
    var->setMax(checkMax) ;	     
    var->setMin(checkMin) ;	     
    std::cout<< Form("Warning: redifine limits var %s: %f with uncertainty %f - limits [%f , %f]",var->GetName(),c_val.getVal(),c_err.getVal(),var->getMin(),var->getMax())<<std::endl;  
    }                        
    return gauss_constr;
}                           
//=========================================================================================
//
//=========================================================================================
// float*  _getFittedVar(const char* varName,RooWorkspace w=0){
//     float out[2];
//     if (&w!=0){
//         out[0]=w->var(varName).getVal();
// 	out[1]=w->var(varName).getError();
//     }else{
//         out[0]=varName.getVal();
// 	out[1]=varName.getError();
//     }
//     return out;	
// }
// double FitMassSpectrumRoofit(UnbinnedDataSet* dataMass, TCanvas* c2, TH1D* masHist, TH1D* pdfHist, TH1D*sigHist, TH1D* bkgHist, int MaxDegreeBckg){
// //
//   if(MaxDegreeBckg<=0) {
//    cout<<"**********************************************************************\n"<<endl;
//    cout<<"Error!! MaxDegree <=0 in   FitMassSpectrumRoofit		       *\n"<<endl;
//    cout<<"**********************************************************************\n"<<endl;
//   }
// //
//   double B0SigmaTemp = 0.
//   double xmeanRTSign = 5.280;
//   RooFit::RooRealVar meanRTSign("meanRTSign"  ,xmeanRTSign,XStepMinuit, 5., 5.5);
//   RooFit::RooRealVar mean_wtSign("mean_wtSign"  ,xmeanRTSign,XStepMinuit, 5., 5.5);
//   RooFit::RooRealVar sigmaRTSign1("sigmaRTSign1",0.0139,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar sigmaRTSign2("sigmaRTSign2",0.0228,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar sigmaRTSign3("sigmaRTSign3",0.0601,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar sigma_wtSignCB1("sigma_wtSignCB1",0.0139,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar sigma_wtSignCB2("sigma_wtSignCB2",0.0228,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar alpha1("alpha1",0.0139,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar alpha2("alpha2",0.0228,XStepMinuit, 0., 1.);
//   RooFit::RooRealVar RTwg1("RTwg1",0.44, 0., 1.);
//   RooFit::RooRealVar RTwg2("RTwg2",0.5 , 0., 1.);
//   RooFit::RooRealVar WTwg1("WTwg1",0.44, 0., 1.);
//   RooFit::RooRealVar WTwg2("WTwg2",0.5 , 0., 1.);
// //
//   RooGaussian *gaussRTSign1 = new RooGaussian("gaussRTSign1","gauss Right Tagged 1"    ,xMassRoofit,meanRTSign,sigmaRTSign1) ;
//   RooGaussian *gaussRTSign2 = new RooGaussian("gaussRTSign2","gauss Right Tagged 2"    ,xMassRoofit,meanRTSign,sigmaRTSign2) ;
//   RooGaussian *gaussRTSign3 = new RooGaussian("gaussRTSign3","gauss Right Tagged 3"    ,xMassRoofit,meanRTSign,sigmaRTSign3) ;
// //  
//   
//   RooGaussian *cstr_sigmaRTSign1 = new RooGaussian("cstr_sigmaRTSign1","sigma1 MC constr." ,xMassRoofit,sigmaRTSign1MC,sigmaRTSign1MC_err) ;
//   RooGaussian *cstr_sigmaRTSign2 = new RooGaussian("cstr_sigmaRTSign2","sigma2 MC constr." ,xMassRoofit,sigmaRTSign2MC,sigmaRTSign2MC_err) ;
//   RooGaussian *cstr_sigmaRTSign3 = new RooGaussian("cstr_sigmaRTSign3","sigma3 MC constr." ,xMassRoofit,sigmaRTSign3MC,sigmaRTSign3MC_err) ;
// //
// //
//   RooGaussian *cstr_meanRTSign1  = new RooGaussian("cstr_meanRTSign1" , "mean1 MC constr." ,xMassRoofit,meanRTSign1MC,meanRTSign1MC_err) ;
//  
//   RooCBShape *crystalBallWTSign1 = new RooCBShape("crystalBallWTSign1", "crystalBall Wrong Tagged Sign1",xMassRoofit,mean_wtSign,sigma_wtSign1,alpha1,n1);
//   RooCBShape *crystalBallWTSign2 = new RooCBShape("crystalBallWTSign2", "crystalBall Wrong Tagged Sign2",xMassRoofit,mean_wtSign,sigma_wtSign2,alpha2,n2);
//   RooGaussian *gaussWTSign = new RooGaussian("gaussWTSign","gauss Right Tagged"    ,xMassRoofit,mean_wtSign,sigma_wtSign) ;
//   
// // Signal Mass RT  
//   RooAddPdf * RTSignalMass = 0;
//   if     (numRTGau=1){
//    RTSignalMass = &gaussRTSign1;
//   else if(numRTGau=2){
//    RTSignalMass = new RooAddPdf("RTSignalMass","RTSignalMass",RooArgList(*gaussRTSign1,*gaussRTSign1),RooArgList(RTwg1));
//   else if(numRTGau=3){
//    RTSignalMass = new RooAddPdf("RTSignalMass","RTSignalMass",RooArgList(*gaussRTSign1,*gaussRTSign2,*gaussRTSign3),RooArgList(RTwg1,RTwg2));
//   } 
// //
//   RooAddPdf * WTSignalMass = 0;
//   if     (numWTMod=1){
//    WTSignalMass = &gaussWTSign1;
//   else if(numWTMod=2){
//    WTSignalMass = new RooAddPdf("WTSignalMass","WTSignalMass",RooArgList(*gaussWTSign,crystalBallWTSign1),RooArgList(WTwg1));
//   else if(numWTMod=3){
//    WTSignalMass = new RooAddPdf("WTSignalMass","WTSignalMass",RooArgList(*gaussWTSign1,*gaussWTSign2,*gaussWTSign3),RooArgList(WTwg1,WTwg2));
//   } 
// //
//    slope	 = RooRealVar	 ("slope"      , "slope"	   ,	0.5,   -10, 10);
//    bkg_exp	 = RooExponential("bkg_exp"    , "exponential"     ,  slope,   tagged_mass  );
//    pol_c1	 = RooRealVar	 ("p1"         , "coeff x^0 term"  ,	0.5,   -10, 10);
//    pol_c2	 = RooRealVar	 ("p2"         , "coeff x^1 term"  ,	0.5,   -10, 10);
//    bkg_pol	 = RooChebychev  ("bkg_pol"    , "2nd order pol"   ,  tagged_mass, RooArgList(pol_c1,pol_c2));
//   
//    nsig 	 = RooRealVar("Yield"	      , "signal frac"	 ,    4000,	0,   1000000);
//    nbkg 	 = RooRealVar("nbkg"	      , "bkg fraction"   ,    1000,	0,   550000);
// 
//    return B0SigmaTemp;
// //
// }


double FitMassSpectrum(UnbinnedDataSet* dataMass, TCanvas* c2, TH1D* masHist, TH1D* pdfHist, TH1D*sigHist, TH1D* bkgHist, int MaxDegreeBckg){

   ///////////////////////////////////////////////////////
   //*****************************************************
   //*
   //*
   //*  	    FIT Mass Spectrum
   //*   
   //*
   //*****************************************************
   ///////////////////////////////////////////////////////
   
     if(MaxDegreeBckg<=0) {
      cout<<"**********************************************************************\n"<<endl;
      cout<<"Error!! MaxDegree <=0 in   FitMassSpectrum                           *\n"<<endl;
      cout<<"**********************************************************************\n"<<endl;
     }
     GooFit::Variable mean  ("mean"  ,5.2762,XStepMinuit, 5., 5.5);
     GooFit::Variable sigma1("sigma1",0.0139,XStepMinuit, 0., 1.);
     GooFit::Variable sigma2("sigma2",0.0228,XStepMinuit, 0., 1.);
     GooFit::Variable sigma3("sigma3",0.0601,XStepMinuit, 0., 1.);
     GooFit::Variable wg1("wg1",0.44, 0., 1.);
     GooFit::Variable wg2("wg2",0.5 , 0., 1.);

     

     GooFit::RGaussianPdf* signalMass1 = new GooFit::RGaussianPdf("signalMass1", xMass, mean, sigma1);
     GooFit::RGaussianPdf* signalMass2 = new GooFit::RGaussianPdf("signalMass2", xMass, mean, sigma2);

     std::vector<GooFit::Variable> weightsSignMass;
     weightsSignMass.push_back(wg1);
     std::vector<PdfBase*> compsSignMass;
     compsSignMass.push_back(signalMass1);
     compsSignMass.push_back(signalMass2);
     GooFit::AddPdf* signalMass= new AddPdf("signalMass", weightsSignMass, compsSignMass);
     GooFit::Variable* signalYield = new GooFit::Variable("signalYield",40000*NFactGen,	     0.,10000000.);
     GooFit::Variable* bckgYield   = new GooFit::Variable("bckgYield"  ,60000*NFactGen ,      0.,20000000.);

     
     GooFit::Variable p0("p0",-3.08433e-01,-1.,1. ); 
     GooFit::Variable p1("p1"	       ,0.,-1.,1. ); 
     GooFit::Variable VMinSign("VMinSign",XMinSign ); 
     GooFit::Variable VMaxSign("VMaxSign",XMaxSign ); 
     SimpleCheby2Pdf* SimpleCheby2  = new SimpleCheby2Pdf("SimpleCheby2", xMass, p0, p1,VMinSign,VMaxSign);
     GooFit::Variable ps0("ps0",1.58302e+01, 8.5 , 30.); 
     GooFit::Variable ps1("ps1",5.11588e+00,   4.0,5.2);
     GooFit::Variable ps2("ps2",1.); 
     GooFit::Variable ps3("ps3",0.);
     ErfcMassPdf* ErfcMassBckg = new ErfcMassPdf("ErfcMassBckg",xMass,ps0,ps1,ps2,ps3);;
     
      GooFit::Variable        wb1("wb1",0.3, 0., 1.);
      std::vector<GooFit::Variable> weightsBckgMass;
      weightsBckgMass.push_back(wb1);
   //	weightsBckgMass.push_back(wb2);
   // //  weightsBckgMass.push_back(wb3);
   // 
   // //  ArgusPdf* argus = new  ArgusPdf("argus", xMass, treshold, aslope, true, apower);  
   // 
      std::vector<PdfBase*> compsBckgMass;
   // //  compsBckgMass.push_back(gaussBckgB0);
   // //  compsBckgMass.push_back(gaussBckgB0);
   // //    compsBckgMass.push_back(argus);
   // //  compsBckgMass.push_back(poly);
      compsBckgMass.push_back(SimpleCheby2);
      compsBckgMass.push_back(ErfcMassBckg);
   //  
   //	GooFit::AddPdf* bckgMass= new AddPdf("bckgMass", weightsBckgMass, compsBckgMass);
   //
     GooFit::Variable b0("b0",	1.      );
     GooFit::Variable b1("b1",	0.     ,0.,1000. );
     GooFit::Variable b2("b2",	0.     ,0.,1000. );
     GooFit::Variable b3("b3",	0.     ,0.,1000. );
     GooFit::Variable b4("b4",	0.      );
     GooFit::Variable b5("b5",	0.     ,0.,1000. );
//      GooFit::Variable b6("b6"	       ,0.,0.,100. ); 
//      GooFit::Variable b7("b7"	       ,0.,0.,100. ); 

     std::vector<GooFit::Variable> ParBernBckg;
     ParBernBckg.push_back(b0);
     if(MaxDegreeBckg>=1) ParBernBckg.push_back(b1);
     if(MaxDegreeBckg>=2) ParBernBckg.push_back(b2);
     if(MaxDegreeBckg>=3) ParBernBckg.push_back(b3);
     if(MaxDegreeBckg>=4) ParBernBckg.push_back(b4);
     if(MaxDegreeBckg>=5) ParBernBckg.push_back(b5);
     std::vector<GooFit::Variable> Limits1D;
     Limits1D.push_back(VMinSign);
     Limits1D.push_back(VMaxSign);
   //
     FastBernsteinPdf* bckgMass  = new FastBernsteinPdf("bckgMass", xMass, ParBernBckg, Limits1D,MaxDegreeBckg);
     
     std::vector<PdfBase*> compsMass;
     compsMass.push_back(signalMass);
     compsMass.push_back(bckgMass);

     std::vector<GooFit::Variable> weightsYield;
     weightsYield.push_back(*signalYield);
     weightsYield.push_back(*bckgYield);

     GooFit::AddPdf modelMass("modelMass", weightsYield, compsMass); 
     modelMass.setData(dataMass);
     
     

   //  GooFit::FitManager fitter(&model);//
     int NumCalls = 12000;

     if(SetMinuit2){
      GooFit::FitManagerMinuit2 fitter(&modelMass);
      fitter.setMaxCalls(NumCalls);
      fitter.setVerbosity(2);
      fitter.fit();
     }else{
      std::cout<<"Warning !!!! bSetting num call for MINUIT :"<<NumCalls  <<std::endl;
      GooFit::FitManagerMinuit1 fitter(&modelMass);
      fitter.setMaxCalls(NumCalls);
      fitter.useHesseBefore(false);
      fitter.useHesse(boolHesse);
      fitter.useMinos(false);
      cout<<"\n"<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"		       ===*** Start Fit ***=== "<<endl;
      cout<<"\n"<<endl;

      Minuit1 * Minuit = fitter.getMinuitObject();
      Minuit->SetPrintLevel(FitPrintLevel);
      fitter.fit();
     }

     UnbinnedDataSet gridMass(xMass);
     double totalDataMass = 0; 
//      double NStepMass = XHScale * xMassHBin;
//      double NBINFactor = NStepMass/xMassHBin2;
     double NStepMass  = pdfHist->GetNbinsX();
     double NBINFactor = NStepMass/masHist->GetNbinsX();
     for (int i = 0; i < NStepMass; ++i) {
       double step = (xMass.getUpperLimit() - xMass.getLowerLimit())/NStepMass;
       xMass.setValue(xMass.getLowerLimit() + (i + 0.5) * step);
       gridMass.addEvent(); 
       totalDataMass++; 
     }

     modelMass.setData(&gridMass);
     std::vector<std::vector<double> > pdfValsMass = modelMass.getCompProbsAtDataPoints();
   //  modelMass.getCompProbsAtDataPoints(pdfValsMass); 
     double totalPdfMass = 0; 
     double totalSigMass = 0; 
     double totalBkgMass = 0; 
     for (int i = 0; i < gridMass.getNumEvents(); ++i) {
       gridMass.loadEvent(i); 
       totalPdfMass += pdfValsMass[0][i]; 
       totalSigMass += pdfValsMass[1][i]; 
       totalBkgMass += pdfValsMass[2][i]; 
     }
     yieldSignal = signalYield->getValue();
     yieldBckg   = bckgYield->getValue();
     double yieldModel  = yieldSignal+yieldBckg;
     for (int i = 0; i < gridMass.getNumEvents(); ++i) {
       gridMass.loadEvent(i); 
       pdfHist->Fill(xMass.getValue(), NBINFactor*yieldModel*pdfValsMass[0][i]/totalPdfMass);
       sigHist->Fill(xMass.getValue(), NBINFactor*yieldSignal*pdfValsMass[1][i]/totalSigMass);
       bkgHist->Fill(xMass.getValue(), NBINFactor*yieldBckg*pdfValsMass[2][i]/totalBkgMass);
     }
     
     std::cout<<"totalPdfMass = "<< totalPdfMass<<std::endl;
     std::cout<<"Signal Yield = "<< yieldSignal<<std::endl;
     std::cout<<"Bckg	Yield = "<< yieldBckg<<std::endl;
     std::cout<<"Tot   Yield  = "<< yieldModel<<std::endl;
     
     double B0SigmaTemp =0.;
     double Sigma1 = sigma1.getValue();
     double Sigma2 = sigma2.getValue();
     double WG1    = wg1.getValue();
 
     c2->cd();
     TLegend* leg_sign = new TLegend(0.30,0.70,0.90,0.90);
     leg_sign->SetTextSize(0.025) ;
     leg_sign->SetTextAlign(31);
     leg_sign->SetBorderSize(0.);
     leg_sign->SetFillStyle(0);
     leg_sign->SetHeader("B^{0} mass spectrum  Fit Projection");
     if(signalYield->getError()!=0){
       leg_sign->AddEntry(masHist ,Form( "Yield_{Sign} =     %5.0f  #pm %5.0f",signalYield->getValue(),signalYield->getError()),"");
     }else{
       leg_sign->AddEntry(masHist ,Form( "Yield_{Sign} =     %5.0f Fixed",signalYield->getValue()),"");
     }
     if(bckgYield->getError()!=0){
       leg_sign->AddEntry(masHist ,Form( "Yield_{Bckg} =     %5.0f  #pm  %5.0f",bckgYield->getValue(),bckgYield->getError()),"");
     }else{
       leg_sign->AddEntry(masHist ,Form( "Yield_{Bckg} =     %5.0f  Fixed",bckgYield->getValue()),"");
     }
 
     if(mean.getError()!=0){
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}} =   %5.5f  #pm %5.5f",mean.getValue(),mean.getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "M_{B^{0}} =   %5.5f Fixed",mean.getValue()),"");
      }
     if(sigma1.getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{1}_{B^{0}} =   %5.5f  #pm %5.5f",sigma1.getValue(),sigma1.getError()),"");
     }else{
      leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{1}_{B^{0}} =   %5.5f Fixed",sigma1.getValue()),"");
     }
     if(sigma2.getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f  #pm %5.5f",sigma2.getValue(),sigma2.getError()),"");
//   	}else{
//   	 leg_sign->AddEntry(masHist ,Form( "#sigma#scale[0.6]{2}_{B^{0}} =   %5.5f Fixed",sigma2.getValue()),"");
     }
     if(wg1.getError()!=XStepMinuit){
      leg_sign->AddEntry(masHist ,Form( "W_{Gaus} =   %5.5f  #pm %5.5f",wg1.getValue(),wg1.getError()),"");
      B0SigmaTemp = sqrt(Sigma1*Sigma1*WG1+(1.-WG1)*Sigma2*Sigma2);
     }else{
      B0SigmaTemp = Sigma1;
     }
     masHist->GetXaxis()->SetTitle("Mass (GeV/c^{2})");
     masHist->SetMarkerStyle(8);
     masHist->SetMarkerSize(MarkerSizeSet);
     masHist->SetTitle("");
     masHist->Draw("E1");
//     masHist.Draw("p");
     pdfHist->SetLineWidth(PlotLineWidth);
     pdfHist->SetFillColor(0);
     pdfHist->SetLineColor(kBlue);
     pdfHist->Draw("same,HIST C");
     sigHist->SetLineWidth(PlotLineWidth);
     sigHist->SetLineColor(kMagenta);
     sigHist->SetLineStyle(kDashed);
     sigHist->SetFillColor(0);
     sigHist->Draw("same,HIST C");
     bkgHist->SetLineWidth(PlotLineWidth);
     bkgHist->SetLineColor(kRed);
     bkgHist->SetLineStyle(kDashed);
     bkgHist->SetFillColor(0);
     bkgHist->Draw("same,HIST C");
     leg_sign->Draw("same");
  
  
  return B0SigmaTemp;

///////////////////////////////////////////////////////
//*****************************************************
//
//  END FIT Mass Spectrum
//
//*****************************************************
///////////////////////////////////////////////////////
  
}
//
//=========================================================================================
//
// Namelist Routine
//
std::map<std::string, std::string> ReadNamelist(int argc, char** argv){
   if ( argc>=1 && (strcmp(argv[0],"namelist")>=0) ){
     std::cout<<"Defined namelist: "<<argv[0]<<std::endl;
   }else{
     std::cout<<"Namelist:"<<argv[0]<<"  should be named/renamed namelist*.list "<<argc<<std::endl;
     exit(1);
   }
   std::vector<std::string> split( char *str, char c = ' ');
   ifstream indata;
   std::map<std::string, std::string> mappa;
   std::string line;
   std::vector<std::string>vstring ;
//
    indata.open(argv[0]);
   if(!indata) { // file couldn't be opened
   	std::cout <<"Line: "<<__LINE__ <<" "<<argv[0]<< " Error: fileList can not be opened" << std::endl;
   	exit(1);
   }
   while(std::getline(indata, line)) {
	 line.erase(std::remove(line.begin(), line.end(), '\t'), line.end());
	 line.erase(std::remove(line.begin(), line.end(), '\n'), line.end());
	 line.erase(std::remove(line.begin(), line.end(), ' ' ), line.end());

 	 char *cstr = new char [line.size()+1];


 	 strcpy (cstr, line.c_str());
//	 cout <<"stringa->"<< cstr << endl;
	 vstring = split(cstr,'=');
	 mappa.insert( std::pair<string,string>(vstring[0],vstring[1]) );
    }
    std::cout<<"//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////"<<std::endl;
    for (map<string,string>::iterator imap = mappa.begin();
    			       imap != mappa.end();
    			       ++imap)
    {
   	std::cout <<"mappa->"<< (*imap).first<<" = "<<(*imap).second << std::endl;
    }
    std::cout<<"//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////"<<std::endl;
    indata.close();	
  return mappa ;
}
//===============================================================================================================
std::vector<std::string> split( char *str, char c = ' ')
{
    std::vector<std::string> result;

    while(1)
    {
         char *begin = str;

        while(*str != c && *str)
                str++;

        result.push_back(string(begin, str));

        if(0 == *str++)
                break;
    }

    return result;
}
//===============================================================================================================
void replaceAll(std::string& str, const std::string& from, const std::string& to) {
    if(from.empty())
        return;
    size_t start_pos = 0;
    while((start_pos = str.find(from, start_pos)) != std::string::npos) {
        str.replace(start_pos, from.length(), to);
        start_pos += to.length(); // In case 'to' contains 'from', like replacing 'x' with 'yx'
    }
}
//===============================================================================================================
void replaceChar(char * txt,const  char * txt1,const  char * txt2) {

  std::stringstream sss,sss1,sss2;
  sss<<txt;
  sss1<<txt1;
  sss2<<txt2;
  std::string ss=sss.str();
  replaceAll( ss,  sss1.str(), sss2.str());
  strcpy(txt,ss.c_str());
  sss.str("");
  sss.clear();
  sss1.str("");
  sss1.clear();
  sss2.str("");
  sss2.clear();
  printf ("replaceChar output=>%s\n",txt);
}  


